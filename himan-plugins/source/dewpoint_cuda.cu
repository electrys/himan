#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "dewpoint_cuda.h"

const double RW = 461.5; // Vesihoyryn kaasuvakio (J / K kg)
const double L = 2.5e6; // Veden hoyrystymislampo (J / kg)
const double RW_div_L = RW / L;

namespace himan
{

namespace plugin
{

namespace dewpoint_cuda
{

__global__ void UnpackAndCalculate(const unsigned char* dTPacked, 
									const unsigned char* dRHPacked,
									double* dT,
									double* dRH,
									double* dTOut,
									dewpoint_cuda_options opts, int* dMissingValuesCount);

__global__ void Calculate(const double* dT, const double* dRH, double* dTD, dewpoint_cuda_options, int* dMissingValueCount);

__device__ void _Calculate(const double* __restrict__ dT, const double* __restrict__ dRH, double* __restrict__ dTD, dewpoint_cuda_options opts, int* dMissingValueCount, int idx);

} // namespace dewpoint
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::dewpoint_cuda::UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dRHPacked,
									double* dT,
									double* dRH,
									double* dTDOut,
									dewpoint_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		SimpleUnpack(dTPacked, dT, opts.N, opts.simplePackedT.bitsPerValue, opts.simplePackedT.binaryScaleFactor, opts.simplePackedT.decimalScaleFactor, opts.simplePackedT.referenceValue, idx);
		SimpleUnpack(dRHPacked, dRH, opts.N, opts.simplePackedRH.bitsPerValue, opts.simplePackedRH.binaryScaleFactor, opts.simplePackedRH.decimalScaleFactor, opts.simplePackedRH.referenceValue, idx);

		_Calculate(dT, dRH, dTDOut, opts, dMissingValuesCount, idx);
	}
}

__global__ void himan::plugin::dewpoint_cuda::Calculate(const double* dT, const double* dRH, double* dTDOut, dewpoint_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		_Calculate(dT, dRH, dTDOut, opts, dMissingValuesCount, idx);
	}
}

__device__ void himan::plugin::dewpoint_cuda::_Calculate(const double* __restrict__ dT,
																const double* __restrict__ dRH,
																double* __restrict__ dTD, dewpoint_cuda_options opts,
																int* dMissingValuesCount, int idx)
{

	if (dT[idx] == kFloatMissing || dRH[idx] == kFloatMissing)
	{
		atomicAdd(dMissingValuesCount, 1);
		dTD[idx] = kFloatMissing;
	}
	else
	{
		dTD[idx] = ((dT[idx]+opts.TBase) / (1 - ((dT[idx]+opts.TBase) * log(dRH[idx]) * (RW_div_L)))) - 273.15 + opts.TBase;
	}
}

void himan::plugin::dewpoint_cuda::DoCuda(dewpoint_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT;
	double* dRH;
	double* dTD;

	unsigned char* dTPacked;
	unsigned char* dRHPacked;
	
	int* dMissingValuesCount;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	CUDA_CHECK(hipMalloc((void **) &dRH, memSize));
	CUDA_CHECK(hipMalloc((void **) &dTD, memSize));

	if (opts.simplePackedT.HasData())
	{
		CUDA_CHECK(hipMalloc((void **) &dTPacked, opts.simplePackedT.dataLength * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dTPacked, opts.simplePackedT.data, opts.simplePackedT.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));		
	}

	if (opts.simplePackedRH.HasData())
	{
		CUDA_CHECK(hipMalloc((void **) &dRHPacked, opts.simplePackedRH.dataLength * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dRHPacked, opts.simplePackedRH.data, opts.simplePackedRH.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dRH, opts.RHIn, memSize, hipMemcpyHostToDevice));
	}
	
	int src = 0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));
	
	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.isPackedData)
	{
		UnpackAndCalculate <<< gridDim, blockDim >>> (dTPacked, dRHPacked, dT, dRH, dTD, opts, dMissingValuesCount);
	}
	else
	{
		Calculate <<< gridDim, blockDim >>> (dT, dRH, dTD, opts, dMissingValuesCount);
	}
	

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.TDOut, dTD, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));
	CUDA_CHECK(hipFree(dRH));
	CUDA_CHECK(hipFree(dTD));
	CUDA_CHECK(hipFree(dMissingValuesCount));

	if (opts.simplePackedT.HasData())
	{
		CUDA_CHECK(hipFree(dTPacked));
	}

	if (opts.simplePackedRH.HasData())
	{
		CUDA_CHECK(hipFree(dRHPacked));
	}

}
