#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "dewpoint_cuda.h"

#define MAPPED_PINNED

const double RW = 461.5; // Vesihoyryn kaasuvakio (J / K kg)
const double L = 2.5e6; // Veden hoyrystymislampo (J / kg)
const double RW_div_L = RW / L;

namespace himan
{

namespace plugin
{

namespace dewpoint_cuda
{

__global__ void Calculate(const double* __restrict__ dT,
							const double* __restrict__ dRH,
							double* __restrict__ dTD, dewpoint_cuda_options opts, int* dMissingValueCount);

} // namespace dewpoint
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::dewpoint_cuda::Calculate(const double* __restrict__ dT,
															const double* __restrict__ dRH,
															double* __restrict__ dTD, dewpoint_cuda_options opts,
															int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (dT[idx] == kFloatMissing || dRH[idx] == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			dTD[idx] = kFloatMissing;
		}
		else
		{
			dTD[idx] = ((dT[idx]+opts.TBase) / (1 - ((dT[idx]+opts.TBase) * log(dRH[idx]) * (RW_div_L)))) - 273.15 + opts.TBase;
		}
	}
}

void himan::plugin::dewpoint_cuda::DoCuda(dewpoint_cuda_options& opts, dewpoint_cuda_data& datas)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT;
	double* dRH;
	double* dTD;

	unsigned char* dpT;
	unsigned char* dpRH;
	int* dbmT;
	int* dbmRH;
	
	int* dMissingValuesCount;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipHostGetDevicePointer(&dTD, datas.TD, 0));

	if (opts.pT)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dT, datas.T, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpT, datas.pT.data, 0));

		if (datas.pT.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmT, datas.pT.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dT, memsize));
		CUDA_CHECK(hipMemcpy(dT, datas.T, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pRH)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dRH, datas.RH, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpRH, datas.pRH.data, 0));

		if (datas.pRH.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmRH, datas.pRH.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dRH, memsize));
		CUDA_CHECK(hipMemcpy(dRH, datas.RH, memsize, hipMemcpyHostToDevice));
	}

	int src = 0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));
	
	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.pT)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpT, dT, dbmT, datas.pT.coefficients, opts.N, datas.pT.HasBitmap());
	}

	if (opts.pRH)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpRH, dRH, dbmRH, datas.pRH.coefficients, opts.N, datas.pRH.HasBitmap());
	}

	Calculate <<< gridDim, blockDim >>> (dT, dRH, dTD, opts, dMissingValuesCount);

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	if (!opts.pT)
	{
		CUDA_CHECK(hipFree(dT));
	}

	if (!opts.pRH)
	{
		CUDA_CHECK(hipFree(dRH));
	}

	CUDA_CHECK(hipFree(dMissingValuesCount));

}
