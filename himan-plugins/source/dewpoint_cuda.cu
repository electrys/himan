#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "dewpoint_cuda.h"
#include "metutil.h"

__global__ void himan::plugin::dewpoint_cuda::Calculate(const double* __restrict__ d_t,
															const double* __restrict__ d_rh,
															double* __restrict__ d_td,
															options opts,
															int* d_missing)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (d_t[idx] == kFloatMissing || d_rh[idx] == kFloatMissing)
		{
			atomicAdd(d_missing, 1);
			d_td[idx] = kFloatMissing;
		}
		else
		{
			// Branching, but first branch is so much simpler in terms of calculation complexity
			// so it's probably worth it

			double RH = d_rh[idx] * opts.rh_scale;
			
			if (RH > 50)
			{
				d_td[idx] = metutil::DewPointFromHighRH_(d_t[idx]+opts.t_base, RH);
			}
			else
			{
				d_td[idx] = metutil::DewPointFromLowRH_(d_t[idx]+opts.t_base, RH);
			}
		}
	}
}

void himan::plugin::dewpoint_cuda::Process(options& opts)
{
	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* d_t = 0;
	double* d_rh = 0;
	double* d_td = 0;
	
	int* d_missing = 0;

	// Allocate memory on device

	CUDA_CHECK(hipMalloc((void **) &d_t, sizeof(double) * memsize));
	CUDA_CHECK(hipMalloc((void **) &d_rh, sizeof(double) * memsize));

	CUDA_CHECK(hipMalloc((void **) &d_missing, sizeof(int)));
	CUDA_CHECK(hipMalloc((void **) &d_td, memsize));

	if (opts.t->packed_values)
	{
		// Unpack data and copy it back to host, we need it because its put back to cache
		opts.t->packed_values->Unpack(d_t, &stream);
		CUDA_CHECK(hipMemcpyAsync(opts.t->values, d_t, memsize, hipMemcpyDeviceToHost, stream));
	}
	else
	{
		CUDA_CHECK(hipMemcpyAsync(d_t, opts.t->values, memsize, hipMemcpyHostToDevice, stream));
	}

	if (opts.rh->packed_values)
	{
		// Unpack data and copy it back to host, we need it because its put back to cache
		opts.rh->packed_values->Unpack(d_rh, &stream);
		CUDA_CHECK(hipMemcpyAsync(opts.rh->values, d_rh, memsize, hipMemcpyDeviceToHost, stream));
	}
	else
	{
		CUDA_CHECK(hipMemcpyAsync(d_rh, opts.rh->values, memsize, hipMemcpyHostToDevice, stream));
	}

	int src = 0;

	CUDA_CHECK(hipMemcpyAsync(d_missing, &src, sizeof(int), hipMemcpyHostToDevice, stream));
	
	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	CUDA_CHECK(hipStreamSynchronize(stream));
	
	Calculate <<< gridSize, blockSize, 0, stream >>> (d_t, d_rh, d_td, opts, d_missing);

	CUDA_CHECK(hipStreamSynchronize(stream));

	// block until the device has completed

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device

	CUDA_CHECK(hipMemcpyAsync(opts.td->values, d_td, memsize, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&opts.missing, d_missing, sizeof(int), hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	// Free device memory

	CUDA_CHECK(hipFree(d_t));
	CUDA_CHECK(hipFree(d_td));
	CUDA_CHECK(hipFree(d_rh));
	CUDA_CHECK(hipFree(d_missing));

	CUDA_CHECK(hipStreamDestroy(stream));
}
