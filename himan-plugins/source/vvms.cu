#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include "cuda_plugin_helper.h"
#include "vvms.cuh"

__global__ void himan::plugin::vvms_cuda::Calculate(cdarr_t d_t, cdarr_t d_vv, cdarr_t d_p, darr_t d_vv_ms,
                                                    options opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		d_vv_ms[idx] = GetKFloatMissing();
		double P = (opts.is_constant_pressure) ? opts.p_const : d_p[idx];

		if (!IsKFloatMissing(d_t[idx]) && !IsKFloatMissing(d_vv[idx]) && !IsKFloatMissing(P))
		{
			const double w = opts.vv_ms_scale *
			                 (287 * -d_vv[idx] * (opts.t_base + d_t[idx]) / (himan::constants::kG * P * opts.p_scale));

			// Some erroneous values of T, P or VV produce infinite values
			if (isfinite(w))
			{
				d_vv_ms[idx] = w;
			}
			else
			{
				d_vv_ms[idx] = GetKFloatMissing();
			}
		}
	}
}

void himan::plugin::vvms_cuda::Process(options& opts)
{
	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* d_t = 0;
	double* d_p = 0;
	double* d_vv = 0;
	double* d_vv_ms = 0;

	CUDA_CHECK(hipMalloc((void**)&d_vv_ms, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_t, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_vv, memsize));

	PrepareInfo(opts.t, d_t, stream);
	PrepareInfo(opts.vv, d_vv, stream);
	PrepareInfo(opts.vv_ms);

	if (!opts.is_constant_pressure)
	{
		CUDA_CHECK(hipMalloc((void**)&d_p, memsize));

		PrepareInfo(opts.p, d_p, stream);
	}

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N / blockSize + (opts.N % blockSize == 0 ? 0 : 1);

	CUDA_CHECK(hipStreamSynchronize(stream));

	Calculate<<<gridSize, blockSize, 0, stream>>>(d_t, d_vv, d_p, d_vv_ms, opts);

	// block until the device has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device

	himan::ReleaseInfo(opts.vv_ms, d_vv_ms, stream);
	himan::ReleaseInfo(opts.t);
	himan::ReleaseInfo(opts.vv);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_t));
	CUDA_CHECK(hipFree(d_vv));
	CUDA_CHECK(hipFree(d_vv_ms));

	if (d_p)
	{
		himan::ReleaseInfo(opts.p);
		CUDA_CHECK(hipFree(d_p));
	}

	CUDA_CHECK(hipStreamDestroy(stream));
}
