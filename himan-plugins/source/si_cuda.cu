#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>

#include "plugin_factory.h"

#include "si_cuda.h"
#include "cuda_helper.h"
#include "metutil.h"
#include "util.h"

#include <NFmiGribPacking.h>

#include "regular_grid.h"
#include "forecast_time.h"
#include "level.h"

#define HIMAN_AUXILIARY_INCLUDE

#include "fetcher.h"
#include "cache.h"

using namespace himan;
using namespace himan::plugin;

template <typename T>
__global__ void InitializeArrayKernel(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(; idx < N; idx += stride)
	{
		d_arr[idx] = val;
	}
}

template <typename T>
void InitializeArray(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	InitializeArrayKernel<T> <<< gridSize, blockSize, 0, stream >>> (d_arr, val, N);

}

template <typename T>
__global__ void MultiplyWith(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(; idx < N; idx += stride)
	{
		d_arr[idx] = d_arr[idx] * val;
	}
}

template <typename T>
void MultiplyWith(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	MultiplyWith<T> <<< gridSize, blockSize, 0, stream >>> (d_arr, val, N);

}

info_simple* PrepareInfo(std::shared_ptr<himan::info> fullInfo, hipStream_t& stream)
{
	auto h_info = fullInfo->ToSimple();
	size_t N = h_info->size_x * h_info->size_y;
	
	assert(N > 0);

	// 1. Reserve memory at device for unpacked data
	double* d_arr = 0;
	CUDA_CHECK(hipMalloc(reinterpret_cast<double**> (&d_arr), N * sizeof(double)));

	// 2. Unpack if needed, leave data to device and simultaneously copy it back to cpu (cache)
	auto tempGrid = dynamic_cast<himan::regular_grid*> (fullInfo->Grid());

	if (tempGrid->IsPackedData())
	{
		assert(tempGrid->PackedData().ClassName() == "simple_packed" || tempGrid->PackedData().ClassName() == "jpeg_packed");
		assert(N > 0);
		assert(tempGrid->Data().Size() == N);

		double* arr = const_cast<double*> (tempGrid->Data().ValuesAsPOD());
		CUDA_CHECK(hipHostRegister(reinterpret_cast<void*> (arr), sizeof(double) * N, 0));

		assert(arr);

		tempGrid->PackedData().Unpack(d_arr, N, &stream);

		CUDA_CHECK(hipMemcpyAsync(arr, d_arr, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

		auto c = GET_PLUGIN(cache);

		CUDA_CHECK(hipStreamSynchronize(stream));

		tempGrid->PackedData().Clear();
		c->Insert(*fullInfo);	

		CUDA_CHECK(hipHostUnregister(arr));

		h_info->packed_values = 0;

	}
	else
	{
		CUDA_CHECK(hipMemcpyAsync(d_arr, fullInfo->Data().ValuesAsPOD(), sizeof(double) * N, hipMemcpyHostToDevice, stream));
	}

	h_info->values = d_arr;
	
	return h_info;
}

void PrepareInfo(std::shared_ptr<himan::info> fullInfo, info_simple** h_info, hipStream_t& stream)
{
	size_t N = (**h_info).size_x * (**h_info).size_y;
	assert(N > 0);

	// 1. Reserve memory at device for unpacked data
	double* d_arr = 0;
	CUDA_CHECK(hipMalloc(reinterpret_cast<double**> (&d_arr), N * sizeof(double)));
	//CUDA_CHECK(hipMalloc((double**) (&(*h_info)->values), N * sizeof(double)));
	(*h_info)->values = d_arr;

	// 2. Unpack if needed, leave data to device and simultaneously copy it back to cpu (cache)
	auto tempGrid = dynamic_cast<himan::regular_grid*> (fullInfo->Grid());

	if (tempGrid->IsPackedData())
	{
		assert(tempGrid->PackedData().ClassName() == "simple_packed" || tempGrid->PackedData().ClassName() == "jpeg_packed");
		assert(N > 0);
		assert(tempGrid->Data().Size() == N);

		double* arr = const_cast<double*> (tempGrid->Data().ValuesAsPOD());
		CUDA_CHECK(hipHostRegister(reinterpret_cast<void*> (arr), sizeof(double) * N, 0));

		assert(arr);

		tempGrid->PackedData().Unpack((*h_info)->values, N, &stream);

		CUDA_CHECK(hipMemcpyAsync(arr, (*h_info)->values, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

		auto c = GET_PLUGIN(cache);

		CUDA_CHECK(hipStreamSynchronize(stream));

		tempGrid->PackedData().Clear();
		c->Insert(*fullInfo);	

		CUDA_CHECK(hipHostUnregister(arr));

		(**h_info).packed_values = 0;

	}
	else
	{
		CUDA_CHECK(hipMemcpyAsync((*h_info)->values, fullInfo->Data().ValuesAsPOD(), sizeof(double) * N, hipMemcpyHostToDevice, stream));
	}

}

std::shared_ptr<himan::info> Fetch(const std::shared_ptr<const plugin_configuration> conf, const himan::forecast_time& theTime, const himan::level& theLevel, const himan::param& theParam, const himan::forecast_type& theType)
{
	try
	{
		auto f = GET_PLUGIN(fetcher);
		return f->Fetch(conf, theTime, theLevel, theParam, theType, true);
	}
	catch (HPExceptionType& e)
	{
		if (e != kFileDataNotFound)
		{
			throw std::runtime_error("si_cuda::Fetch(): Unable to proceed");
		}
		
		return std::shared_ptr<info> ();
	}
}

__global__
void CopyLFCIteratorValuesKernel(double* __restrict__ d_Titer, const double* __restrict__ d_Tparcel, double* __restrict__ d_Piter, info_simple d_Penv)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Penv.size_x * d_Penv.size_y)
	{
		if (d_Tparcel[idx] != kFloatMissing && d_Penv.values[idx] != kFloatMissing)
		{
			d_Titer[idx] = d_Tparcel[idx];
			d_Piter[idx] = d_Penv.values[idx];
		}
	}
}
__global__
void MoistLiftKernel(const double* __restrict__ d_T, const double* __restrict__  d_P, info_simple d_Ptarget, double* __restrict__ d_Tparcel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	assert(d_T);
	assert(d_P);

	if (idx < d_Ptarget.size_x * d_Ptarget.size_y)
	{
		assert(d_P[idx] > 10);
		assert(d_P[idx] < 1500);

		assert(d_Ptarget.values[idx] > 10);
		assert(d_Ptarget.values[idx] < 1500);

		assert(d_T[idx] > 100);
		assert(d_T[idx] < 350 || d_T[idx] == kFloatMissing);

		double T = metutil::MoistLift_(d_P[idx]*100, d_T[idx], d_Ptarget.values[idx]*100);

		assert(T > 100);
		assert(T < 350 || T == kFloatMissing);

		d_Tparcel[idx] = T;
	}
}

__global__
void LFCKernel(info_simple d_T, info_simple d_P, info_simple d_prevT, info_simple d_prevP, double* __restrict__ d_Tparcel, double* __restrict__ d_LCLP, double* __restrict__ d_LFCT, double* __restrict__ d_LFCP, unsigned char* __restrict__ d_found, int d_curLevel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	assert(d_T.values);
	assert(d_P.values);

	if (idx < d_T.size_x * d_T.size_y)
	{
		double Tparcel = d_Tparcel[idx];
		double Tenv = d_T.values[idx];
		
		assert(Tenv < 350.);
		assert(Tenv > 100.);
		
		double prevTenv = d_prevT.values[idx];
		assert(prevTenv < 350.);
		assert(prevTenv > 100.);

		double Penv = d_P.values[idx];
		double LCLP = d_LCLP[idx];
		
		if (Tparcel != kFloatMissing && d_curLevel < 95 && (Tenv - Tparcel) > 30.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}
		
		if (Tparcel != kFloatMissing && Penv <= LCLP)
		{
			if (Tparcel >= Tenv && d_found[idx] == 0)
			{
				d_found[idx] = 1;

				// We have no specific information on the precise height where the temperature has crossed
				// Or we could if we'd integrate it but it makes the calculation more complex. So maybe in the
				// future. For now just take an average of upper and lower level values.
				
				if (prevTenv == kFloatMissing) prevTenv = Tenv;

				d_LFCT[idx] = (Tenv + prevTenv) * 0.5; // K

				assert(d_LFCT[idx] > 100);
				assert(d_LFCT[idx] < 350);

				// Never allow LFC pressure to be bigger than LCL pressure; bound lower level (with larger pressure value)
				// to LCL level if it below LCL

				double prevPenv = d_prevP.values[idx];
				prevPenv = min(prevPenv, LCLP);
				assert(prevPenv > 10);
				assert(prevPenv < 1500);

				d_LFCP[idx] = (Penv + prevPenv) * 0.5; // hPa
			}
		}
	}
}


__global__
void ThetaEKernel(info_simple d_T, info_simple d_RH, info_simple d_P, double* __restrict__ d_maxThetaE, double* __restrict__ d_Tresult, double* __restrict__ d_TDresult, unsigned char* __restrict__ d_found)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	assert(d_T.values);
	assert(d_RH.values);
	assert(d_P.values);
	
	if (idx < d_T.size_x * d_T.size_y)
	{
		double T = d_T.values[idx];
		double P = d_P.values[idx];
		double RH = d_RH.values[idx];
		double TD = metutil::DewPointFromRH_(T, RH);
		
		double& refThetaE = d_maxThetaE[idx];
		double ThetaE = metutil::ThetaE_(T, TD, P*100);

		if (P == kFloatMissing || P < 600.)
		{
			d_found[idx] = 1;
		}
		else
		{
			if (ThetaE >= refThetaE)
			{
				refThetaE = ThetaE;

				d_Tresult[idx] = T;
				d_TDresult[idx] = TD;
			}
		}
	}
}

std::pair<std::vector<double>,std::vector<double>> si_cuda::GetHighestThetaETAndTDGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo)
{
	himan::level curLevel(kHybrid, 137);
	
	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));
	
	double* d_maxThetaE = 0;
	double* d_Tresult = 0;
	double* d_TDresult = 0;
	unsigned char* d_found = 0;
	
	CUDA_CHECK(hipMalloc((double**) &d_maxThetaE, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Tresult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_TDresult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_found, sizeof(unsigned char) * N));

	InitializeArray<double> (d_maxThetaE, -1, N, stream);
	InitializeArray<double> (d_Tresult, kFloatMissing, N, stream);
	InitializeArray<double> (d_TDresult, kFloatMissing, N, stream);
	InitializeArray<unsigned char> (d_found, 0, N, stream);
	
	while (curLevel.Value() > 90)
	{
		auto TInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto RHInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("RH-PRCNT"), myTargetInfo->ForecastType());
		auto PInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

		assert(TInfo && RHInfo && PInfo);
		assert(TInfo->Data().MissingCount() == 0);

		auto h_T = TInfo->ToSimple();
		PrepareInfo(TInfo, &h_T, stream);
		
		auto h_P = PInfo->ToSimple();
		PrepareInfo(PInfo, &h_P, stream);
		
		auto h_RH = RHInfo->ToSimple();
		PrepareInfo(RHInfo, &h_RH, stream);

		assert(h_T->values);
		assert(h_RH->values);
		assert(h_P->values);

		ThetaEKernel <<< gridSize, blockSize, 0, stream >>> (*h_T, *h_RH, *h_P, d_maxThetaE, d_Tresult, d_TDresult, d_found);

		std::vector<unsigned char> found(N, 0);
		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipStreamSynchronize(stream));

		CUDA_CHECK(hipFree(h_P->values));
		CUDA_CHECK(hipFree(h_RH->values));
		CUDA_CHECK(hipFree(h_T->values));
		
		delete h_P;
		delete h_T;
		delete h_RH;

		curLevel.Value(curLevel.Value()-1);

		size_t foundCount = std::count(found.begin(), found.end(), 1);

		if (foundCount == found.size()) break;
	}
	
	std::vector<double> Tsurf(myTargetInfo->Data().Size());
	std::vector<double> TDsurf(myTargetInfo->Data().Size());

	CUDA_CHECK(hipMemcpyAsync(&Tsurf[0], d_Tresult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&TDsurf[0], d_TDresult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));
	
	CUDA_CHECK(hipFree(d_maxThetaE));
	CUDA_CHECK(hipFree(d_Tresult));
	CUDA_CHECK(hipFree(d_TDresult));
	CUDA_CHECK(hipFree(d_found));
	
	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_pair(Tsurf, TDsurf);
	
}

std::pair<std::vector<double>,std::vector<double>> si_cuda::GetLFCGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo, std::vector<double>& T, std::vector<double>& P, std::vector<double>& TenvLCL)
{
	//auto h = GET_PLUGIN(hitool);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_TenvLCL = 0;
	double* d_Titer = 0;
	double* d_Piter = 0;
	double* d_LCLP = 0;
	double* d_LFCT = 0;
	double* d_LFCP = 0;
	double* d_Tparcel = 0;

	unsigned char* d_found = 0;
	
	CUDA_CHECK(hipMalloc((double**) &d_TenvLCL, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Piter, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Titer, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LCLP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LFCT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LFCP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Tparcel, sizeof(double) * N));

	CUDA_CHECK(hipMemcpyAsync(d_TenvLCL, &TenvLCL[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Titer, &T[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	
	CUDA_CHECK(hipMemcpyAsync(d_LCLP, d_Piter, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
	
	InitializeArray<double> (d_LFCT, kFloatMissing, N, stream);
	InitializeArray<double> (d_LFCP, kFloatMissing, N, stream);
	InitializeArray<unsigned char> (d_found, 0, N, stream);

	// For each grid point find the hybrid level that's below LCL and then pick the lowest level
	// among all grid points; most commonly it's the lowest hybrid level

	//auto levels = h->LevelForHeight(myTargetInfo->Producer(), CAPE::Max(P));

	//level curLevel = levels.first;
	level curLevel = level(kHybrid,137);
	auto prevPenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());
	auto prevTenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());

	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);

	assert(h_prevTenv->values);
	assert(h_prevPenv->values);

	curLevel.Value(curLevel.Value()-1);

	std::vector<unsigned char> found(N, 0);
	std::vector<double> LFCT(N, kFloatMissing);
	std::vector<double> LFCP(N, kFloatMissing);

	for (size_t i = 0; i < N; i++)
	{
		if (T[i] >= TenvLCL[i])
		{
			found[i] = true;
			LFCT[i] = T[i];
			LFCP[i] = P[i];
			//Piter[i] = kFloatMissing;
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCT, &LFCT[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCP, &LFCP[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	while (curLevel.Value() > 70)
	{	
		// Get environment temperature and pressure values for this level
		auto TenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto PenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);

		// Lift the particle from previous level to this level. In the first revolution
		// of this loop the starting level is LCL. If target level level is below current level
		// (ie. we would be lowering the particle) missing value is returned.

		MoistLiftKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Piter, *h_Penv, d_Tparcel);

		LFCKernel <<< gridSize, blockSize, 0, stream >>> (*h_Tenv, *h_Penv, *h_prevTenv, *h_prevPenv, d_Tparcel, d_LCLP, d_LFCT, d_LFCP, d_found, curLevel.Value());

		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipFree(h_prevPenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));

		delete h_prevPenv;
		delete h_prevTenv;

		h_prevPenv = h_Penv;
		h_prevTenv = h_Tenv;

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (static_cast<size_t> (std::count(found.begin(), found.end(), 1)) == found.size()) break;

		// preserve starting position for those grid points that have value

		CopyLFCIteratorValuesKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Tparcel, d_Piter, *h_Penv);
		
		curLevel.Value(curLevel.Value() - 1);	

	}

	CUDA_CHECK(hipFree(h_prevPenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));

	delete h_prevPenv;
	delete h_prevTenv;

	CUDA_CHECK(hipMemcpyAsync(&LFCT[0], d_LFCT, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&LFCP[0], d_LFCP, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_LCLP));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_Titer));
	CUDA_CHECK(hipFree(d_Piter));
	CUDA_CHECK(hipFree(d_TenvLCL));

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_pair(LFCT, LFCP);
}
