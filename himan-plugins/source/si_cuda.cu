#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>

#include "plugin_factory.h"

#include "si_cuda.h"
#include "cuda_helper.h"
#include "metutil.h"
#include "util.h"

#include <NFmiGribPacking.h>

#include "regular_grid.h"
#include "forecast_time.h"
#include "level.h"

#define HIMAN_AUXILIARY_INCLUDE

#include "fetcher.h"
#include "cache.h"
#include "hitool.h"


using namespace himan;
using namespace himan::plugin;

level si_cuda::itsBottomLevel;

const unsigned char FCAPE		= (1 << 2);
const unsigned char FCAPE3km	= (1 << 0);

extern double Max(const std::vector<double>& vec);

template <typename T>
__global__ void InitializeArrayKernel(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(; idx < N; idx += stride)
	{
		d_arr[idx] = val;
	}
}

template <typename T>
void InitializeArray(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	InitializeArrayKernel<T> <<< gridSize, blockSize, 0, stream >>> (d_arr, val, N);

}

template <typename T>
__global__ void MultiplyWith(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for(; idx < N; idx += stride)
	{
		d_arr[idx] = d_arr[idx] * val;
	}
}

template <typename T>
void MultiplyWith(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	MultiplyWith<T> <<< gridSize, blockSize, 0, stream >>> (d_arr, val, N);

}

info_simple* PrepareInfo(std::shared_ptr<himan::info> fullInfo, hipStream_t& stream)
{
	auto h_info = fullInfo->ToSimple();
	size_t N = h_info->size_x * h_info->size_y;
	
	assert(N > 0);

	// 1. Reserve memory at device for unpacked data
	double* d_arr = 0;
	CUDA_CHECK(hipMalloc(reinterpret_cast<double**> (&d_arr), N * sizeof(double)));

	// 2. Unpack if needed, leave data to device and simultaneously copy it back to cpu (himan cache)
	auto tempGrid = dynamic_cast<himan::regular_grid*> (fullInfo->Grid());

	if (tempGrid->IsPackedData())
	{
		assert(tempGrid->PackedData().ClassName() == "simple_packed" || tempGrid->PackedData().ClassName() == "jpeg_packed");
		assert(N > 0);
		assert(tempGrid->Data().Size() == N);

		double* arr = const_cast<double*> (tempGrid->Data().ValuesAsPOD());
		CUDA_CHECK(hipHostRegister(reinterpret_cast<void*> (arr), sizeof(double) * N, 0));

		assert(arr);

		tempGrid->PackedData().Unpack(d_arr, N, &stream);

		CUDA_CHECK(hipMemcpyAsync(arr, d_arr, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

		tempGrid->PackedData().Clear();

		auto c = GET_PLUGIN(cache);

		CUDA_CHECK(hipStreamSynchronize(stream));

		c->Insert(*fullInfo);	

		CUDA_CHECK(hipHostUnregister(arr));

		h_info->packed_values = 0;
	}
	else
	{
		CUDA_CHECK(hipMemcpyAsync(d_arr, fullInfo->Data().ValuesAsPOD(), sizeof(double) * N, hipMemcpyHostToDevice, stream));
	}

	h_info->values = d_arr;
	
	return h_info;
}

std::shared_ptr<himan::info> Fetch(const std::shared_ptr<const plugin_configuration> conf, const himan::forecast_time& theTime, const himan::level& theLevel, const himan::param& theParam, const himan::forecast_type& theType)
{
	try
	{
		auto f = GET_PLUGIN(fetcher);
		return f->Fetch(conf, theTime, theLevel, theParam, theType, true);
	}
	catch (HPExceptionType& e)
	{
		if (e != kFileDataNotFound)
		{
			throw std::runtime_error("si_cuda::Fetch(): Unable to proceed");
		}
		
		return std::shared_ptr<info> ();
	}
}

__global__
void CopyLFCIteratorValuesKernel(double* __restrict__ d_Titer, const double* __restrict__ d_Tparcel, double* __restrict__ d_Piter, info_simple d_Penv)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Penv.size_x * d_Penv.size_y)
	{
		if (d_Tparcel[idx] != kFloatMissing && d_Penv.values[idx] != kFloatMissing)
		{
			d_Titer[idx] = d_Tparcel[idx];
			d_Piter[idx] = d_Penv.values[idx];
		}
	}
}

__global__
void LiftLCLKernel(const double* __restrict__ d_P, const double* __restrict__ d_T, const double* __restrict__ d_PLCL, info_simple d_Ptarget, double* __restrict__ d_Tparcel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Ptarget.size_x * d_Ptarget.size_y)
	{
		assert(d_P[idx] > 10);
		assert(d_P[idx] < 1500 || d_P[idx] == kFloatMissing);

		assert(d_Ptarget.values[idx] > 10);
		assert(d_Ptarget.values[idx] < 1500 || d_Ptarget.values[idx] == kFloatMissing);

		assert(d_T[idx] > 100);
		assert(d_T[idx] < 350 || d_T[idx] == kFloatMissing);

		double T = metutil::LiftLCL_(d_P[idx]*100, d_T[idx], d_PLCL[idx]*100, d_Ptarget.values[idx]*100);

		assert(T > 100);
		assert(T < 350 || T == kFloatMissing);

		d_Tparcel[idx] = T;
	}
}

__global__
void MoistLiftKernel(const double* __restrict__ d_T, const double* __restrict__  d_P, info_simple d_Ptarget, double* __restrict__ d_Tparcel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	assert(d_T);
	assert(d_P);

	if (idx < d_Ptarget.size_x * d_Ptarget.size_y)
	{
		assert(d_P[idx] > 10);
		assert(d_P[idx] < 1500 || d_P[idx] == kFloatMissing);

		assert(d_Ptarget.values[idx] > 10);
		assert(d_Ptarget.values[idx] < 1500 || d_Ptarget.values[idx] == kFloatMissing);

		assert(d_T[idx] > 100);
		assert(d_T[idx] < 350 || d_T[idx] == kFloatMissing);

		double T = metutil::MoistLift_(d_P[idx]*100, d_T[idx], d_Ptarget.values[idx]*100);

		assert(T > 100);
		assert(T < 350 || T == kFloatMissing);

		d_Tparcel[idx] = T;
	}
}

__global__
void CAPEKernel(info_simple d_Tenv, info_simple d_Penv, info_simple d_Zenv, info_simple d_prevTenv, info_simple d_prevPenv, info_simple d_prevZenv, 
		const double* __restrict d_Tparcel, const double* __restrict d_prevTparcel, const double* __restrict__ d_LFCP, double* __restrict__ d_CAPE, 
		double* __restrict__ d_CAPE1040, double* __restrict__ d_CAPE3km, double* __restrict__ d_ELT, double* __restrict__ d_ELP, unsigned char* __restrict__ d_found, int d_curLevel, int d_breakLevel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Tenv.size_x * d_Tenv.size_y && d_found[idx] != 4)
	{
		double Tenv = d_Tenv.values[idx];
		assert(Tenv > 100.);

		double Penv = d_Penv.values[idx]; // hPa
		assert(Penv < 1200.);
		
		double Zenv = d_Zenv.values[idx]; // m
		
		double prevTenv = d_prevTenv.values[idx]; // K
		assert(prevTenv > 100.);

		double prevPenv = d_prevPenv.values[idx]; // hPa
		assert(prevPenv < 1200.);
		
		double prevZenv = d_prevZenv.values[idx]; // m

		double Tparcel = d_Tparcel[idx]; // K
		assert(Tparcel > 100. || Tparcel == kFloatMissing);

		double prevTparcel = d_prevTparcel[idx]; // K
		assert(prevTparcel > 100. || Tparcel == kFloatMissing);

		double LFCP = d_LFCP[idx]; // hPa
		assert(LFCP < 1200.);
				
		if (Penv == kFloatMissing || Tenv == kFloatMissing || Zenv == kFloatMissing || prevZenv == kFloatMissing || Tparcel == kFloatMissing || prevTparcel == kFloatMissing || Penv > LFCP)
		{
			// Missing data or current grid point is below LFC
			;
		}
		else if (d_curLevel < d_breakLevel && (Tenv - Tparcel) > 25.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] |= FCAPE;
		}
		else
		{
			if (prevZenv >= 3000. && Zenv >= 3000.)
			{
				d_found[idx] |= FCAPE3km;
			}

			if ((d_found[idx] & FCAPE3km) == 0)
			{
				double C = CAPE::CalcCAPE3km(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

				d_CAPE3km[idx] += C;

				assert(d_CAPE3km[idx] < 3000.); // 3000J/kg, not 3000m
				assert(d_CAPE3km[idx] >= 0);
			}

			double C = CAPE::CalcCAPE1040(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

			d_CAPE1040[idx] += C;

			assert(d_CAPE1040[idx] < 5000.);
			assert(d_CAPE1040[idx] >= 0);

			double CAPE, ELT, ELP;
			CAPE::CalcCAPE(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv, CAPE, ELT, ELP);

			d_CAPE[idx] += CAPE;

			assert(CAPE >= 0.);				
			assert(d_CAPE[idx] < 8000);

			if (ELT != kFloatMissing)
			{
				d_ELT[idx] = ELT;
				d_ELP[idx] = ELP;
			}
		}
	}
}

__global__
void CINKernel(info_simple d_Tenv, info_simple d_Penv, const double* __restrict__ d_Titer, const double* __restrict__ d_Piter, info_simple d_Zenv, info_simple d_prevZenv, const double* __restrict__ d_Tparcel, const double* __restrict__ d_PLCL, const double* __restrict__ d_PLFC, double* __restrict__ d_cinh, unsigned char* __restrict__ d_found)
{
	
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Tenv.size_x * d_Tenv.size_y && d_found[idx] == 0)
	{
		
		double Tenv = d_Tenv.values[idx]; // K
		assert(Tenv >= 150.);

		double Penv = d_Penv.values[idx]; // hPa
		assert(Penv < 1200. || Penv == kFloatMissing);

		double Pparcel = d_Piter[idx]; // hPa
		assert(Pparcel < 1200. || Pparcel == kFloatMissing);

		double Tparcel = d_Titer[idx]; // K
		assert(Tparcel >= 150. || Tparcel == kFloatMissing);

		double PLFC = d_PLFC[idx]; // hPa
		assert(PLFC < 1200. || PLFC == kFloatMissing);
		
		double PLCL = d_PLCL[idx]; // hPa
		assert(PLCL < 1200. || PLCL == kFloatMissing);
		
		double Zenv = d_Zenv.values[idx]; // m
		double prevZenv = d_prevZenv.values[idx]; // m

		if (
				PLFC == kFloatMissing || // No LFC
				Penv <= PLFC // reached max height; TODO: final piece integration
				)
		{
			d_found[idx] = 1;
		}
		else
		{
			if (Penv < PLCL)
			{
				// Above LCL, switch to virtual temperature
				
				Tparcel = metutil::VirtualTemperature_(Tparcel, Penv * 100);
				Tenv = metutil::VirtualTemperature_(Tenv, Penv * 100);
			}

			if (Tparcel != kFloatMissing && Tparcel <= Tenv)
			{
				d_cinh[idx] += constants::kG * (Zenv - prevZenv) * ((Tparcel - Tenv) / Tenv);
				assert(d_cinh[idx] <= 0);
			}
			else if (d_cinh[idx] != 0)
			{
				// Parcel buoyant --> cape layer, no more CIN. We stop integration here.
				// TODO: final piece integration
				d_found[idx] = 1;
			}
		}	
	}
}

__global__
void LFCKernel(info_simple d_T, info_simple d_P, info_simple d_prevT, info_simple d_prevP, double* __restrict__ d_Tparcel, double* __restrict__ d_LCLP, double* __restrict__ d_LFCT, double* __restrict__ d_LFCP, unsigned char* __restrict__ d_found, int d_curLevel, int d_breakLevel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	assert(d_T.values);
	assert(d_P.values);

	if (idx < d_T.size_x * d_T.size_y && d_found[idx] == 0)
	{
		double Tparcel = d_Tparcel[idx];
		double Tenv = d_T.values[idx];
		
		assert(Tenv < 350.);
		assert(Tenv > 100.);
		
		double prevTenv = d_prevT.values[idx];
		assert(prevTenv < 350.);
		assert(prevTenv > 100.);

		double Penv = d_P.values[idx];
		double LCLP = d_LCLP[idx];
		
		if (Tparcel != kFloatMissing && d_curLevel < d_breakLevel && (Tenv - Tparcel) > 30.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}
		
		if (Tparcel != kFloatMissing && Penv <= LCLP)
		{
			if (Tparcel >= Tenv && d_found[idx] == 0)
			{
				d_found[idx] = 1;

				// We have no specific information on the precise height where the temperature has crossed
				// Or we could if we'd integrate it but it makes the calculation more complex. So maybe in the
				// future. For now just take an average of upper and lower level values.
				
				if (prevTenv == kFloatMissing) prevTenv = Tenv;

				d_LFCT[idx] = (Tenv + prevTenv) * 0.5; // K

				assert(d_LFCT[idx] > 100);
				assert(d_LFCT[idx] < 350);

				// Never allow LFC pressure to be bigger than LCL pressure; bound lower level (with larger pressure value)
				// to LCL level if it below LCL

				double prevPenv = d_prevP.values[idx];
				prevPenv = min(prevPenv, LCLP);
				assert(prevPenv > 10);
				assert(prevPenv < 1500);

				d_LFCP[idx] = (Penv + prevPenv) * 0.5; // hPa
			}
		}
	}
}


__global__
void ThetaEKernel(info_simple d_T, info_simple d_RH, info_simple d_P, info_simple d_prevT, info_simple d_prevRH, info_simple d_prevP, double* __restrict__ d_maxThetaE, double* __restrict__ d_Tresult, double* __restrict__ d_TDresult, unsigned char* __restrict__ d_found)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	assert(d_T.values);
	assert(d_RH.values);
	assert(d_P.values);
	
	if (idx < d_T.size_x * d_T.size_y && d_found[idx] == 0)
	{
		double T = d_T.values[idx];
		double P = d_P.values[idx];
		double RH = d_RH.values[idx];
		
		if (P == kFloatMissing || T == kFloatMissing || RH == kFloatMissing)
		{
			d_found[idx] = 1;
		}
		else
		{
			if (P < 600.)
			{
				// Cut search if reach level 600hPa

				// Linearly interpolate temperature and humidity values to 600hPa, to check
				// if highest theta e is found there

				T = numerical_functions::interpolation::Linear(600., P, d_prevP.values[idx], T, d_prevT.values[idx]);
				RH = numerical_functions::interpolation::Linear(600., P, d_prevP.values[idx], RH, d_prevRH.values[idx]);

				d_found[idx] = 1; // Make sure this is the last time we access this grid point
				P = 600.;
			}

			double TD = metutil::DewPointFromRH_(T, RH);

			double& refThetaE = d_maxThetaE[idx];
			double ThetaE = metutil::ThetaE_(T, TD, P*100);

			if (ThetaE >= refThetaE)
			{
				refThetaE = ThetaE;
				d_Tresult[idx] = T;
				d_TDresult[idx] = TD;
			}
		}
	}
}

std::pair<std::vector<double>,std::vector<double>> si_cuda::GetHighestThetaETAndTDGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo)
{
	himan::level curLevel = itsBottomLevel;
	
	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));
	
	double* d_maxThetaE = 0;
	double* d_Tresult = 0;
	double* d_TDresult = 0;
	unsigned char* d_found = 0;
	
	CUDA_CHECK(hipMalloc((double**) &d_maxThetaE, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Tresult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_TDresult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_found, sizeof(unsigned char) * N));

	InitializeArray<double> (d_maxThetaE, -1, N, stream);
	InitializeArray<double> (d_Tresult, kFloatMissing, N, stream);
	InitializeArray<double> (d_TDresult, kFloatMissing, N, stream);
	InitializeArray<unsigned char> (d_found, 0, N, stream);
	
	info_simple* h_prevT = 0;
	info_simple* h_prevP = 0;
	info_simple* h_prevRH = 0;
	
	while (true)
	{

		auto TInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto RHInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("RH-PRCNT"), myTargetInfo->ForecastType());
		auto PInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

		if (!TInfo || !RHInfo || !PInfo)
		{
			return std::make_pair(std::vector<double>(),std::vector<double>());
		}

		auto h_T = PrepareInfo(TInfo, stream);
		auto h_P = PrepareInfo(PInfo, stream);
		auto h_RH = PrepareInfo(RHInfo, stream);

		assert(h_T->values);
		assert(h_RH->values);
		assert(h_P->values);

		bool release = true;

		if (!h_prevT)
		{
			// first time
			h_prevT = new info_simple(*h_T);
			h_prevP = new info_simple(*h_P);
			h_prevRH = new info_simple(*h_RH);

			release = false;
		}

		ThetaEKernel <<< gridSize, blockSize, 0, stream >>> (*h_T, *h_RH, *h_P, *h_prevT, *h_prevRH, *h_prevP, d_maxThetaE, d_Tresult, d_TDresult, d_found);

		std::vector<unsigned char> found(N, 0);
		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipStreamSynchronize(stream));

		if (release)
		{
			CUDA_CHECK(hipFree(h_prevP->values));
			CUDA_CHECK(hipFree(h_prevRH->values));
			CUDA_CHECK(hipFree(h_prevT->values));
		}
				
		delete h_prevP;
		delete h_prevT;
		delete h_prevRH;

		h_prevP = h_P;
		h_prevRH = h_RH;
		h_prevT = h_T;
		
		curLevel.Value(curLevel.Value()-1);

		size_t foundCount = std::count(found.begin(), found.end(), 1);

		if (foundCount == found.size()) break;
	}
	
	CUDA_CHECK(hipFree(h_prevP->values));
	CUDA_CHECK(hipFree(h_prevRH->values));
	CUDA_CHECK(hipFree(h_prevT->values));
	
	delete h_prevP;
	delete h_prevT;
	delete h_prevRH;

	std::vector<double> Tsurf(myTargetInfo->Data().Size());
	std::vector<double> TDsurf(myTargetInfo->Data().Size());

	CUDA_CHECK(hipMemcpyAsync(&Tsurf[0], d_Tresult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&TDsurf[0], d_TDresult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));
	
	CUDA_CHECK(hipFree(d_maxThetaE));
	CUDA_CHECK(hipFree(d_Tresult));
	CUDA_CHECK(hipFree(d_TDresult));
	CUDA_CHECK(hipFree(d_found));
	
	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_pair(Tsurf, TDsurf);
	
}

std::pair<std::vector<double>,std::vector<double>> si_cuda::GetLFCGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo, std::vector<double>& T, std::vector<double>& P, std::vector<double>& TenvLCL)
{
	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->HeightUnit(kHPa);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_TenvLCL = 0;
	double* d_Titer = 0;
	double* d_Piter = 0;
	double* d_LCLP = 0;
	double* d_LFCT = 0;
	double* d_LFCP = 0;
	double* d_Tparcel = 0;

	unsigned char* d_found = 0;
	
	CUDA_CHECK(hipMalloc((double**) &d_TenvLCL, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Piter, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Titer, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LCLP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LFCT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LFCP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Tparcel, sizeof(double) * N));

	CUDA_CHECK(hipMemcpyAsync(d_TenvLCL, &TenvLCL[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Titer, &T[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	
	CUDA_CHECK(hipMemcpyAsync(d_LCLP, d_Piter, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
	
	InitializeArray<double> (d_LFCT, kFloatMissing, N, stream);
	InitializeArray<double> (d_LFCP, kFloatMissing, N, stream);
	InitializeArray<unsigned char> (d_found, 0, N, stream);

	// For each grid point find the hybrid level that's below LCL and then pick the lowest level
	// among all grid points; most commonly it's the lowest hybrid level

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P));

	level curLevel = levels.first;
	
	auto prevPenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());
	auto prevTenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());

	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);

	assert(h_prevTenv->values);
	assert(h_prevPenv->values);

	curLevel.Value(curLevel.Value()-1);

	std::vector<unsigned char> found(N, 0);
	std::vector<double> LFCT(N, kFloatMissing);
	std::vector<double> LFCP(N, kFloatMissing);

	for (size_t i = 0; i < N; i++)
	{
		if (T[i] >= TenvLCL[i])
		{
			found[i] = 1;
			LFCT[i] = T[i];
			LFCP[i] = P[i];
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCT, &LFCT[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCP, &LFCP[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	auto hPa450 = h->LevelForHeight(myTargetInfo->Producer(), 450.);
	auto hPa150 = h->LevelForHeight(myTargetInfo->Producer(), 150.);

	while (curLevel.Value() > hPa150.first.Value())
	{	
		// Get environment temperature and pressure values for this level
		auto TenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto PenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);

		// Lift the particle from previous level to this level. In the first revolution
		// of this loop the starting level is LCL. If target level level is below current level
		// (ie. we would be lowering the particle) missing value is returned.

		MoistLiftKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Piter, *h_Penv, d_Tparcel);

		LFCKernel <<< gridSize, blockSize, 0, stream >>> (*h_Tenv, *h_Penv, *h_prevTenv, *h_prevPenv, d_Tparcel, d_LCLP, d_LFCT, d_LFCP, d_found, curLevel.Value(), hPa450.first.Value());

		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipFree(h_prevPenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));

		delete h_prevPenv;
		delete h_prevTenv;

		h_prevPenv = h_Penv;
		h_prevTenv = h_Tenv;

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (static_cast<size_t> (std::count(found.begin(), found.end(), 1)) == found.size()) break;

		// preserve starting position for those grid points that have value

		CopyLFCIteratorValuesKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Tparcel, d_Piter, *h_Penv);
		
		curLevel.Value(curLevel.Value() - 1);	

	}

	CUDA_CHECK(hipFree(h_prevPenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));

	delete h_prevPenv;
	delete h_prevTenv;

	CUDA_CHECK(hipMemcpyAsync(&LFCT[0], d_LFCT, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&LFCP[0], d_LFCP, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_LCLP));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_Titer));
	CUDA_CHECK(hipFree(d_Piter));
	CUDA_CHECK(hipFree(d_TenvLCL));

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_pair(LFCT, LFCP);
}


void si_cuda::GetCINGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo, const std::vector<double>& Tsurf, const std::vector<double>& TLCL, const std::vector<double>& PLCL, const std::vector<double>& PLFC, param CINParam)
{
	const params PParams({param("PGR-PA"), param("P-PA")});

	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->HeightUnit(kHPa);

	forecast_time ftime = myTargetInfo->Time();
	forecast_type ftype = myTargetInfo->ForecastType();

	/*
	 * Modus operandi:
	 * 
	 * 1. Integrate from ground to LCL dry adiabatically
	 * 
	 * This can be done always since LCL is known at all grid points 
	 * (that have source data values defined).
	 * 
	 * 2. Integrate from LCL to LFC moist adiabatically
	 * 
	 * Note! For some points integration will fail (no LFC found)
	 * 
	 * We stop integrating at first time CAPE area is found!
	 */
	
	// Get LCL and LFC heights in meters

	auto ZLCL = h->VerticalValue(param("HL-M"), PLCL);
	auto ZLFC = h->VerticalValue(param("HL-M"), PLFC);

	level curLevel = itsBottomLevel;
	
	auto basePenvInfo = Fetch(conf, ftime, curLevel, param("P-HPA"), ftype);
	auto prevZenvInfo = Fetch(conf, ftime, curLevel, param("HL-M"), ftype);
	auto prevTenvInfo = Fetch(conf, ftime, curLevel, param("T-K"), ftype);
	auto prevPenvInfo = Fetch(conf, ftime, curLevel, param("P-HPA"), ftype);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	auto h_basePenv = PrepareInfo(basePenvInfo, stream);
	auto h_prevZenv = PrepareInfo(prevZenvInfo, stream);
	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);
	
	double* d_Tparcel = 0;
	double* d_Piter = 0;
	double* d_Titer = 0;
	double* d_PLCL = 0;
	double* d_PLFC = 0;
	double* d_cinh = 0;
	unsigned char* d_found = 0;
	
	CUDA_CHECK(hipMalloc((double**) &d_Tparcel, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**) &d_Piter, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**) &d_Titer, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**) &d_PLCL, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**) &d_PLFC, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**) &d_cinh, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((unsigned char**) &d_found, N * sizeof(unsigned char)));
	
	InitializeArray<double>(d_cinh, 0., N, stream);
	InitializeArray<double>(d_Tparcel, kFloatMissing, N, stream);
	InitializeArray<unsigned char>(d_found, 0, N, stream);

	CUDA_CHECK(hipMemcpyAsync(d_Titer, &Tsurf[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, h_basePenv->values, sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLCL, &PLCL[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLFC, &PLFC[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	
	std::vector<unsigned char> found(N, 0);
	
	curLevel.Value(curLevel.Value()-1);

	auto hPa150 = h->LevelForHeight(myTargetInfo->Producer(), 150.);

	while (curLevel.Value() > hPa150.first.Value())
	{

		auto ZenvInfo = Fetch(conf, ftime, curLevel, param("HL-M"), ftype);
		auto TenvInfo = Fetch(conf, ftime, curLevel, param("T-K"), ftype);
		auto PenvInfo = Fetch(conf, ftime, curLevel, param("P-HPA"), ftype);
	
		auto h_Zenv = PrepareInfo(ZenvInfo, stream);
		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);
		
		LiftLCLKernel <<< gridSize, blockSize, 0, stream >>> (d_Piter, d_Titer, d_PLCL, *h_Penv, d_Tparcel);
		
		CINKernel <<< gridSize, blockSize, 0, stream >>> (*h_Tenv, *h_Penv, d_Titer, d_Piter, *h_Zenv, *h_prevZenv, d_Tparcel, d_PLCL, d_PLFC, d_cinh, d_found);
		
		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipFree(h_prevPenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));
		CUDA_CHECK(hipFree(h_prevZenv->values));

		delete h_prevPenv;
		delete h_prevTenv;
		delete h_prevZenv;

		h_prevPenv = h_Penv;
		h_prevTenv = h_Tenv;
		h_prevZenv = h_Zenv;

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (static_cast<size_t> (std::count(found.begin(), found.end(), 1)) == found.size()) break;

		// preserve starting position for those grid points that have value

		CopyLFCIteratorValuesKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Tparcel, d_Piter, *h_Penv);
		
		curLevel.Value(curLevel.Value() - 1);

		
	}
	
	CUDA_CHECK(hipFree(h_prevPenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));
	CUDA_CHECK(hipFree(h_prevZenv->values));

	delete h_prevPenv;
	delete h_prevTenv;
	delete h_prevZenv;

	std::vector<double> cinh(N, 0);
	
	CUDA_CHECK(hipMemcpyAsync(&cinh[0], d_cinh, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_cinh));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_Piter));
	CUDA_CHECK(hipFree(d_Titer));
	CUDA_CHECK(hipFree(d_PLCL));
	CUDA_CHECK(hipFree(d_PLFC));
	CUDA_CHECK(hipFree(d_found));

	CUDA_CHECK(hipStreamDestroy(stream));

	myTargetInfo->Param(CINParam);
	myTargetInfo->Data().Set(cinh);

}

void si_cuda::GetCAPEGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo, const std::vector<double>& T, const std::vector<double>& P, param ELTParam, param ELPParam, param CAPEParam, param CAPE1040Param, param CAPE3kmParam)
{
	assert(T.size() == P.size());

	auto h = GET_PLUGIN(hitool);
	
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->HeightUnit(kHPa);

	// Found count determines if we have calculated all three CAPE variation for a single grid point
	std::vector<unsigned char> found(T.size(), 0);

	// No LFC --> No CAPE
	
	for (size_t i = 0; i < P.size(); i++)
	{
		if (P[i] == kFloatMissing)
		{
			found[i] |= FCAPE;
		}
	}
	
	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N/blockSize + (N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_CAPE = 0;
	double* d_CAPE1040 = 0;
	double* d_CAPE3km = 0;
	double* d_ELT = 0;
	double* d_ELP = 0;
	double* d_Titer = 0;
	double* d_Piter = 0;
	double* d_prevTparcel = 0;
	double* d_Tparcel = 0;
	double* d_LFC = 0;
	
	unsigned char* d_found = 0;
	
	CUDA_CHECK(hipMalloc((double**) &d_CAPE, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_CAPE1040, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_CAPE3km, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_ELP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_ELT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Piter, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Titer, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_Tparcel, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_prevTparcel, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**) &d_LFC, sizeof(double) * N));
	
	CUDA_CHECK(hipMalloc((double**) &d_found, sizeof(unsigned char) * N));
	
	InitializeArray<unsigned char> (d_found, 0, N, stream);

	//CUDA_CHECK(hipMemcpyAsync(d_TenvLCL, &TenvLCL[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Titer, &T[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Titer, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFC, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	
	//CUDA_CHECK(hipMemcpyAsync(d_LCLP, d_Piter, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
	
	InitializeArray<double> (d_CAPE, 0., N, stream);
	InitializeArray<double> (d_CAPE1040, 0., N, stream);
	InitializeArray<double> (d_CAPE3km, 0., N, stream);
	
	InitializeArray<double> (d_ELP, kFloatMissing, N, stream);
	InitializeArray<double> (d_ELT, kFloatMissing, N, stream);

	// For each grid point find the hybrid level that's below LFC and then pick the lowest level
	// among all grid points
		
	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P));
	
	level curLevel = levels.first;

	auto prevZenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("HL-M"), myTargetInfo->ForecastType());
	auto prevTenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
	auto prevPenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());
	
	auto h_prevZenv = PrepareInfo(prevZenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);
	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);
	
	curLevel.Value(curLevel.Value() - 1);
	
	auto hPa100 = h->LevelForHeight(myTargetInfo->Producer(), 100.);

	while (curLevel.Value() > hPa100.first.Value())
	{
		auto PenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());
		auto TenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto ZenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("HL-M"), myTargetInfo->ForecastType());

		auto h_Zenv = PrepareInfo(ZenvInfo, stream);
		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);
	
		MoistLiftKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Piter, *h_Penv, d_Tparcel);
			
		CAPEKernel <<< gridSize, blockSize, 0, stream >>> (*h_Tenv, *h_Penv, *h_Zenv, *h_prevTenv, *h_prevPenv, *h_prevZenv, d_Tparcel, d_prevTparcel, d_LFC, d_CAPE, d_CAPE1040, d_CAPE3km, d_ELT, d_ELP, d_found, curLevel.Value(), hPa100.first.Value());
		
		CUDA_CHECK(hipFree(h_prevZenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));
		CUDA_CHECK(hipFree(h_prevPenv->values));
		
		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
		
		delete h_prevZenv;
		delete h_prevPenv;
		delete h_prevTenv;
		
		h_prevZenv = h_Zenv;
		h_prevTenv = h_Tenv;
		h_prevPenv = h_Penv;

		CopyLFCIteratorValuesKernel <<< gridSize, blockSize, 0, stream >>> (d_Titer, d_Tparcel, d_Piter, *h_Penv);
		
		curLevel.Value(curLevel.Value() - 1);

	}
	
	CUDA_CHECK(hipFree(h_prevZenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));
	CUDA_CHECK(hipFree(h_prevPenv->values));

	delete h_prevZenv;
	delete h_prevPenv;
	delete h_prevTenv;

#if 0
		
	// If the CAPE area is continued all the way to level 60 and beyond, we don't have an EL for that
	// (since integration is forcefully stopped)
	// In this case level 60 = EL
	
	for (size_t i = 0; i < CAPE.size(); i++)
	{
		if (CAPE[i] > 0 && ELT[i] == kFloatMissing)
		{
			TenvInfo->LocationIndex(i);
			PenvInfo->LocationIndex(i);
			
			ELT[i] = TenvInfo->Value();
			ELP[i] = PenvInfo->Value();
		}
	}
#endif
	
	std::vector<double> CAPE(T.size(), 0);
	std::vector<double> CAPE1040(T.size(), 0);
	std::vector<double> CAPE3km(T.size(), 0);
	std::vector<double> ELT(T.size(), kFloatMissing);
	std::vector<double> ELP(T.size(), kFloatMissing);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipMemcpyAsync(&CAPE[0], d_CAPE, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&CAPE1040[0], d_CAPE1040, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&CAPE3km[0], d_CAPE3km, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&ELT[0], d_ELT, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&ELP[0], d_ELP, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_LFC));
	CUDA_CHECK(hipFree(d_found));
	
	CUDA_CHECK(hipStreamSynchronize(stream));
	
	CUDA_CHECK(hipFree(d_CAPE));
	CUDA_CHECK(hipFree(d_CAPE1040));
	CUDA_CHECK(hipFree(d_CAPE3km));
	CUDA_CHECK(hipFree(d_ELT));
	CUDA_CHECK(hipFree(d_ELP));
	
	myTargetInfo->Param(ELTParam);
	myTargetInfo->Data().Set(ELT);
	
	myTargetInfo->Param(ELPParam);
	myTargetInfo->Data().Set(ELP);

	myTargetInfo->Param(CAPEParam);
	myTargetInfo->Data().Set(CAPE);
	
	myTargetInfo->Param(CAPE1040Param);
	myTargetInfo->Data().Set(CAPE1040);

	myTargetInfo->Param(CAPE3kmParam);
	myTargetInfo->Data().Set(CAPE3km);

}