#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include <hip/hip_runtime.h>
#include <thrust/count.h>
#include <thrust/device_vector.h>

#include "plugin_factory.h"

#include "cape.cuh"
#include "cuda_helper.h"
#include "metutil.h"
#include "util.h"

#include <NFmiGribPacking.h>

#include "forecast_time.h"
#include "level.h"

#define HIMAN_AUXILIARY_INCLUDE

#include "cache.h"
#include "fetcher.h"
#include "hitool.h"

#include "debug.h"

#undef HIMAN_AUXILIARY_INCLUDE

using namespace himan;
using namespace himan::plugin;

himan::level cape_cuda::itsBottomLevel;

const unsigned char FCAPE = (1 << 2);
const unsigned char FCAPE3km = (1 << 0);

extern double Max(const std::vector<double>& vec);

template <typename T>
__global__ void InitializeArrayKernel(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = val;
	}
}

template <typename T>
void InitializeArray(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	InitializeArrayKernel<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

template <typename T>
__global__ void MultiplyWith(T* d_arr, T val, size_t N)
{
	int idx = threadIdx.x + blockDim.x * blockIdx.x;
	int stride = blockDim.x * gridDim.x;

	for (; idx < N; idx += stride)
	{
		d_arr[idx] = d_arr[idx] * val;
	}
}

template <typename T>
void MultiplyWith(T* d_arr, T val, size_t N, hipStream_t& stream)
{
	const int blockSize = 128;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	MultiplyWith<T><<<gridSize, blockSize, 0, stream>>>(d_arr, val, N);
}

info_simple* PrepareInfo(std::shared_ptr<himan::info> fullInfo, hipStream_t& stream)
{
	auto h_info = fullInfo->ToSimple();
	size_t N = h_info->size_x * h_info->size_y;

	ASSERT(N > 0);

	// 1. Reserve memory at device for unpacked data
	double* d_arr = 0;
	CUDA_CHECK(hipMalloc(reinterpret_cast<double**>(&d_arr), N * sizeof(double)));

	// 2. Unpack if needed, leave data to device and simultaneously copy it back to cpu (himan cache)
	auto tempGrid = fullInfo->Grid();

	if (tempGrid->IsPackedData())
	{
		ASSERT(tempGrid->PackedData().ClassName() == "simple_packed" ||
		       tempGrid->PackedData().ClassName() == "jpeg_packed");
		ASSERT(N > 0);
		ASSERT(tempGrid->Data().Size() == N);

		double* arr = const_cast<double*>(tempGrid->Data().ValuesAsPOD());
		CUDA_CHECK(hipHostRegister(reinterpret_cast<void*>(arr), sizeof(double) * N, 0));

		ASSERT(arr);

		tempGrid->PackedData().Unpack(d_arr, N, &stream);

		CUDA_CHECK(hipMemcpyAsync(arr, d_arr, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

		tempGrid->PackedData().Clear();

		auto c = GET_PLUGIN(cache);

		CUDA_CHECK(hipStreamSynchronize(stream));

		c->Insert(*fullInfo);

		CUDA_CHECK(hipHostUnregister(arr));

		h_info->packed_values = 0;
	}
	else
	{
		CUDA_CHECK(
		    hipMemcpyAsync(d_arr, fullInfo->Data().ValuesAsPOD(), sizeof(double) * N, hipMemcpyHostToDevice, stream));
	}

	h_info->values = d_arr;

	return h_info;
}

std::shared_ptr<himan::info> Fetch(const std::shared_ptr<const plugin_configuration> conf,
                                   const himan::forecast_time& theTime, const himan::level& theLevel,
                                   const himan::param& theParam, const himan::forecast_type& theType)
{
	try
	{
		auto f = GET_PLUGIN(fetcher);
		return f->Fetch(conf, theTime, theLevel, theParam, theType, true);
	}
	catch (HPExceptionType& e)
	{
		if (e != kFileDataNotFound)
		{
			throw std::runtime_error("cape_cuda::Fetch(): Unable to proceed");
		}

		return std::shared_ptr<info>();
	}
}

__global__ void CopyLFCIteratorValuesKernel(double* __restrict__ d_Titer, const double* __restrict__ d_Tparcel,
                                            double* __restrict__ d_Piter, info_simple d_Penv)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Penv.size_x * d_Penv.size_y)
	{
		if (!IsMissingDouble(d_Tparcel[idx]) && !IsMissingDouble(d_Penv.values[idx]))
		{
			d_Titer[idx] = d_Tparcel[idx];
			d_Piter[idx] = d_Penv.values[idx];
		}
	}
}

__global__ void LiftLCLKernel(const double* __restrict__ d_P, const double* __restrict__ d_T,
                              const double* __restrict__ d_PLCL, info_simple d_Ptarget, double* __restrict__ d_Tparcel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Ptarget.size_x * d_Ptarget.size_y)
	{
		ASSERT(d_P[idx] > 10);
		ASSERT(d_P[idx] < 1500 || IsMissingDouble(d_P[idx]));

		ASSERT(d_Ptarget.values[idx] > 10);
		ASSERT(d_Ptarget.values[idx] < 1500 || IsMissingDouble(d_Ptarget.values[idx]));

		ASSERT(d_T[idx] > 100);
		ASSERT(d_T[idx] < 350 || IsMissingDouble(d_T[idx]));

		double T = metutil::LiftLCL_(d_P[idx] * 100, d_T[idx], d_PLCL[idx] * 100, d_Ptarget.values[idx] * 100);

		ASSERT(T > 100);
		ASSERT(T < 350 || IsMissingDouble(T));

		d_Tparcel[idx] = T;
	}
}

__global__ void MoistLiftKernel(const double* __restrict__ d_T, const double* __restrict__ d_P, info_simple d_Ptarget,
                                double* __restrict__ d_Tparcel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T);
	ASSERT(d_P);

	if (idx < d_Ptarget.size_x * d_Ptarget.size_y)
	{
		ASSERT((d_P[idx] > 10 && d_P[idx] < 1500) || IsMissingDouble(d_P[idx]));
		ASSERT((d_Ptarget.values[idx] > 10 && d_Ptarget.values[idx] < 1500) || IsMissingDouble(d_Ptarget.values[idx]));

		ASSERT((d_T[idx] > 100 && d_T[idx] < 350) || IsMissingDouble(d_T[idx]));

		double T = metutil::MoistLiftA_(d_P[idx] * 100, d_T[idx], d_Ptarget.values[idx] * 100);

		ASSERT((T > 100 && T < 350) || IsMissingDouble(T));

		d_Tparcel[idx] = T;
	}
}

__global__ void CAPEKernel(info_simple d_Tenv, info_simple d_Penv, info_simple d_Zenv, info_simple d_prevTenv,
                           info_simple d_prevPenv, info_simple d_prevZenv, const double* __restrict d_Tparcel,
                           const double* __restrict d_prevTparcel, const double* __restrict__ d_LFCT,
                           const double* __restrict__ d_LFCP, double* __restrict__ d_CAPE,
                           double* __restrict__ d_CAPE1040, double* __restrict__ d_CAPE3km, double* __restrict__ d_ELT,
                           double* __restrict__ d_ELP, double* __restrict__ d_LastELT, double* __restrict__ d_LastELP,
                           unsigned char* __restrict__ d_found, int d_curLevel, int d_breakLevel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Tenv.size_x * d_Tenv.size_y && d_found[idx] != 4)
	{
		double Tenv = d_Tenv.values[idx];
		ASSERT(Tenv > 100.);

		double Penv = d_Penv.values[idx];  // hPa
		ASSERT(Penv < 1200.);

		double Zenv = d_Zenv.values[idx];  // m

		double prevTenv = d_prevTenv.values[idx];  // K
		ASSERT(prevTenv > 100.);

		double prevPenv = d_prevPenv.values[idx];  // hPa
		ASSERT(prevPenv < 1200.);

		double prevZenv = d_prevZenv.values[idx];  // m

		double Tparcel = d_Tparcel[idx];  // K
		ASSERT(Tparcel > 100. || IsMissingDouble(Tparcel));

		double prevTparcel = d_prevTparcel[idx];  // K
		ASSERT(prevTparcel > 100. || IsMissingDouble(Tparcel));

		double LFCP = d_LFCP[idx];  // hPa
		ASSERT(LFCP < 1200.);

		double LFCT = d_LFCT[idx];  // K
		ASSERT(LFCT > 100.);

		if (IsMissingDouble(Penv) || IsMissingDouble(Tenv) || IsMissingDouble(Zenv) || IsMissingDouble(prevZenv) ||
		    IsMissingDouble(Tparcel) || Penv > LFCP)
		{
			// Missing data or current grid point is below LFC
			return;
		}

		if (IsMissingDouble(prevTparcel) && !IsMissingDouble(Tparcel))
		{
			// When rising above LFC, get accurate value of Tenv at that level so that even small amounts of CAPE
			// (and EL!) values can be determined.

			prevTenv = himan::numerical_functions::interpolation::Linear(LFCP, prevPenv, Penv, prevTenv, Tenv);
			prevZenv = himan::numerical_functions::interpolation::Linear(LFCP, prevPenv, Penv, prevZenv, Zenv);
			prevPenv = LFCP;     // LFC pressure
			prevTparcel = LFCT;  // LFC temperature

			// If LFC was found close to lower hybrid level, the linear interpolation and moist lift will result
			// to same values. In this case CAPE integration fails as there is no area formed between environment
			// and parcel temperature. The result for this is that LFC is found but EL is not found. To prevent
			// this, warm the parcel value just slightly so that a miniscule CAPE area is formed and EL is found.

			if (fabs(prevTparcel - prevTenv) < 0.0001)
			{
				prevTparcel += 0.0001;
			}
		}

		if (d_curLevel < d_breakLevel && (Tenv - Tparcel) > 25.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] |= FCAPE;
		}
		else
		{
			if (prevZenv >= 3000. && Zenv >= 3000.)
			{
				d_found[idx] |= FCAPE3km;
			}

			if ((d_found[idx] & FCAPE3km) == 0)
			{
				double C = CAPE::CalcCAPE3km(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

				d_CAPE3km[idx] += C;

				ASSERT(d_CAPE3km[idx] < 3000.);  // 3000J/kg, not 3000m
				ASSERT(d_CAPE3km[idx] >= 0);
			}

			double C = CAPE::CalcCAPE1040(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);

			d_CAPE1040[idx] += C;

			ASSERT(d_CAPE1040[idx] < 5000.);
			ASSERT(d_CAPE1040[idx] >= 0);

			double CAPE, ELT, ELP;
			CAPE::CalcCAPE(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv, CAPE, ELT, ELP);

			d_CAPE[idx] += CAPE;

			ASSERT(CAPE >= 0.);
			ASSERT(d_CAPE[idx] < 8000);

			if (!IsMissingDouble(ELT))
			{
				if (IsMissingDouble(d_ELT[idx]))
				{
					d_ELT[idx] = ELT;
					d_ELP[idx] = ELP;
				}

				d_LastELT[idx] = ELT;
				d_LastELP[idx] = ELP;
			}
		}
	}
}

__global__ void CINKernel(info_simple d_Tenv, info_simple d_prevTenv, info_simple d_Penv, info_simple d_prevPenv,
                          info_simple d_Zenv, info_simple d_prevZenv, const double* __restrict__ d_Tparcel,
                          const double* __restrict__ d_prevTparcel, const double* __restrict__ d_PLCL,
                          const double* __restrict__ d_PLFC, const double* __restrict__ d_Psource,
                          double* __restrict__ d_cinh, unsigned char* __restrict__ d_found)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < d_Tenv.size_x * d_Tenv.size_y && d_found[idx] == 0)
	{
		double Tenv = d_Tenv.values[idx];  // K
		ASSERT(Tenv >= 150.);

		const double prevTenv = d_prevTenv.values[idx];

		double Penv = d_Penv.values[idx];  // hPa
		ASSERT(Penv < 1200. || IsMissingDouble(Penv));

		const double prevPenv = d_prevPenv.values[idx];

		double Tparcel = d_Tparcel[idx];  // K
		ASSERT(Tparcel >= 150. || IsMissingDouble(Tparcel));

		const double prevTparcel = d_prevTparcel[idx];

		double PLFC = d_PLFC[idx];  // hPa
		ASSERT(PLFC < 1200. || IsMissingDouble(PLFC));

		double PLCL = d_PLCL[idx];  // hPa
		ASSERT(PLCL < 1200. || IsMissingDouble(PLCL));

		double Zenv = d_Zenv.values[idx];          // m
		double prevZenv = d_prevZenv.values[idx];  // m

		// Make sure we have passed the starting level
		if (Penv <= d_Psource[idx])
		{
			if (Penv <= PLFC)
			{
				// reached max height
				d_found[idx] = 1;

				// Integrate the final piece from previous level to LFC level

				if (IsMissingDouble(prevTparcel) || IsMissingDouble(prevPenv) || IsMissingDouble(prevTenv))
				{
					Tparcel = MissingDouble();  // unable to proceed with CIN integration
				}
				else
				{
					// First get LFC height in meters
					Zenv = numerical_functions::interpolation::Linear(PLFC, prevPenv, Penv, prevZenv, Zenv);

					// LFC environment temperature value
					Tenv = numerical_functions::interpolation::Linear(PLFC, prevPenv, Penv, prevTenv, Tenv);

					// LFC T parcel value
					Tparcel = numerical_functions::interpolation::Linear(PLFC, prevPenv, Penv, prevTparcel, Tparcel);

					Penv = PLFC;

					ASSERT(Zenv > prevZenv);
				}
			}

			if (Penv < PLCL && !IsMissingDouble(Tparcel))
			{
				// Above LCL, switch to virtual temperature

				Tparcel = metutil::VirtualTemperature_(Tparcel, Penv * 100);
				Tenv = metutil::VirtualTemperature_(Tenv, Penv * 100);
			}

			if (!IsMissingDouble(Tparcel))
			{
				d_cinh[idx] += CAPE::CalcCIN(Tenv, prevTenv, Tparcel, prevTparcel, Penv, prevPenv, Zenv, prevZenv);
				ASSERT(d_cinh[idx] <= 0);
			}
		}
	}
}

__global__ void LFCKernel(info_simple d_T, info_simple d_P, info_simple d_prevT, info_simple d_prevP,
                          double* __restrict__ d_Tparcel, const double* __restrict__ d_prevTparcel,
                          const double* __restrict__ d_LCLT, const double* __restrict__ d_LCLP,
                          double* __restrict__ d_LFCT, double* __restrict__ d_LFCP, unsigned char* __restrict__ d_found,
                          int d_curLevel, int d_breakLevel)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T.values);
	ASSERT(d_P.values);

	if (idx < d_T.size_x * d_T.size_y && d_found[idx] == 0)
	{
		double Tparcel = d_Tparcel[idx];
		double prevTparcel = d_prevTparcel[idx];
		double Tenv = d_T.values[idx];

		ASSERT(Tenv < 350.);
		ASSERT(Tenv > 100.);

		double prevTenv = d_prevT.values[idx];
		ASSERT(prevTenv < 350.);
		ASSERT(prevTenv > 100.);

		double Penv = d_P.values[idx];
		double prevPenv = d_prevP.values[idx];

		double LCLP = d_LCLP[idx];

		if (!IsMissingDouble(Tparcel) && d_curLevel < d_breakLevel && (Tenv - Tparcel) > 30.)
		{
			// Temperature gap between environment and parcel too large --> abort search.
			// Only for values higher in the atmosphere, to avoid the effects of inversion

			d_found[idx] = 1;
		}

		if (!IsMissingDouble(Tparcel) && Penv <= LCLP && Tparcel > Tenv && d_found[idx] == 0)
		{
			d_found[idx] = 1;

			if (IsMissingDouble(prevTparcel))
			{
				prevTparcel = d_LCLT[idx];  // previous is LCL
				ASSERT(!IsMissingDouble(d_LCLT[idx]));
			}

			if (fabs(prevTparcel - prevTenv) < 0.0001)
			{
				d_LFCT[idx] = Tparcel;
				d_LFCP[idx] = Penv;
			}
			else
			{
				auto intersection = CAPE::GetPointOfIntersection(point(Tenv, Penv), point(prevTenv, prevPenv),
				                                                 point(Tparcel, Penv), point(prevTparcel, prevPenv));

				d_LFCT[idx] = intersection.X();
				d_LFCP[idx] = intersection.Y();

				if (IsMissingDouble(d_LFCT[idx]))
				{
					// Intersection not found, use exact level value
					d_LFCT[idx] = Tenv;
					d_LFCP[idx] = Penv;
				}
			}

			ASSERT(d_LFCT[idx] > 100);
			ASSERT(d_LFCT[idx] < 350);
		}
	}
}

__global__ void ThetaEKernel(info_simple d_T, info_simple d_RH, info_simple d_P, info_simple d_prevT,
                             info_simple d_prevRH, info_simple d_prevP, double* __restrict__ d_maxThetaE,
                             double* __restrict__ d_Tresult, double* __restrict__ d_TDresult,
                             double* __restrict__ d_Presult, unsigned char* __restrict__ d_found)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T.values);
	ASSERT(d_RH.values);
	ASSERT(d_P.values);

	if (idx < d_T.size_x * d_T.size_y && d_found[idx] == 0)
	{
		double T = d_T.values[idx];
		double P = d_P.values[idx];
		double RH = d_RH.values[idx];

		if (IsMissingDouble(P) || IsMissingDouble(T) || IsMissingDouble(RH))
		{
			d_found[idx] = 1;
		}
		else
		{
			if (P < 600.)
			{
				// Cut search if reach level 600hPa

				// Linearly interpolate temperature and humidity values to 600hPa, to check
				// if highest theta e is found there

				T = numerical_functions::interpolation::Linear(600., P, d_prevP.values[idx], T, d_prevT.values[idx]);
				RH = numerical_functions::interpolation::Linear(600., P, d_prevP.values[idx], RH, d_prevRH.values[idx]);

				d_found[idx] = 1;  // Make sure this is the last time we access this grid point
				P = 600.;
			}

			double TD = metutil::DewPointFromRH_(T, RH);

			double& refThetaE = d_maxThetaE[idx];
			double ThetaE = metutil::smarttool::ThetaE_(T, RH, P * 100);

			if (ThetaE >= refThetaE)
			{
				refThetaE = ThetaE;
				d_Tresult[idx] = T;
				d_TDresult[idx] = TD;
				d_Presult[idx] = P;
			}
		}
	}
}

__global__ void MixingRatioKernel(const double* __restrict__ d_T, double* __restrict__ d_P,
                                  const double* __restrict__ d_RH, double* __restrict__ d_Tpot,
                                  double* __restrict__ d_MR, size_t N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T);
	ASSERT(d_RH);
	ASSERT(d_P);

	if (idx < N)
	{
		double T = d_T[idx];
		double P = d_P[idx];
		double RH = d_RH[idx];

		ASSERT((T > 150 && T < 350) || IsMissingDouble(T));
		ASSERT((P > 100 && P < 1500) || IsMissingDouble(P));
		ASSERT((RH >= 0 && RH < 102) || IsMissingDouble(RH));

		if (IsMissingDouble(T) || IsMissingDouble(P) || IsMissingDouble(RH))
		{
			d_P[idx] = MissingDouble();
		}
		else
		{
			d_Tpot[idx] = metutil::Theta_(T, 100 * P);
			d_MR[idx] = metutil::smarttool::MixingRatio_(T, RH, 100 * P);

			d_P[idx] = P - 2.0;
		}
	}
}

__global__ void MixingRatioFinalizeKernel(double* __restrict__ d_T, double* __restrict__ d_TD, info_simple d_P,
                                          const double* __restrict__ d_Tpot, const double* __restrict__ d_MR, size_t N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	ASSERT(d_T);
	ASSERT(d_P.values);

	if (idx < N)
	{
		double P = d_P.values[idx];

		double MR = d_MR[idx];
		double Tpot = d_Tpot[idx];

		ASSERT((P > 100 && P < 1500) || IsMissingDouble(P));

		if (!IsMissingDouble(Tpot) && !IsMissingDouble(P))
		{
			d_T[idx] = Tpot * pow((P / 1000.), 0.2854);
		}

		double T = d_T[idx];

		if (!IsMissingDouble(T) && !IsMissingDouble(MR) && !IsMissingDouble(P))
		{
			double Es = metutil::Es_(T);  // Saturated water vapor pressure
			double E = metutil::E_(MR, 100 * P);

			double RH = E / Es * 100;
			d_TD[idx] = metutil::DewPointFromRH_(T, RH);
		}
	}
}

cape_source cape_cuda::GetHighestThetaEValuesGPU(const std::shared_ptr<const plugin_configuration> conf,
                                                 std::shared_ptr<info> myTargetInfo)
{
	himan::level curLevel = itsBottomLevel;

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_maxThetaE = 0;
	double* d_Tresult = 0;
	double* d_TDresult = 0;
	double* d_Presult = 0;
	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((double**)&d_maxThetaE, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Tresult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_TDresult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Presult, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_found, sizeof(unsigned char) * N));

	InitializeArray<double>(d_maxThetaE, -1, N, stream);
	InitializeArray<double>(d_Tresult, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_TDresult, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_Presult, himan::MissingDouble(), N, stream);
	InitializeArray<unsigned char>(d_found, 0, N, stream);

	info_simple* h_prevT = 0;
	info_simple* h_prevP = 0;
	info_simple* h_prevRH = 0;

	while (true)
	{
		auto TInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto RHInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("RH-PRCNT"), myTargetInfo->ForecastType());
		auto PInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

		if (!TInfo || !RHInfo || !PInfo)
		{
			return std::make_tuple(std::vector<double>(), std::vector<double>(), std::vector<double>());
		}

		auto h_T = PrepareInfo(TInfo, stream);
		auto h_P = PrepareInfo(PInfo, stream);
		auto h_RH = PrepareInfo(RHInfo, stream);

		ASSERT(h_T->values);
		ASSERT(h_RH->values);
		ASSERT(h_P->values);

		bool release = true;

		if (!h_prevT)
		{
			// first time
			h_prevT = new info_simple(*h_T);
			h_prevP = new info_simple(*h_P);
			h_prevRH = new info_simple(*h_RH);

			release = false;
		}

		ThetaEKernel<<<gridSize, blockSize, 0, stream>>>(*h_T, *h_RH, *h_P, *h_prevT, *h_prevRH, *h_prevP, d_maxThetaE,
		                                                 d_Tresult, d_TDresult, d_Presult, d_found);

		std::vector<unsigned char> found(N, 0);
		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipStreamSynchronize(stream));

		if (release)
		{
			CUDA_CHECK(hipFree(h_prevP->values));
			CUDA_CHECK(hipFree(h_prevRH->values));
			CUDA_CHECK(hipFree(h_prevT->values));
		}

		delete h_prevP;
		delete h_prevT;
		delete h_prevRH;

		h_prevP = h_P;
		h_prevRH = h_RH;
		h_prevT = h_T;

		curLevel.Value(curLevel.Value() - 1);

		size_t foundCount = std::count(found.begin(), found.end(), 1);

		if (foundCount == found.size()) break;
	}

	CUDA_CHECK(hipFree(h_prevP->values));
	CUDA_CHECK(hipFree(h_prevRH->values));
	CUDA_CHECK(hipFree(h_prevT->values));

	delete h_prevP;
	delete h_prevT;
	delete h_prevRH;

	std::vector<double> Tthetae(myTargetInfo->Data().Size());
	std::vector<double> TDthetae(myTargetInfo->Data().Size());
	std::vector<double> Pthetae(myTargetInfo->Data().Size());

	CUDA_CHECK(hipMemcpyAsync(&Tthetae[0], d_Tresult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&TDthetae[0], d_TDresult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&Pthetae[0], d_Presult, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_maxThetaE));
	CUDA_CHECK(hipFree(d_Tresult));
	CUDA_CHECK(hipFree(d_TDresult));
	CUDA_CHECK(hipFree(d_Presult));
	CUDA_CHECK(hipFree(d_found));

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_tuple(Tthetae, TDthetae, Pthetae);
}

cape_source cape_cuda::Get500mMixingRatioValuesGPU(std::shared_ptr<const plugin_configuration> conf,
                                                   std::shared_ptr<info> myTargetInfo)
{
	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	level curLevel = itsBottomLevel;

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());

	modifier_mean tp, mr;

	tp.HeightInMeters(false);
	mr.HeightInMeters(false);

	auto f = GET_PLUGIN(fetcher);
	auto PInfo = f->Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType(), false);

	if (!PInfo)
	{
		return std::make_tuple(std::vector<double>(), std::vector<double>(), std::vector<double>());
	}
	else
	{
		// Himan specialty: empty data grid

		size_t miss = 0;
		for (auto& val : VEC(PInfo))
		{
			if (IsMissingDouble(val)) miss++;
		}

		if (PInfo->Data().MissingCount() == PInfo->Data().Size())
		{
			return std::make_tuple(std::vector<double>(), std::vector<double>(), std::vector<double>());
		}
	}

	auto PVec = VEC(PInfo);

	auto P500m = h->VerticalValue(param("P-HPA"), 500.);

	h->HeightUnit(kHPa);

	tp.LowerHeight(PVec);
	mr.LowerHeight(PVec);

	tp.UpperHeight(P500m);
	mr.UpperHeight(P500m);

	double* d_Tpot = 0;
	double* d_MR = 0;
	double* d_T = 0;
	double* d_RH = 0;
	double* d_P = 0;
	double* d_TD = 0;

	CUDA_CHECK(hipMalloc((double**)&d_Tpot, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_MR, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_T, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_RH, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_P, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_TD, N * sizeof(double)));

	InitializeArray<double>(d_Tpot, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_MR, himan::MissingDouble(), N, stream);

	while (true)
	{
		auto TVec = h->VerticalValue(param("T-K"), PVec);
		auto RHVec = h->VerticalValue(param("RH-PRCNT"), PVec);

		CUDA_CHECK(hipMemcpyAsync(d_T, &TVec[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_RH, &RHVec[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
		CUDA_CHECK(hipMemcpyAsync(d_P, &PVec[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));

		MixingRatioKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_P, d_RH, d_Tpot, d_MR, N);

		std::vector<double> Tpot(N, himan::MissingDouble());
		std::vector<double> MR(N, himan::MissingDouble());

		CUDA_CHECK(hipStreamSynchronize(stream));

		CUDA_CHECK(hipMemcpyAsync(&Tpot[0], d_Tpot, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(&MR[0], d_MR, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipStreamSynchronize(stream));

		tp.Process(Tpot, PVec);
		mr.Process(MR, PVec);

		size_t foundCount = tp.HeightsCrossed();

		ASSERT(tp.HeightsCrossed() == mr.HeightsCrossed());

		if (foundCount == N)
		{
			break;
		}

		CUDA_CHECK(hipMemcpyAsync(&PVec[0], d_P, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	}

	CUDA_CHECK(hipStreamSynchronize(stream));

	// Calculate averages

	auto Tpot = tp.Result();
	auto MR = mr.Result();

	// Copy averages to GPU for final calculation
	CUDA_CHECK(hipMemcpyAsync(d_Tpot, &Tpot[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_MR, &MR[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));

	auto Psurf = Fetch(conf, myTargetInfo->Time(), itsBottomLevel, param("P-HPA"), myTargetInfo->ForecastType());
	auto h_P = PrepareInfo(Psurf, stream);

	InitializeArray<double>(d_T, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_TD, himan::MissingDouble(), N, stream);

	std::vector<double> T(Tpot.size(), himan::MissingDouble());
	std::vector<double> TD(T.size(), himan::MissingDouble());

	MixingRatioFinalizeKernel<<<gridSize, blockSize, 0, stream>>>(d_T, d_TD, *h_P, d_Tpot, d_MR, N);

	CUDA_CHECK(hipMemcpyAsync(&T[0], d_T, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&TD[0], d_TD, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_Tpot));
	CUDA_CHECK(hipFree(d_MR));
	CUDA_CHECK(hipFree(d_RH));
	CUDA_CHECK(hipFree(d_P));
	CUDA_CHECK(hipFree(d_T));
	CUDA_CHECK(hipFree(d_TD));

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_tuple(T, TD, VEC(Psurf));
}

std::pair<std::vector<double>, std::vector<double>> cape_cuda::GetLFCGPU(
    const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo, std::vector<double>& T,
    std::vector<double>& P, std::vector<double>& TenvLCL)
{
	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_TenvLCL = 0;
	double* d_Titer = 0;
	double* d_Piter = 0;
	double* d_LCLP = 0;
	double* d_LCLT = 0;
	double* d_LFCT = 0;
	double* d_LFCP = 0;
	double* d_Tparcel = 0;
	double* d_prevTparcel = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((double**)&d_TenvLCL, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Piter, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Titer, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LCLT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LCLP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LFCT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LFCP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_found, sizeof(unsigned char) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Tparcel, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_prevTparcel, sizeof(double) * N));

	CUDA_CHECK(hipMemcpyAsync(d_TenvLCL, &TenvLCL[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Titer, &T[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_LCLT, d_Titer, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LCLP, d_Piter, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));

	InitializeArray<double>(d_LFCT, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_LFCP, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_prevTparcel, himan::MissingDouble(), N, stream);
	InitializeArray<unsigned char>(d_found, 0, N, stream);

	// For each grid point find the hybrid level that's below LCL and then pick the lowest level
	// among all grid points; most commonly it's the lowest hybrid level

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P));

	level curLevel = levels.first;

	auto prevPenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());
	auto prevTenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());

	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);

	ASSERT(h_prevTenv->values);
	ASSERT(h_prevPenv->values);

	curLevel.Value(curLevel.Value() - 1);

	std::vector<unsigned char> found(N, 0);
	std::vector<double> LFCT(N, himan::MissingDouble());
	std::vector<double> LFCP(N, himan::MissingDouble());

	for (size_t i = 0; i < N; i++)
	{
		if ((T[i] - TenvLCL[i]) > 0.001)
		{
			found[i] = 1;
			LFCT[i] = T[i];
			LFCP[i] = P[i];
		}
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCT, &LFCT[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCP, &LFCP[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipStreamSynchronize(stream));

	auto hPa450 = h->LevelForHeight(myTargetInfo->Producer(), 450.);
	auto hPa150 = h->LevelForHeight(myTargetInfo->Producer(), 150.);

	while (curLevel.Value() > hPa150.first.Value())
	{
		// Get environment temperature and pressure values for this level
		auto TenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto PenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);

		// Lift the particle from previous level to this level. In the first revolution
		// of this loop the starting level is LCL. If target level level is below current level
		// (ie. we would be lowering the particle) missing value is returned.

		MoistLiftKernel<<<gridSize, blockSize, 0, stream>>>(d_Titer, d_Piter, *h_Penv, d_Tparcel);

		LFCKernel<<<gridSize, blockSize, 0, stream>>>(*h_Tenv, *h_Penv, *h_prevTenv, *h_prevPenv, d_Tparcel,
		                                              d_prevTparcel, d_LCLT, d_LCLP, d_LFCT, d_LFCP, d_found,
		                                              curLevel.Value(), hPa450.first.Value());

		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));

		CUDA_CHECK(hipFree(h_prevPenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));

		delete h_prevPenv;
		delete h_prevTenv;

		h_prevPenv = h_Penv;
		h_prevTenv = h_Tenv;

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (static_cast<size_t>(std::count(found.begin(), found.end(), 1)) == found.size()) break;

		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
		curLevel.Value(curLevel.Value() - 1);
	}

	CUDA_CHECK(hipFree(h_prevPenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));

	delete h_prevPenv;
	delete h_prevTenv;

	CUDA_CHECK(hipMemcpyAsync(&LFCT[0], d_LFCT, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&LFCP[0], d_LFCP, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_LCLT));
	CUDA_CHECK(hipFree(d_LCLP));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_found));
	CUDA_CHECK(hipFree(d_Titer));
	CUDA_CHECK(hipFree(d_Piter));
	CUDA_CHECK(hipFree(d_TenvLCL));

	CUDA_CHECK(hipStreamDestroy(stream));

	return std::make_pair(LFCT, LFCP);
}

void cape_cuda::GetCINGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo,
                          const std::vector<double>& Tsource, const std::vector<double>& Psource,
                          const std::vector<double>& TLCL, const std::vector<double>& PLCL,
                          const std::vector<double>& PLFC)
{
	const params PParams({param("PGR-PA"), param("P-PA")});

	auto h = GET_PLUGIN(hitool);
	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	forecast_time ftime = myTargetInfo->Time();
	forecast_type ftype = myTargetInfo->ForecastType();

	/*
	 * Modus operandi:
	 *
	 * 1. Integrate from ground to LCL dry adiabatically
	 *
	 * This can be done always since LCL is known at all grid points
	 * (that have source data values defined).
	 *
	 * 2. Integrate from LCL to LFC moist adiabatically
	 *
	 * Note! For some points integration will fail (no LFC found)
	 *
	 * We stop integrating at first time CAPE area is found!
	 */

	// Get LCL and LFC heights in meters

	auto ZLCL = h->VerticalValue(param("HL-M"), PLCL);
	auto ZLFC = h->VerticalValue(param("HL-M"), PLFC);

	level curLevel = itsBottomLevel;

	auto prevZenvInfo = Fetch(conf, ftime, curLevel, param("HL-M"), ftype);
	auto prevTenvInfo = Fetch(conf, ftime, curLevel, param("T-K"), ftype);
	auto prevPenvInfo = Fetch(conf, ftime, curLevel, param("P-HPA"), ftype);

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	auto h_prevZenv = PrepareInfo(prevZenvInfo, stream);
	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);

	double* d_Psource = 0;
	double* d_Tparcel = 0;
	double* d_prevTparcel = 0;
	double* d_Piter = 0;
	double* d_prevPiter = 0;
	double* d_Titer = 0;
	double* d_prevTiter = 0;
	double* d_PLCL = 0;
	double* d_PLFC = 0;
	double* d_cinh = 0;
	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((double**)&d_Psource, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_Tparcel, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_prevTparcel, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_Piter, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_Titer, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_PLCL, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_PLFC, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((double**)&d_cinh, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((unsigned char**)&d_found, N * sizeof(unsigned char)));

	InitializeArray<double>(d_cinh, 0., N, stream);
	InitializeArray<double>(d_Tparcel, himan::MissingDouble(), N, stream);

	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, &Tsource[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Psource, &Psource[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Titer, &Tsource[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, &Psource[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLCL, &PLCL[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_PLFC, &PLFC[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));

	std::vector<unsigned char> found(N, 0);

	for (size_t i = 0; i < PLFC.size(); i++)
	{
		if (IsMissingDouble(PLFC[i])) found[i] = true;
	}

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));

	curLevel.Value(curLevel.Value() - 1);

	auto hPa100 = h->LevelForHeight(myTargetInfo->Producer(), 100.);

	while (curLevel.Value() > hPa100.first.Value())
	{
		auto ZenvInfo = Fetch(conf, ftime, curLevel, param("HL-M"), ftype);
		auto TenvInfo = Fetch(conf, ftime, curLevel, param("T-K"), ftype);
		auto PenvInfo = Fetch(conf, ftime, curLevel, param("P-HPA"), ftype);

		auto h_Zenv = PrepareInfo(ZenvInfo, stream);
		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);

		LiftLCLKernel<<<gridSize, blockSize, 0, stream>>>(d_Piter, d_Titer, d_PLCL, *h_Penv, d_Tparcel);

		CINKernel<<<gridSize, blockSize, 0, stream>>>(*h_Tenv, *h_prevTenv, *h_Penv, *h_prevPenv, *h_Zenv, *h_prevZenv,
		                                              d_Tparcel, d_prevTparcel, d_PLCL, d_PLFC, d_Psource, d_cinh,
		                                              d_found);

		CUDA_CHECK(hipMemcpyAsync(&found[0], d_found, sizeof(unsigned char) * N, hipMemcpyDeviceToHost, stream));
		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));

		CUDA_CHECK(hipFree(h_prevPenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));
		CUDA_CHECK(hipFree(h_prevZenv->values));

		delete h_prevPenv;
		delete h_prevTenv;
		delete h_prevZenv;

		h_prevPenv = h_Penv;
		h_prevTenv = h_Tenv;
		h_prevZenv = h_Zenv;

		CUDA_CHECK(hipStreamSynchronize(stream));

		if (static_cast<size_t>(std::count(found.begin(), found.end(), 1)) == found.size()) break;

		// preserve starting position for those grid points that have value

		CopyLFCIteratorValuesKernel<<<gridSize, blockSize, 0, stream>>>(d_Titer, d_Tparcel, d_Piter, *h_Penv);

		curLevel.Value(curLevel.Value() - 1);
	}

	CUDA_CHECK(hipFree(h_prevPenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));
	CUDA_CHECK(hipFree(h_prevZenv->values));

	delete h_prevPenv;
	delete h_prevTenv;
	delete h_prevZenv;

	std::vector<double> cinh(N, 0);

	CUDA_CHECK(hipMemcpyAsync(&cinh[0], d_cinh, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK(hipFree(d_cinh));
	CUDA_CHECK(hipFree(d_Psource));
	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_Piter));
	CUDA_CHECK(hipFree(d_prevPiter));
	CUDA_CHECK(hipFree(d_Titer));
	CUDA_CHECK(hipFree(d_prevTiter));
	CUDA_CHECK(hipFree(d_PLCL));
	CUDA_CHECK(hipFree(d_PLFC));
	CUDA_CHECK(hipFree(d_found));

	CUDA_CHECK(hipStreamDestroy(stream));

	myTargetInfo->Param(CINParam);
	myTargetInfo->Data().Set(cinh);
}

void cape_cuda::GetCAPEGPU(const std::shared_ptr<const plugin_configuration> conf, std::shared_ptr<info> myTargetInfo,
                           const std::vector<double>& T, const std::vector<double>& P)
{
	ASSERT(T.size() == P.size());

	auto h = GET_PLUGIN(hitool);

	h->Configuration(conf);
	h->Time(myTargetInfo->Time());
	h->ForecastType(myTargetInfo->ForecastType());
	h->HeightUnit(kHPa);

	// Found count determines if we have calculated all three CAPE variation for a single grid point
	std::vector<unsigned char> found(T.size(), 0);

	// No LFC --> No CAPE

	for (size_t i = 0; i < P.size(); i++)
	{
		if (IsMissingDouble(P[i]))
		{
			found[i] |= FCAPE;
		}
	}

	const size_t N = myTargetInfo->Data().Size();
	const int blockSize = 256;
	const int gridSize = N / blockSize + (N % blockSize == 0 ? 0 : 1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_CAPE = 0;
	double* d_CAPE1040 = 0;
	double* d_CAPE3km = 0;
	double* d_ELT = 0;
	double* d_ELP = 0;
	double* d_LastELT = 0;
	double* d_LastELP = 0;
	double* d_Titer = 0;
	double* d_Piter = 0;
	double* d_prevTparcel = 0;
	double* d_Tparcel = 0;
	double* d_LFCT = 0;
	double* d_LFCP = 0;

	unsigned char* d_found = 0;

	CUDA_CHECK(hipMalloc((double**)&d_CAPE, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_CAPE1040, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_CAPE3km, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_ELP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_ELT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LastELP, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LastELT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Piter, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Titer, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_Tparcel, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_prevTparcel, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LFCT, sizeof(double) * N));
	CUDA_CHECK(hipMalloc((double**)&d_LFCP, sizeof(double) * N));

	CUDA_CHECK(hipMalloc((double**)&d_found, sizeof(unsigned char) * N));

	InitializeArray<unsigned char>(d_found, 0, N, stream);

	CUDA_CHECK(hipMemcpyAsync(d_Titer, &T[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Titer, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_Piter, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCT, &T[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));
	CUDA_CHECK(hipMemcpyAsync(d_LFCP, &P[0], sizeof(double) * N, hipMemcpyHostToDevice, stream));

	CUDA_CHECK(hipMemcpyAsync(d_found, &found[0], sizeof(unsigned char) * N, hipMemcpyHostToDevice, stream));

	InitializeArray<double>(d_CAPE, 0., N, stream);
	InitializeArray<double>(d_CAPE1040, 0., N, stream);
	InitializeArray<double>(d_CAPE3km, 0., N, stream);

	InitializeArray<double>(d_ELP, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_ELT, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_LastELP, himan::MissingDouble(), N, stream);
	InitializeArray<double>(d_LastELT, himan::MissingDouble(), N, stream);

	// For each grid point find the hybrid level that's below LFC and then pick the lowest level
	// among all grid points

	auto levels = h->LevelForHeight(myTargetInfo->Producer(), ::Max(P));

	level curLevel = levels.first;

	auto prevZenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("HL-M"), myTargetInfo->ForecastType());
	auto prevTenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
	auto prevPenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());

	auto h_prevZenv = PrepareInfo(prevZenvInfo, stream);
	auto h_prevPenv = PrepareInfo(prevPenvInfo, stream);
	auto h_prevTenv = PrepareInfo(prevTenvInfo, stream);

	curLevel.Value(curLevel.Value());

	auto hPa100 = h->LevelForHeight(myTargetInfo->Producer(), 100.);

	while (curLevel.Value() > hPa100.first.Value())
	{
		auto PenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("P-HPA"), myTargetInfo->ForecastType());
		auto TenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("T-K"), myTargetInfo->ForecastType());
		auto ZenvInfo = Fetch(conf, myTargetInfo->Time(), curLevel, param("HL-M"), myTargetInfo->ForecastType());

		auto h_Zenv = PrepareInfo(ZenvInfo, stream);
		auto h_Penv = PrepareInfo(PenvInfo, stream);
		auto h_Tenv = PrepareInfo(TenvInfo, stream);

		MoistLiftKernel<<<gridSize, blockSize, 0, stream>>>(d_Titer, d_Piter, *h_Penv, d_Tparcel);

		CAPEKernel<<<gridSize, blockSize, 0, stream>>>(*h_Tenv, *h_Penv, *h_Zenv, *h_prevTenv, *h_prevPenv, *h_prevZenv,
		                                               d_Tparcel, d_prevTparcel, d_LFCT, d_LFCP, d_CAPE, d_CAPE1040,
		                                               d_CAPE3km, d_ELT, d_ELP, d_LastELT, d_LastELP, d_found,
		                                               curLevel.Value(), hPa100.first.Value());

		CUDA_CHECK(hipFree(h_prevZenv->values));
		CUDA_CHECK(hipFree(h_prevTenv->values));
		CUDA_CHECK(hipFree(h_prevPenv->values));

		CUDA_CHECK(hipMemcpyAsync(d_prevTparcel, d_Tparcel, sizeof(double) * N, hipMemcpyDeviceToDevice, stream));

		delete h_prevZenv;
		delete h_prevPenv;
		delete h_prevTenv;

		h_prevZenv = h_Zenv;
		h_prevTenv = h_Tenv;
		h_prevPenv = h_Penv;

		curLevel.Value(curLevel.Value() - 1);
	}

	CUDA_CHECK(hipFree(h_prevZenv->values));
	CUDA_CHECK(hipFree(h_prevTenv->values));
	CUDA_CHECK(hipFree(h_prevPenv->values));

	delete h_prevZenv;
	delete h_prevPenv;
	delete h_prevTenv;

#if 0
		
	// If the CAPE area is continued all the way to level 60 and beyond, we don't have an EL for that
	// (since integration is forcefully stopped)
	// In this case level 60 = EL
	
	for (size_t i = 0; i < CAPE.size(); i++)
	{
		if (CAPE[i] > 0 && ELT[i] == MissingDouble())
		{
			TenvInfo->LocationIndex(i);
			PenvInfo->LocationIndex(i);
			
			ELT[i] = TenvInfo->Value();
			ELP[i] = PenvInfo->Value();
		}
	}
#endif

	std::vector<double> CAPE(T.size(), 0);
	std::vector<double> CAPE1040(T.size(), 0);
	std::vector<double> CAPE3km(T.size(), 0);
	std::vector<double> ELT(T.size(), himan::MissingDouble());
	std::vector<double> ELP(T.size(), himan::MissingDouble());
	std::vector<double> LastELT(T.size(), himan::MissingDouble());
	std::vector<double> LastELP(T.size(), himan::MissingDouble());

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipMemcpyAsync(&CAPE[0], d_CAPE, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&CAPE1040[0], d_CAPE1040, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&CAPE3km[0], d_CAPE3km, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&ELT[0], d_ELT, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&ELP[0], d_ELP, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&LastELT[0], d_LastELT, sizeof(double) * N, hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(&LastELP[0], d_LastELP, sizeof(double) * N, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipFree(d_Tparcel));
	CUDA_CHECK(hipFree(d_prevTparcel));
	CUDA_CHECK(hipFree(d_LFCT));
	CUDA_CHECK(hipFree(d_LFCP));
	CUDA_CHECK(hipFree(d_found));

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipFree(d_CAPE));
	CUDA_CHECK(hipFree(d_CAPE1040));
	CUDA_CHECK(hipFree(d_CAPE3km));
	CUDA_CHECK(hipFree(d_ELT));
	CUDA_CHECK(hipFree(d_ELP));
	CUDA_CHECK(hipFree(d_LastELT));
	CUDA_CHECK(hipFree(d_LastELP));

	myTargetInfo->Param(ELTParam);
	myTargetInfo->Data().Set(ELT);

	myTargetInfo->Param(ELPParam);
	myTargetInfo->Data().Set(ELP);

	myTargetInfo->Param(LastELTParam);
	myTargetInfo->Data().Set(LastELT);

	myTargetInfo->Param(LastELPParam);
	myTargetInfo->Data().Set(LastELP);

	myTargetInfo->Param(CAPEParam);
	myTargetInfo->Data().Set(CAPE);

	myTargetInfo->Param(CAPE1040Param);
	myTargetInfo->Data().Set(CAPE1040);

	myTargetInfo->Param(CAPE3kmParam);
	myTargetInfo->Data().Set(CAPE3km);
}
