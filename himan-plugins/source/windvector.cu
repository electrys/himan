#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

#include "cuda_plugin_helper.h"
#include "windvector.cuh"

#define MIN(a, b) (((a) < (b)) ? (a) : (b))
#define MAX(a, b) (((a) > (b)) ? (a) : (b))

/*
 * Calculate results. At this point it as assumed that U and V are in correct form.
 */

__global__ void Calculate(cdarr_t d_u, cdarr_t d_v, darr_t d_speed, darr_t d_dir,
                          himan::plugin::windvector_cuda::options opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	using himan::kFloatMissing;

	if (idx < opts.N)
	{
		double U = d_u[idx], V = d_v[idx];
		d_speed[idx] = kFloatMissing;
		if (d_dir) d_dir[idx] = kFloatMissing;

		if (U != kFloatMissing && V != kFloatMissing)
		{
			double speed = sqrt(U * U + V * V);

			d_speed[idx] = speed;

			double dir = 0;

			if (opts.target_type != himan::plugin::kGust)
			{
				int offset = 180;

				if (opts.target_type == himan::plugin::kSea || opts.target_type == himan::plugin::kIce)
				{
					offset = 0;
				}

				if (speed > 0)
				{
					dir = himan::constants::kRad * atan2(U, V) + offset;

					// modulo operator is supposedly slow on cuda ?

					/*
					 * quote:
					 *
					 * Integer division and modulo operation are costly: tens of instructions on devices of
					 * compute capability 1.x, below 20 instructions on devices of compute capability 2.x and
					 * higher.
					 */

					// reduce the angle
					while (dir > 360)
					{
						dir -= 360;
					}

					// force it to be the positive remainder, so that 0 <= dir < 360

					while (dir < 0)
					{
						dir += 360;
					}
				}

				d_dir[idx] = round(dir);
			}
		}
	}
}

/*
 * Rotate U and V vectors that are grid-relative (U is pointing to grid north, V to grid east) to earth
 * relative form (U points to earth or map north etc).
 */

__global__ void RotateLambert(double* __restrict__ d_u, double* __restrict__ d_v, const double* __restrict__ d_lon,
                              const double* __restrict__ d_lat, double cone, double orientation,
                              himan::info_simple opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.size_x * opts.size_y)
	{
		double U = d_u[idx];
		double V = d_v[idx];

		if (U != himan::kFloatMissing && V != himan::kFloatMissing)
		{
			int i = fmod(static_cast<double>(idx), static_cast<double>(opts.size_x));  // idx - j * opts.size_x;
			int j = floor(static_cast<double>(idx / opts.size_x));

			double londiff = d_lon[idx] - orientation;
			const double angle = cone * londiff * himan::constants::kDeg;
			double sinx, cosx;
			sincos(angle, &sinx, &cosx);
			d_u[idx] = -1 * cosx * U + sinx * V;
			d_v[idx] = -1 * -sinx * U + cosx * V;
		}
	}
}

__global__ void Rotate(double* __restrict__ d_u, double* __restrict__ d_v, himan::info_simple opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.size_x * opts.size_y)
	{
		double U = d_u[idx];
		double V = d_v[idx];

		if (U != himan::kFloatMissing && V != himan::kFloatMissing)
		{
			int i = fmod(static_cast<double>(idx), static_cast<double>(opts.size_x));  // idx - j * opts.size_x;
			int j = floor(static_cast<double>(idx / opts.size_x));

			double lon = opts.first_lon + i * opts.di;

			double lat = himan::kFloatMissing;

			if (opts.j_scans_positive)
			{
				lat = opts.first_lat + j * opts.dj;
			}
			else
			{
				lat = opts.first_lat - j * opts.dj;
			}

			double SinYPole = sin((opts.south_pole_lat + 90.) * himan::constants::kDeg);
			double CosYPole = cos((opts.south_pole_lat + 90.) * himan::constants::kDeg);

			double SinXRot, CosXRot, SinYRot, CosYRot;

			sincos(lon * himan::constants::kDeg, &SinXRot, &CosXRot);
			sincos(lat * himan::constants::kDeg, &SinYRot, &CosYRot);

			double SinYReg = CosYPole * SinYRot + SinYPole * CosYRot * CosXRot;

			SinYReg = MIN(MAX(SinYReg, -1.), 1.);

			double YReg = asin(SinYReg) * himan::constants::kRad;

			double CosYReg = cos(YReg * himan::constants::kDeg);

			double CosXReg = (CosYPole * CosYRot * CosXRot - SinYPole * SinYRot) / CosYReg;

			CosXReg = MIN(MAX(CosXReg, -1.), 1.);
			double SinXReg = CosYRot * SinXRot / CosYReg;

			double XReg = acos(CosXReg) * himan::constants::kRad;

			if (SinXReg < 0.)
			{
				XReg = -XReg;
			}

			XReg += opts.south_pole_lon;

			// UV to earth relative

			double zxmxc = himan::constants::kDeg * (XReg - opts.south_pole_lon);

			double sinxmxc, cosxmxc;

			sincos(zxmxc, &sinxmxc, &cosxmxc);

			double PA = cosxmxc * CosXRot + CosYPole * sinxmxc * SinXRot;
			double PB =
			    CosYPole * sinxmxc * CosXRot * SinYRot + SinYPole * sinxmxc * CosYRot - cosxmxc * SinXRot * SinYRot;
			double PC = (-SinYPole) * SinXRot / CosYReg;
			double PD = (CosYPole * CosYRot - SinYPole * CosXRot * SinYRot) / CosYReg;

			double newU = PA * U + PB * V;
			double newV = PC * U + PD * V;

			d_u[idx] = newU;
			d_v[idx] = newV;
		}
	}
}

void himan::plugin::windvector_cuda::Process(options& opts)
{
	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	// Allocate device arrays

	double* d_u = 0;
	double* d_v = 0;
	double* d_speed = 0;
	double* d_dir = 0;
	double* d_lon = 0;
	double* d_lat = 0;

	// Allocate memory on device

	size_t memsize = opts.N * sizeof(double);

	CUDA_CHECK(hipMalloc((void**)&d_u, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_v, memsize));

	CUDA_CHECK(hipMalloc((void**)&d_speed, memsize));

	if (opts.target_type != kGust)
	{
		CUDA_CHECK(hipMalloc((void**)&d_dir, memsize));
		PrepareInfo(opts.dir);
	}

	// Copy data to device

	PrepareInfo(opts.u, d_u, stream);
	PrepareInfo(opts.v, d_v, stream);
	PrepareInfo(opts.speed);

	// dims

	const int blockSize = 256;
	const int gridSize = opts.N / blockSize + (opts.N % blockSize == 0 ? 0 : 1);

	/*
	 *  If calculating gust, do not ever rotate grid since we don't calculate
	 * direction for it.
	*/

	CUDA_CHECK(hipStreamSynchronize(stream));

	if (opts.target_type != kGust && opts.need_grid_rotation)
	{
		if (opts.u->projection == kRotatedLatitudeLongitude)
		{
			if (opts.u->south_pole_lat > 0)
			{
				opts.u->south_pole_lat = -opts.u->south_pole_lat;
				opts.u->south_pole_lon = 0;
			}

			Rotate<<<gridSize, blockSize, 0, stream>>>(d_u, d_v, *opts.u);
		}
		else if (opts.u->projection == kLambertConformalConic)
		{
			const double latin1 = opts.u->latin1;
			const double latin2 = opts.u->latin2;
			assert(latin1 != kFloatMissing);
			double cone;
			if (latin1 == latin2 && latin2 != kFloatMissing)
			{
				cone = sin(latin1 * constants::kDeg);
			}
			else
			{
				cone = (log(cos(latin1 * constants::kDeg)) - log(cos(latin2 * constants::kDeg))) /
				       (log(tan((90 - fabs(latin1)) * constants::kDeg * 0.5)) -
				        log(tan(90 - fabs(latin2)) * constants::kDeg * 0.5));
			}

			CUDA_CHECK(hipMalloc((void**)&d_lon, memsize));
			CUDA_CHECK(hipMalloc((void**)&d_lat, memsize));

			CUDA_CHECK(hipMemcpyAsync(d_lon, opts.lon, memsize, hipMemcpyHostToDevice, stream));
			CUDA_CHECK(hipMemcpyAsync(d_lat, opts.lat, memsize, hipMemcpyHostToDevice, stream));

			RotateLambert<<<gridSize, blockSize, 0, stream>>>(d_u, d_v, d_lon, d_lat, cone, opts.u->orientation,
			                                                  *opts.u);
		}
	}

	Calculate<<<gridSize, blockSize, 0, stream>>>(d_u, d_v, d_speed, d_dir, opts);

	// block until the stream has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	ReleaseInfo(opts.u);
	ReleaseInfo(opts.v);
	ReleaseInfo(opts.speed, d_speed, stream);

	if (opts.target_type != kGust)
	{
		ReleaseInfo(opts.dir, d_dir, stream);
	}

	CUDA_CHECK(hipStreamSynchronize(stream));

	// Free device memory

	CUDA_CHECK(hipFree(d_u));
	CUDA_CHECK(hipFree(d_v));
	CUDA_CHECK(hipFree(d_speed));

	if (d_dir)
	{
		CUDA_CHECK(hipFree(d_dir));
	}

	CUDA_CHECK(hipStreamDestroy(stream));
}
