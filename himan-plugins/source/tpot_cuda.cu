#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "tpot_cuda.h"

namespace himan
{

namespace plugin
{

namespace tpot_cuda
{

__global__ void UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dPPacked,
									double* dT,
									double* dP,
									double* dTP,
									tpot_cuda_options opts, int* dMissingValuesCount);

__global__ void Calculate(double* dT,
							double* dP,
							double* dTP,
							tpot_cuda_options opts, int* dMissingValuesCount);

__device__ void _Calculate(const double* __restrict__ dT,
							const double* __restrict__ dP,
							double* __restrict__ dTP,
							tpot_cuda_options opts, 
							int* dMissingValuesCount,
							int idx);

} // namespace tpot
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::tpot_cuda::UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dPPacked,
									double* dT,
									double* dP,
									double* dTP,
									tpot_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (opts.simplePackedT.HasData())
		{
			SimpleUnpack(dTPacked, dT, opts.N, opts.simplePackedT.bitsPerValue, opts.simplePackedT.binaryScaleFactor, opts.simplePackedT.decimalScaleFactor, opts.simplePackedT.referenceValue, idx);
		}

		if (!opts.isConstantPressure && opts.simplePackedP.HasData())
		{
			SimpleUnpack(dPPacked, dP, opts.N, opts.simplePackedP.bitsPerValue, opts.simplePackedP.binaryScaleFactor, opts.simplePackedP.decimalScaleFactor, opts.simplePackedP.referenceValue, idx);
		}

		_Calculate(dT, dP, dTP, opts, dMissingValuesCount, idx);
	}
}

__global__ void himan::plugin::tpot_cuda::Calculate(double* dT,	double* dP, double* dTP,
							tpot_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		_Calculate(dT, dP, dTP, opts, dMissingValuesCount, idx);
	}
}

__device__ void himan::plugin::tpot_cuda::_Calculate(const double* __restrict__ dT,
														const double* __restrict__ dP,
														double* __restrict__ dTP,
														tpot_cuda_options opts,
														int* dMissingValuesCount, int idx)
{
	double P = (opts.isConstantPressure) ? opts.PConst : dP[idx];

	if (dT[idx] == kFloatMissing || P == kFloatMissing)
	{
		atomicAdd(dMissingValuesCount, 1);
		dTP[idx] = kFloatMissing;
	}
	else
	{
		dTP[idx] = (opts.TBase + dT[idx]) * powf((1000 / (opts.PScale * P)), 0.286);
	}
}

void himan::plugin::tpot_cuda::DoCuda(tpot_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	unsigned char* dTPacked;
	unsigned char* dPPacked;

	double* dT;
	double* dP;
	double* dTP;

	int *dMissingValuesCount;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	CUDA_CHECK(hipMalloc((void **) &dTP, memSize));

	if (opts.simplePackedT.HasData())
	{
		CUDA_CHECK(hipMalloc((void **) &dTPacked, opts.N * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dTPacked, opts.simplePackedT.data, opts.N * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));
	}

	if (!opts.isConstantPressure)
	{
		if (opts.simplePackedT.HasData())
		{
			CUDA_CHECK(hipMalloc((void **) &dPPacked, opts.N * sizeof(unsigned char)));
			CUDA_CHECK(hipMemcpy(dPPacked, opts.simplePackedP.data, opts.N * sizeof(unsigned char), hipMemcpyHostToDevice));
		}
		else
		{
			CUDA_CHECK(hipMalloc((void **) &dP, memSize));
			CUDA_CHECK(hipMemcpy(dP, opts.PIn, memSize, hipMemcpyHostToDevice));
		}
	}

	int src=0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.isPackedData)
	{
		UnpackAndCalculate <<< gridDim, blockDim >>> (dTPacked, dPPacked, dT, dP, dTP, opts, dMissingValuesCount);
	}
	else
	{
		Calculate <<< gridDim, blockDim >>> (dT, dP, dTP, opts, dMissingValuesCount);
	}

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.TpOut, dTP, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));
	CUDA_CHECK(hipFree(dTP));

	CUDA_CHECK(hipFree(dMissingValuesCount));

	if (opts.simplePackedT.HasData())
	{
		CUDA_CHECK(hipFree(dTPacked));
	}
	
	if (!opts.isConstantPressure)
	{
		if (opts.simplePackedP.HasData())
		{
			CUDA_CHECK(hipFree(dPPacked));
		}
		
		CUDA_CHECK(hipFree(dP));
	}


}
