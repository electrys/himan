#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifdef DEBUG
#include "timer_factory.h"
#endif

//#define CUDA_CHECK(a,msg) checkCUDAError(#a,__FILE__,__LINE__,a,msg)
//#define CUDA_STREAMS

namespace himan
{

namespace plugin
{

namespace tpot_cuda
{

void doCuda(const float* Tin, float TBase, const float* Pin, float PScale, float* TPout, size_t N, float PConst, unsigned short index);
void checkCUDAError(const std::string& msg);
__global__ void kernel_constant_pressure(float* Tin, float TBase, float P, float* TPout, size_t N);
__global__ void kernel_varying_pressure(float* Tin, float TBase, float* Pin, float PScale, float* TPout, size_t N);


} // namespace tpot
} // namespace plugin
} // namespace himan


const float kFloatMissing = 32700.f;

__global__ void himan::plugin::tpot_cuda::kernel_constant_pressure(float* Tin, float TBase, float P, float* TPout, size_t N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {

        if (Tin[idx] == kFloatMissing || P == kFloatMissing)
        {
            TPout[idx] = kFloatMissing;
        }
        else
        {
            TPout[idx] = (TBase + Tin[idx]) * powf((1000 / P), 0.286f);
        }
    }
}

__global__ void himan::plugin::tpot_cuda::kernel_varying_pressure(float* Tin, float TBase, float* Pin, float PScale, float* TPout, size_t N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {

        if (Tin[idx] == kFloatMissing || Pin[idx] == kFloatMissing)
        {
            TPout[idx] = kFloatMissing;
        }
        else
        {
            TPout[idx] = (TBase + Tin[idx]) * powf((1000 / (PScale * Pin[idx])), 0.286f);
        }
    }
}


void himan::plugin::tpot_cuda::doCuda(const float* Tin, float TBase, const float* Pin, float PScale, float* TPout, size_t N, float PConst, unsigned short index)
{

    //hipSetDevice(deviceIndex);
    //hipSetDevice(0); // this laptop has only one GPU

#ifdef CUDA_STREAMS
    hipStream_t stream;

    hipStreamCreate(&stream);

    hipError_t err;

#endif

    std::cout << "P " << PConst << std::endl;
    // Allocate host arrays and convert input data to float

    size_t size = N * sizeof(float);

    bool isConstantPressure = (Pin == 0 && PConst > 0);

    // Allocate device arrays

    float* dT;
    hipMalloc((void **) &dT, size);
    checkCUDAError("malloc dT");

    float* dP;

    if (!isConstantPressure)
    {
        hipMalloc((void **) &dP, size);
        checkCUDAError("malloc dP");
    }

#ifdef CUDA_STREAMS
    float *TPpinned;
    checkCudaErrors(hipHostMalloc((void **)&TPpinned, size));
#endif

    float *dTP;

    hipMalloc((void **) &dTP, size);
    checkCUDAError("malloc dTP");

#ifdef CUDA_STREAMS
    hipMemcpyAsync(dT, Tin, size, hipMemcpyHostToDevice, stream);
#else
    hipMemcpy(dT, Tin, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy Tin");
#endif


    if (!isConstantPressure)
    {
#ifdef CUDA_STREAMS
        hipMemcpyAsync(dP, Pin, size, hipMemcpyHostToDevice, stream);
#else
        hipMemcpy(dP, Pin, size, hipMemcpyHostToDevice);
        checkCUDAError("memcpy Pin");
#endif

    }

#ifdef CUDA_STREAMS
    hipMemcpyAsync(dTP, TPout, size, hipMemcpyHostToDevice, stream);
#else
    hipMemcpy(dTP, TPout, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy TPout");
#endif

    // dims

    const int n_threads_per_block = 512;
    int n_blocks = N/n_threads_per_block + (N%n_threads_per_block == 0?0:1);

    dim3 dimGrid(n_blocks,1);
    dim3 dimBlock(n_threads_per_block, 1);

#ifdef DEBUG
    timer* t = timer_factory::Instance()->GetTimer();
    t->Start();
#endif

    if (isConstantPressure)
    {
#ifdef CUDA_STREAMS
        kernel_constant_pressure <<< dimGrid, dimBlock, 0, stream >>> (dT, TBase, PConst, dTP, N);
#else
        kernel_constant_pressure <<< dimGrid, dimBlock >>> (dT, TBase, PConst, dTP, N);
#endif
    }
    else
    {
#ifdef CUDA_STREAMS
        kernel_varying_pressure <<< dimGrid, dimBlock, 0, stream >>> (dT, TBase, dP, PScale, dTP, N);
#else
        kernel_varying_pressure <<< dimGrid, dimBlock >>> (dT, TBase, dP, PScale, dTP, N);
#endif
    }

    // block until the device has completed

#ifdef CUDA_STREAMS
    hipMemcpyAsync(TPpinned, dTP, size, hipMemcpyDeviceToHost, stream);

    if ((err = hipStreamSynchronize(stream)) != hipSuccess)
    {
        std::cout << "Cuda error: " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
#else

    hipDeviceSynchronize();
    // check if kernel execution generated an error

#ifdef DEBUG
    t->Stop();

    std::cout << "cudaDebug::tpot_cuda Kernel execution took took " << t->GetTime() << " microseconds" << std::endl;

    delete t;
#endif

    checkCUDAError("kernel invocation");

    // Retrieve result from device
    hipMemcpy(TPout, dTP, size, hipMemcpyDeviceToHost);

    checkCUDAError("memcpy dTP");

#endif

    hipFree(dT);

    if (!isConstantPressure)
    {
        hipFree(dP);
    }

    hipFree(dTP);

#ifdef CUDA_STREAMS
    TPout = TPpinned;

    hipHostFree(TPpinned);
    hipStreamDestroy(stream);
#endif
}

void himan::plugin::tpot_cuda::checkCUDAError(const std::string& msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        std::cout << "Cuda error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}
