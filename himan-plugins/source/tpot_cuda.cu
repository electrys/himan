#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>
#include <math.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

const float kFloatMissing = 32700.f;

timespec start_ts;
timespec stop_ts;

void checkCUDAError(const std::string& msg);
void StartTimer();
void StopTimer();
long GetTime();

/*
namespace himan
{

namespace plugin
{
*/
__global__ void tpot_kernel(float* Tin, float* Pin, float* TPout, int N)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		//TPout[idx] = 273.15f + Tin[idx] * powf((1000 / Pin[idx]), 0.286f);
		
		// Assume K for now (since it actually is K when read from grib)
		
		if (Tin[idx] == kFloatMissing || Pin[idx] == kFloatMissing)
		{
		  TPout[idx] = kFloatMissing;
		}
		else
		{		
		  TPout[idx] = Tin[idx] * powf((1000 / Pin[idx]), 0.286f);
		}
	}
}


void tpot_cuda(const float* Tin, const float* Pin, float* TPout, int N)
{

	// Allocate host arrays and convert input data to float
	
	size_t size = N * sizeof(float);
	
	float* hP = (float*) malloc(size);

	size_t i = 0;
	
	for (i = 0; i < N; i++)
	{
		hP[i] = 850.f; // Hard coded pressure level 850
		
		TPout[i] = kFloatMissing;
	}
	
	// Allocate device arrays
	
	float* dT; 
	hipMalloc((void **) &dT, size);
	checkCUDAError("malloc");
	
	float* dP;
  	hipMalloc((void **) &dP, size);
  	checkCUDAError("malloc");

  	float *dTP;
  	hipMalloc((void **) &dTP, size);
  	checkCUDAError("malloc");
  
    hipMemcpy(dT, Tin, size, hipMemcpyHostToDevice);
  	checkCUDAError("memcpy");
  
  	hipMemcpy(dP, Pin, size, hipMemcpyHostToDevice);
  	checkCUDAError("memcpy");
  
  	hipMemcpy(dTP, TPout, size, hipMemcpyHostToDevice);
  	checkCUDAError("memcpy");
  	
    // dims
     	
    const int n_threads_per_block = 512;
    int n_blocks = N/n_threads_per_block + (N%n_threads_per_block == 0?0:1);  	
  	
  	std::cout << "threads_per_block: " << n_threads_per_block << " number of blocks " << n_blocks << std::endl;
  	
  	dim3 dimGrid(n_blocks);
  	dim3 dimBlock(n_threads_per_block);
    
  	StartTimer();
  
  	tpot_kernel <<< dimGrid, dimBlock >>> (dT, dT, dTP, N);
  
  	// block until the device has completed
  	hipDeviceSynchronize();

  	// check if kernel execution generated an error

  	checkCUDAError("kernel invocation");
  
  	// Retrieve result from device 
  	hipMemcpy(TPout, dTP, size, hipMemcpyDeviceToHost);
  
  	checkCUDAError("memcpy");
  
  	StopTimer();
  
  	printf ("Calculation and data transfer took %ld microseconds on GPU\n", GetTime());
  	
  	hipFree(dT);
  	hipFree(dP);
  	hipFree(dTP);
  	
}

void checkCUDAError(const std::string& msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        std::cout << "Cuda error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        exit(1);
    }                        
}

void StartTimer() {
  clock_gettime(CLOCK_REALTIME, &start_ts);
}

void StopTimer() {
  clock_gettime(CLOCK_REALTIME, &stop_ts);
}

long GetTime() {
  return ((stop_ts.tv_sec*1e9 + stop_ts.tv_nsec) - (start_ts.tv_sec*1e9 + start_ts.tv_nsec))/1e3;
}

/*
} // namespace plugin
} // namespace himan

*/