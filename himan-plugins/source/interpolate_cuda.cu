#include "hip/hip_runtime.h"
#include "info_simple.h"
#include <NFmiLatLonArea.h>
#include <NFmiRotatedLatLonArea.h>
#include <NFmiStereographicArea.h>
#include <NFmiGrid.h>
#include "cuda_plugin_helper.h"
#include <thrust/sort.h>
#include "numerical_functions.h"

const double kEpsilon = 1e-6;

struct point
{
	double x;
	double y;

	__host__ __device__
	point() : x(kFloatMissing), y(kFloatMissing) {}
	__host__ __device__
	point(double _x, double _y) : x(_x), y(_y) {}

};

__host__ __device__ 
unsigned int Index(unsigned int x, unsigned int y, unsigned int sx)
{
	return y * sx + x;
}

__host__ __device__ 
unsigned int Index(point p, unsigned int sx)
{
	return Index(static_cast<unsigned int> (p.x),static_cast<unsigned int> (p.y), sx);
}

__global__
void Swap(double* __restrict__ arr, size_t ni, size_t nj)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Flip with regards to x axis

	if (idx < nj * ni * 0.5)
	{
		const int i = fmod(static_cast<double> (idx), static_cast<double> (ni));
		const int j = floor(static_cast<double> (idx / ni));

		double upper = arr[idx];
		double lower = arr[Index(i,nj-1-j,ni)];

		arr[idx] = lower;
		arr[Index(i,nj-1-j,ni)] = upper;
	}
}

__global__ void Print(double* __restrict__ arr, int i) { printf("%d %f\n", i, arr[i]); }

 NFmiArea* CreateArea(himan::info_simple* info)
{
	NFmiPoint bl, tr;
	
	bl.X(info->first_lon);
	tr.X(bl.X() + (info->size_x - 1) * info->di);

	if (info->j_scans_positive)
	{	
		bl.Y(info->first_lat);
		tr.Y(bl.Y() + (info->size_y - 1) * info->dj);
	}
	else
	{
		tr.Y(info->first_lat);
		bl.Y(tr.Y() - (info->size_y - 1) * info->dj);
	}

	NFmiArea* area = 0;

	if (info->projection == himan::kLatLonProjection)
	{
		area = new NFmiLatLonArea(bl, tr);
	}
	else if (info->projection == himan::kRotatedLatLonProjection)
	{
		NFmiPoint sp(info->south_pole_lon, info->south_pole_lat);
		area = new NFmiRotatedLatLonArea(bl, tr, sp, NFmiPoint(0,0), NFmiPoint(1,1), true);
	}
	else if (info->projection == himan::kStereographicProjection)
	{
		area = new NFmiStereographicArea(bl, (info->size_x - 1) * info->di, (info->size_y - 1) * info->dj, info->orientation);
	}
	else
	{
		throw std::runtime_error("Invalid projection for cuda interpolation");
	}

	info->wraps_globally = area->PacificView();

	assert(area);
	return area;
}

point* CreateGrid(himan::info_simple* sourceInfo, himan::info_simple* targetInfo)
{

	NFmiArea* sourceArea = CreateArea(sourceInfo);
	NFmiArea* targetArea = CreateArea(targetInfo);
	
	NFmiGrid sourceGrid(sourceArea, sourceInfo->size_x, sourceInfo->size_y, kBottomLeft);
	NFmiGrid targetGrid(targetArea, targetInfo->size_x, targetInfo->size_y, kBottomLeft);
/*
	std::cout	<< "Source area BL: " << sourceArea->BottomLeftLatLon()
				<< "Source area TR: " << sourceArea->TopRightLatLon()
				<< "Source grid BL: " << sourceGrid.LatLonToGrid(sourceArea->BottomLeftLatLon())
				<< "Source grid TR: " << sourceGrid.LatLonToGrid(sourceArea->TopRightLatLon())
				<< "Source J scans positive: " << sourceInfo->j_scans_positive << std::endl
				<< "Target area BL: " << targetArea->BottomLeftLatLon()
				<< "Target area TR: " << targetArea->TopRightLatLon()
				<< "Target grid BL (relative): " << sourceGrid.LatLonToGrid(targetArea->BottomLeftLatLon())
				<< "Target grid TR (relative): " << sourceGrid.LatLonToGrid(targetArea->TopRightLatLon())
				<< "Target J scans positive: " << targetInfo->j_scans_positive << std::endl
				;
	*/	
	point* ret = new point[targetGrid.XNumber() * targetGrid.YNumber()];
	
	targetGrid.Reset();
	
	int i = 0;

	
	while(targetGrid.Next())
	{
		NFmiPoint gp = sourceGrid.LatLonToGrid(targetGrid.LatLon());

#ifdef EXTRADEBUG
		NFmiPoint latlon = targetGrid.LatLon();

		if (!sourceArea->IsInside(latlon))
		{
			std::cout << "Latlon " << latlon << " is outside source area!" << std::endl;
		}
		if (!sourceGrid.IsInsideGrid(gp))
		{
			std::cout << "Gridpoint " << gp << " is outside source grid!" << std::endl;
		}
		
#endif
		ret[i].x = gp.X();
		ret[i].y = gp.Y();

		i++;
	}

	delete (sourceArea);
	delete (targetArea);

	return ret;
}

__device__
double Mode(double* arr)
{
	thrust::sort(thrust::seq, arr, arr + 4);

	double num = arr[0];
	double mode = kFloatMissing;
	
	int count = 1;
	int modeCount = 0;
	
	bool multiModal = false;
	
	for (int i = 1; i < 4; i++)
	{
		double val = arr[i];
		
		if (fabs(val - num) < kEpsilon)
		{
			// increase occurrences for this number
			count++;
		
			if (count == modeCount)
			{
				multiModal = true;
			}
			else if (count > modeCount)
			{
				modeCount = count;
				mode = num;
				multiModal = false;
			}
		}
		else
		{
			// value changed
			count = 1;
			num = val;
		}
	}
	
	double ret = kFloatMissing;
	
	if (!multiModal)
	{
		ret = mode;
	}
	
	return ret;
}

__device__
bool IsInsideGrid(point& gp, size_t size_x, size_t size_y)
{
	// if interpolated grid points are negative, it means that we are outside the grid
	
	// sometime first grid point is -0, so we subtract a small value from first
	// grid point accept that value as well
	
	if (gp.x >= (0 - kEpsilon) && gp.y >= (0 - kEpsilon) &&
				
		// if interpolated grid points are larger than source grid in x or y
		// direction, it means again that we are outside of the area
			
		((fabs(gp.x - (size_x - 1)) < kEpsilon || __double2uint_ru(gp.x) < size_x) && (fabs(gp.y - (size_y - 1)) < kEpsilon || __double2uint_ru(gp.y) < size_y)))
	{
		return true;
	}

#ifdef EXTRADEBUG
	bool lc = gp.x >= (0 - kEpsilon) && gp.y >= (0 - kEpsilon);
	bool uc = (fabs(gp.x - (size_x - 1)) < kEpsilon || __double2uint_ru(gp.x) < size_x) && (fabs(gp.y - (size_y - 1)) < kEpsilon || __double2uint_ru(gp.y) < size_y);
	
	printf("gp x:%f y:%f discarded [%ld,%ld]: lower cond --> x:%d y:%d upper cond x:%d y:%d\n", 
		gp.x, gp.y, size_x, size_y, gp.x >= (0 - kEpsilon), gp.y >= (0 - kEpsilon), lc, uc);
	
#endif

	return false;
}

__device__
double NearestPointInterpolation(const double* __restrict__ d_source, himan::info_simple& sourceInfo, const point& gp)
{
	int rx = rint(gp.x);
	int ry = rint(gp.y);

	assert(rx >= 0 && rx <= sourceInfo.size_x);
	assert(ry >= 0 && ry <= sourceInfo.size_y);

	return d_source[Index(rx,ry,sourceInfo.size_x)];

}

__device__
double BiLinearInterpolation(const double* __restrict__ d_source, himan::info_simple& sourceInfo, const point& gp)
{
	double ret = kFloatMissing;

	// Find all four neighboring points

	point a(floor(gp.x), ceil(gp.y));
	point b(ceil(gp.x), ceil(gp.y));
	point c(floor(gp.x), floor(gp.y));
	point d(ceil(gp.x), floor(gp.y));
	
	// Assure neighboring points are inside grid and get values
	
	size_t size_x = sourceInfo.size_x;
	size_t size_y = sourceInfo.size_y;
	
	double av = kFloatMissing, bv = kFloatMissing, cv = kFloatMissing, dv = kFloatMissing;
	
	if (IsInsideGrid(a, size_x, size_y))
	{
		av = d_source[Index(a,size_x)];
	}
	if (IsInsideGrid(b, size_x, size_y))
	{
		bv = d_source[Index(b,size_x)];
	}
	if (IsInsideGrid(c, size_x, size_y))
	{
		cv = d_source[Index(c,size_x)];
	}
	if (IsInsideGrid(d, size_x, size_y))
	{
		dv = d_source[Index(d,size_x)];
	}

	// Distance of interpolated point to neighboring points

	point dist(gp.x - c.x, gp.y - c.y);

	assert(dist.x >= 0 && dist.x <= 1);
	assert(dist.y >= 0 && dist.y <= 1);
	
	// If interpolated point is very close to source grid point, pick 
	// the point value directly
	
	// This is preferred since nearest point is faster than bilinear, and
	// if wanted grid point =~ source grid point, the bilinear interpolation
	// value will be very close to nearest point value
	
	using namespace himan::numerical_functions::interpolation;

	if ( 
			(dist.x < kEpsilon || fabs(dist.x-1) < kEpsilon) && 
			(dist.y < kEpsilon || fabs(dist.y-1) < kEpsilon) )
	{
		ret = NearestPointInterpolation(d_source, sourceInfo, gp);
	}
	
	// All values present, regular bilinear interpolation
	
	else if (av != kFloatMissing && bv != kFloatMissing && cv != kFloatMissing && dv != kFloatMissing)
	{
		ret = BiLinear(dist.x, dist.y, av, bv, cv, dv);
	}
	
	// x or y is at grid edge

	else if (fabs(dist.y) < kEpsilon && cv != kFloatMissing && dv != kFloatMissing)
	{
		ret = Linear(dist.x, cv, dv);
	}

	else if (fabs(dist.y - 1) < kEpsilon && av != kFloatMissing && bv != kFloatMissing)
	{
		ret = Linear(dist.x, av, bv);
	}

	else if (fabs(dist.x) < kEpsilon && cv != kFloatMissing && av != kFloatMissing)
	{
		ret = Linear(dist.y, cv, av);
	}

	else if (fabs(dist.x - 1) < kEpsilon && av != kFloatMissing && bv != kFloatMissing)
	{
		ret = Linear(dist.y, dv, bv);
	}
		
	// One point missing; these "triangulation" methods have been copied from NFmiInterpolation.cpp

	else if (av == kFloatMissing && bv != kFloatMissing && cv != kFloatMissing && dv != kFloatMissing)
	{
		double wsum = (dist.x * dist.y + (1 - dist.x) * (1 - dist.y) + dist.x * (1 - dist.y));

		ret = ((1 - dist.x) * (1 - dist.y) * cv + dist.x * (1 - dist.y) * dv + dist.x * dist.y * bv) / wsum;
	}
	else if (av != kFloatMissing && bv == kFloatMissing && cv != kFloatMissing && dv != kFloatMissing)
	{
		double wsum = ((1 - dist.x) * dist.y + (1 - dist.x) * (1 - dist.y) + dist.x * (1 - dist.y));

		ret = ((1 - dist.x) * (1 - dist.y) * cv + dist.x * (1 - dist.y) * dv + (1 - dist.x) * dist.y * av) / wsum;
	}
	else if (av != kFloatMissing && bv != kFloatMissing && cv == kFloatMissing && dv != kFloatMissing)
	{
		double wsum = ((1 - dist.x) * dist.y + dist.x * dist.y + dist.x * (1 - dist.y));
		
		ret = (dist.x * (1 - dist.y) * dv + (1 - dist.x) * dist.y * av + dist.x * dist.y * bv) / wsum;
	}
	else if (av != kFloatMissing && bv != kFloatMissing && cv != kFloatMissing && dv == kFloatMissing)
	{
		double wsum = ((1 - dist.x) * (1 - dist.y) + (1 - dist.x) * dist.y + dist.x * dist.y);
		
		ret = ((1 - dist.x) * (1 - dist.y) * cv + (1 - dist.x) * dist.y * av + dist.x * dist.y * bv) / wsum;
	}

#ifdef EXTRADEBUG
	else
	{
		printf("More than one point missing for gp x: %f y:%f --> a:%f b:%f c:%f d:%f | dx:%f dy:%f\n", gp.x, gp.y, av, bv, cv, dv, dist.x, dist.y);
	}

	if (ret == kFloatMissing) { 
	
		printf("gpx:%f gpy:%f [%ld %ld] |  dist x:%f y:%f\n", gp.x, gp.y, size_x, size_y, dist.x, dist.y);
		printf("av:%f bv:%f cv:%f dv:%f | interp:%f\n", av, bv, cv, dv, ret);
		printf("ax:%f ay:%f bx:%f by:%f cx:%f cy:%f dx:%f dy:%f\n", a.x, a.y, b.x, b.y, c.x, c.y, d.x, d.y);
		printf("is inside grid: a:%d b:%d c:%d d:%d\n", IsInsideGrid(a, size_x, size_y), IsInsideGrid(b, size_x, size_y), IsInsideGrid(c, size_x, size_y), IsInsideGrid(d, size_x, size_y));
	
	}
	
#endif

	return ret;
}

__device__
double NearestPointValueInterpolation(const double* __restrict__ d_source, himan::info_simple& sourceInfo, const point& gp)
{
	double ret = kFloatMissing;

	// Find all four neighboring points

	point a(floor(gp.x), ceil(gp.y));
	point b(ceil(gp.x), ceil(gp.y));
	point c(floor(gp.x), floor(gp.y));
	point d(ceil(gp.x), floor(gp.y));

	// Assure neighboring points are inside grid
	
	size_t size_x = sourceInfo.size_x;
	size_t size_y = sourceInfo.size_y;
	
	if (!IsInsideGrid(a, size_x, size_y) || !IsInsideGrid(b, size_x, size_y) || !IsInsideGrid(c, size_x, size_y) || !IsInsideGrid(d, size_x, size_y))
	{
		return ret;
	}
	
	// Neighbor values

	double av = d_source[Index(a,size_x)];
	double bv = d_source[Index(b,size_x)];
	double cv = d_source[Index(c,size_x)];
	double dv = d_source[Index(d,size_x)];	

	// Find mode of neighboring points

	double arr[4] = {av, bv, cv, dv};
	double mode = Mode(arr);

	if (mode != kFloatMissing)
	{
		return mode;
	}

	double bilin = BiLinearInterpolation(d_source, sourceInfo, gp);

	arr[0] = fabs(av - bilin);
	arr[1] = fabs(bv - bilin);
	arr[2] = fabs(cv - bilin);
	arr[3] = fabs(dv - bilin);
	
	mode = Mode(arr);
	
	if (mode != kFloatMissing)
	{
		double min = fmin(arr[0], fmin(arr[1], fmin(arr[2], arr[3])));

		if (fabs(mode - min) < kEpsilon)
		{
			ret = bilin - mode;
		}
		else
		{
			ret = bilin - min;
		}

	}
	else
	{
		// no mode
		double min = fmin(arr[0], fmin(arr[1], fmin(arr[2], arr[3])));
		ret = bilin - min;
	}

	return ret;
}


__global__ 
void InterpolateCudaKernel(const double* __restrict__ d_source, 
							double* __restrict__ d_target,
							const point* __restrict__ d_grid,
							himan::info_simple sourceInfo, 
							himan::info_simple targetInfo)
{

	// idx is our pointer to the TARGET data in linear format

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < targetInfo.size_x * targetInfo.size_y)
	{
		// next we need to get x and y of the 'idx' in the source grid coordinates
		// to do that we first determine the i and j of the target grid coordinates

		const int i = fmod(static_cast<double> (idx), static_cast<double> (targetInfo.size_x));
		const int j = floor(static_cast<double> (idx / targetInfo.size_x));

		// with i and j we can get the grid point coordinates in the source grid
		
		point gp = d_grid[Index(i,j,targetInfo.size_x)];

		if (sourceInfo.wraps_globally && (gp.x < 0 || gp.x > sourceInfo.size_x - 1))
		{
			// wrap x if necessary
			// this might happen f.ex. with EC where grid start at 0 meridian and 
			// we interpolate from say -10 to 40 longitude

			while (gp.x < 0) gp.x += sourceInfo.size_x;
			while (gp.x > sourceInfo.size_x-1) gp.x -= sourceInfo.size_x-1;
		}
		
		double interp = kFloatMissing;
		
		if (IsInsideGrid(gp, sourceInfo.size_x, sourceInfo.size_y))
		{
			
			//targetInfo.interpolation = himan::kNearestPointValue;

			switch (targetInfo.interpolation)
			{
				case himan::kBiLinear:
					interp = BiLinearInterpolation(d_source, sourceInfo, gp);
					break;
					
				case himan::kNearestPoint:
					interp = NearestPointInterpolation(d_source, sourceInfo, gp);
					break;
					
				case himan::kNearestPointValue:
					interp = NearestPointValueInterpolation(d_source, sourceInfo, gp);
					break;
			}
		}
#ifdef EXTRADEBUG
		else
		{
			printf("grid point x:%f y:%f discarded [%ld,%ld]\n", gp.x, gp.y, sourceInfo.size_x, sourceInfo.size_y);
		}
#endif
		d_target[idx] = interp ;

		assert(interp == interp); // no NaN
		assert(interp < 1e30); // No crazy values

	}
}


bool InterpolateCuda(himan::info_simple* sourceInfo, himan::info_simple* targetInfo)
{
	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	if (targetInfo->interpolation == himan::kUnknownInterpolationMethod)
	{
		targetInfo->interpolation = himan::kBiLinear;
	}

	/* Determine all grid point coordinates that need to be interpolated.
	 * This is done with newbase by explicitly looping through the grid.
	 * Initially I tried to implement it with just starting point and offset
	 * but the code was awkward and would not work with stereographic projections
	 * anyway.
	 */

	point* grid = CreateGrid(sourceInfo, targetInfo);
	
	const size_t N = targetInfo->size_x * targetInfo->size_y;
	
	point* d_grid = 0;
	CUDA_CHECK(hipMalloc((void**) &d_grid, sizeof(point) * N));
	CUDA_CHECK(hipMemcpyAsync(d_grid, grid, sizeof(point) * N, hipMemcpyHostToDevice, stream));

	double* d_source = 0;
	double* d_target = 0;

	CUDA_CHECK(hipMalloc((void **) &d_source, sourceInfo->size_x * sourceInfo->size_y * sizeof(double)));
	CUDA_CHECK(hipMalloc((void **) &d_target, N * sizeof(double)));

#ifdef DEBUG
	CUDA_CHECK(hipMemset(d_target, 0, targetInfo->size_x * targetInfo->size_y * 8));
#endif
	
	PrepareInfo(sourceInfo, d_source, stream);
	PrepareInfo(targetInfo);

	if (!sourceInfo->j_scans_positive)
	{
		// Force +x-y --> +x+y

		// This is needed because latlon coordinates are created from newbase area
		// and they are in +x+y. This also means that we have to flip the data
		// back after interpolation.

		// TODO: Do not flip the data twice, but create the grid in the correct scanning mode!
		
		size_t N = sourceInfo->size_x * sourceInfo->size_y * 0.5 ;

		int bs = 256;
		int gs = N/bs + (N % bs == 0?0:1);
		Swap <<<gs,bs,0,stream>>>(d_source, sourceInfo->size_x, sourceInfo->size_y);

		sourceInfo->j_scans_positive = true;
	}
	
	const int bs = 256;
	const int gs = N/bs + (N % bs == 0?0:1);

	// Do bilinear transform on CUDA device
	InterpolateCudaKernel <<<gs,bs,0,stream>>>(d_source, d_target, d_grid, *sourceInfo, *targetInfo);

	if (!targetInfo->j_scans_positive)
	{
		// Flip data back

		size_t N = targetInfo->size_x * targetInfo->size_y * 0.5 ;

		int bs = 256;
		int gs = N/bs + (N % bs == 0?0:1);
		Swap <<<gs,bs,0,stream>>>(d_target, targetInfo->size_x, targetInfo->size_y);
	}

	CUDA_CHECK(hipStreamSynchronize(stream));
	
	himan::ReleaseInfo(sourceInfo);
	himan::ReleaseInfo(targetInfo, d_target, stream);
	
	CUDA_CHECK(hipFree(d_source));
	CUDA_CHECK(hipFree(d_target));
	CUDA_CHECK(hipFree(d_grid));
	
	CUDA_CHECK(hipStreamDestroy(stream));
	
	return true;
}
