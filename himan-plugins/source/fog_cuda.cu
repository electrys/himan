#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include <cuda_helper.h>
#include <fog_cuda.h>

namespace himan
{

namespace plugin
{

namespace fog_cuda
{

	__global__ void Calculate(	const double* __restrict__ dDTC2M, 
								const double* __restrict__ dTKGround, 
								const double* __restrict__ dFF10M, 
								double* __restrict__ dF, 
								fog_cuda_options opts, 
								int* dMissingValuesCount);

} // namespace fog_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::fog_cuda::Calculate( const double* __restrict__ dDTC2M, 
													const double* __restrict__ dTKGround, 
													const double* __restrict__ dFF10M, 
													double* __restrict__ dF, 
													fog_cuda_options opts, 
													int* dMissingValuesCount )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (dDTC2M[idx] == kFloatMissing || dTKGround[idx] == kFloatMissing || dFF10M[idx] == kFloatMissing )
		{
			atomicAdd(dMissingValuesCount, 1);
			dF[idx] = kFloatMissing;
		}
		else
		{
			dF[idx] = dDTC2M[idx] - dTKGround[idx] -0.3 && dFF10M[idx] < 5) ? 607 : 0;
			
		}
	}
}

void himan::plugin::fog_cuda::DoCuda(fog_cuda_options& opts, fog_cuda_data& datas)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dDTC2M;
	double* dTKGround;
	double* dFF10M;
	
	double* dF;

	unsigned char* dpDTC2M;
	unsigned char* dpTKGround;
	unsigned char* dpFF10M;

	int* dbmDTC2M;
	int* dbmTKGround;
	int* dbmFF10M;
	
	int* dMissingValuesCount;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipHostGetDevicePointer(&dF, datas.F, 0));

	if (opts.pDTC2M)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dDTC2M, datas.DTC2M, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpDTC2M, datas.pDTC2M.data, 0));

		if (datas.pDTC2M.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmT, datas.pDTC2M.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dDTC2M, memsize));
		CUDA_CHECK(hipMemcpy(dDTC2M, datas.DTC2MT, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pTKGround)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dTKGround, datas.TKGround, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpTKGround, datas.pTKGround.data, 0));

		if (datas.pTKGround.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmTKGround, datas.pTKGround.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dTKGround, memsize));
		CUDA_CHECK(hipMemcpy(dTKGround, datas.TKGround, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pFF10M)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dFF10M, datas.FF10M, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpFF10M, datas.pFF10M.data, 0));

		if (datas.pFF10M.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmFF10M, datas.pFF10M.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dFF10M, memsize));
		CUDA_CHECK(hipMemcpy(dFF10M, datas.FF10M, memsize, hipMemcpyHostToDevice));
	}

	int src = 0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));
	
	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.pDTC2M)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpDTC2M, dDTC2M, dbmDTC2M, datas.pDTC2M.coefficients, opts.N, datas.pDTC2M.HasBitmap());
	}

	if (opts.pTKGround)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpTKGround, dTKGround, dbmTKGround, datas.pTKGround.coefficients, opts.N, datas.pTKGround.HasBitmap());
	}

	if (opts.pFF10M)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpFF10M, dFF10M, dbmFF10M, datas.pFF10M.coefficients, opts.N, datas.pFF10M.HasBitmap());
	}

	Calculate <<< gridDim, blockDim >>> (dDTC2M, dTKGround, dFF10M, dF, opts, dMissingValuesCount);

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	if (!opts.pDTC2M)
	{
		CUDA_CHECK(hipFree(dDTC2M));
	}

	if (!opts.pTKGround)
	{
		CUDA_CHECK(hipFree(dTKGround));
	}

	if (!opts.pFF10M)
	{
		CUDA_CHECK(hipFree(dFF10M));
	}

	CUDA_CHECK(hipFree(dMissingValuesCount));

}