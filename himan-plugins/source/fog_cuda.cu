#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include <cuda_helper.h>
#include <fog_cuda.h>

namespace himan
{

namespace plugin
{

namespace fog_cuda
{

	__global__ void Calculate(	const double* __restrict__ dDTC2M, 
								const double* __restrict__ dTKGround, 
								const double* __restrict__ dFF10M, 
								double* __restrict__ dF, 
								fog_cuda_options opts, 
								int* dMissingValuesCount);

} // namespace fog_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::fog_cuda::Calculate( const double* __restrict__ dDTC2M, 
													const double* __restrict__ dTKGround, 
													const double* __restrict__ dFF10M, 
													double* __restrict__ dF, 
													fog_cuda_options opts, 
													int* dMissingValuesCount )
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (dDTC2M[idx] == kFloatMissing || dTKGround[idx] == kFloatMissing || dFF10M[idx] == kFloatMissing )
		{
			atomicAdd(dMissingValuesCount, 1);
			dF[idx] = kFloatMissing;
		}
		else
		{
			dF[idx] = (dDTC2M[idx] - dTKGround[idx] > -0.3 && dFF10M[idx] < 5) ? 607 : 0;
			
		}
	}
}

void himan::plugin::fog_cuda::DoCuda(fog_cuda_options& opts, fog_cuda_data& datas)
{
#if 0
	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dDTC2M;
	double* dTKGround;
	double* dFF10M;
	
	double* dF;
	
	int* dMissingValuesCount;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipHostGetDevicePointer(&dF, datas.F, 0));

	if (opts.pDTC2M)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dDTC2M, datas.DTC2M, 0));
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dDTC2M, memsize));
		CUDA_CHECK(hipMemcpy(dDTC2M, datas.DTC2M, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pTKGround)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dTKGround, datas.TKGround, 0));
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dTKGround, memsize));
		CUDA_CHECK(hipMemcpy(dTKGround, datas.TKGround, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pFF10M)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dFF10M, datas.FF10M, 0));
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dFF10M, memsize));
		CUDA_CHECK(hipMemcpy(dFF10M, datas.FF10M, memsize, hipMemcpyHostToDevice));
	}

	int src = 0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));
	
	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));
	
	if (opts.pDTC2M)
	{
		datas.pDTC2M->Unpack(dDTC2M, &stream);
	}

	if (opts.pTKGround)
	{
		datas.pTKGround->Unpack(dTKGround, &stream);
	}

	if (opts.pFF10M)
	{
		datas.pFF10M->Unpack(dFF10M, &stream);
	}

	Calculate <<< gridSize, blockSize, 0, stream >>> (dDTC2M, dTKGround, dFF10M, dF, opts, dMissingValuesCount);

	// block until the device has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	if (!opts.pDTC2M)
	{
		CUDA_CHECK(hipFree(dDTC2M));
	}

	if (!opts.pTKGround)
	{
		CUDA_CHECK(hipFree(dTKGround));
	}

	if (!opts.pFF10M)
	{
		CUDA_CHECK(hipFree(dFF10M));
	}

	CUDA_CHECK(hipFree(dMissingValuesCount));
	CUDA_CHECK(hipStreamDestroy(stream));
#endif
}
