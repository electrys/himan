#include "hip/hip_runtime.h"
#include "cuda_plugin_helper.h"
#include "dewpoint.cuh"
#include "moisture.h"

__global__ void himan::plugin::dewpoint_cuda::Calculate(cdarr_t d_t, cdarr_t d_rh, darr_t d_td, options opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		double RH = d_rh[idx] * opts.rh_scale;
		d_td[idx] = metutil::DewPointFromRH_<double>(d_t[idx] + opts.t_base, RH);
	}
}

void himan::plugin::dewpoint_cuda::Process(options& opts)
{
	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* d_t = 0;
	double* d_rh = 0;
	double* d_td = 0;

	// Allocate memory on device

	CUDA_CHECK(hipMalloc((void**)&d_t, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_rh, memsize));
	CUDA_CHECK(hipMalloc((void**)&d_td, memsize));

	himan::PrepareInfo(opts.t, d_t, stream);
	himan::PrepareInfo(opts.rh, d_rh, stream);
	himan::PrepareInfo(opts.td);

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N / blockSize + (opts.N % blockSize == 0 ? 0 : 1);

	CUDA_CHECK(hipStreamSynchronize(stream));

	Calculate<<<gridSize, blockSize, 0, stream>>>(d_t, d_rh, d_td, opts);

	CUDA_CHECK(hipStreamSynchronize(stream));
	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	himan::ReleaseInfo(opts.t);
	himan::ReleaseInfo(opts.rh);
	himan::ReleaseInfo(opts.td, d_td, stream);

	CUDA_CHECK(hipStreamSynchronize(stream));

	// Free device memory

	CUDA_CHECK(hipFree(d_t));
	CUDA_CHECK(hipFree(d_td));
	CUDA_CHECK(hipFree(d_rh));

	CUDA_CHECK(hipStreamDestroy(stream));
}
