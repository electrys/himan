#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "vvms_cuda.h"
#include "cuda_helper.h"

namespace himan
{

namespace plugin
{

namespace vvms_cuda
{

__global__ void Calculate(const double* __restrict__ dT,
							const double* __restrict__ dVV,
							const double* __restrict__ dP,
							double* __restrict__ dVVOut,
							vvms_cuda_options opts,
							int* dMissingValuesCount);

} // namespace vvms_cuda
} // namespace plugin
} // namespace himan


__global__ void himan::plugin::vvms_cuda::Calculate(const double* __restrict__ dT,
														const double* __restrict__ dVV,
														const double* __restrict__ dP,
														double* __restrict__ VVMS,
														vvms_cuda_options opts,
														int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		double P = (opts.isConstantPressure) ? opts.PConst : dP[idx];

		if (dT[idx] == kFloatMissing || dVV[idx] == kFloatMissing || P == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			VVMS[idx] = kFloatMissing;
		}
		else
		{
			VVMS[idx] = 287 * -dVV[idx] * (opts.TBase + dT[idx]) / (9.81 * P);
		}
	}
}

void himan::plugin::vvms_cuda::DoCuda(vvms_cuda_options& opts, vvms_cuda_data& datas)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT = NULL;
	double* dP = NULL;
	double* dVV = NULL;
	double* dVVMS = NULL;

	unsigned char* dpT = NULL;
	unsigned char* dpP = NULL;
	unsigned char* dpVV = NULL;
	int* dbmT = NULL;
	int* dbmP = NULL;
	int* dbmVV = NULL;

	int *dMissingValuesCount = NULL;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipHostGetDevicePointer(&dVVMS, datas.VVMS, 0));

	if (opts.pT)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dT, datas.T, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpT, datas.pT.data, 0));

		if (datas.pT.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmT, datas.pT.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dT, memsize));
		CUDA_CHECK(hipMemcpy(dT, datas.T, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pVV)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dVV, datas.VV, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpVV, datas.pVV.data, 0));

		if (datas.pVV.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmVV, datas.pVV.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dVV, memsize));
		CUDA_CHECK(hipMemcpy(dVV, datas.VV, memsize, hipMemcpyHostToDevice));
	}

	if (!opts.isConstantPressure)
	{
		if (opts.pP)
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dP, datas.P, 0));
			CUDA_CHECK(hipHostGetDevicePointer(&dpP, datas.pP.data, 0));

			if (datas.pP.HasBitmap())
			{
				CUDA_CHECK(hipHostGetDevicePointer(&dbmP, datas.pP.bitmap, 0));
			}
		}
		else
		{
			CUDA_CHECK(hipMalloc((void **) &dP, memsize));
			CUDA_CHECK(hipMemcpy(dP, datas.P, memsize, hipMemcpyHostToDevice));
		}
	}

	int src=0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.pT)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpT, dT, dbmT, datas.pT.coefficients, opts.N, datas.pT.HasBitmap());
	}

	if (opts.pVV)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpVV, dVV, dbmVV, datas.pVV.coefficients, opts.N, datas.pVV.HasBitmap());
	}

	if (opts.pP)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpP, dP, dbmP, datas.pP.coefficients, opts.N, datas.pP.HasBitmap());
	}

	Calculate <<< gridDim, blockDim >>> (dT, dVV, dP, dVVMS, opts, dMissingValuesCount);
	
	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	if (!opts.pT)
	{
		CUDA_CHECK(hipFree(dT));
	}

	if (!opts.pVV)
	{
		CUDA_CHECK(hipFree(dVV));
	}

	if (!opts.isConstantPressure && !opts.pP)
	{
		CUDA_CHECK(hipFree(dP));
	}

	CUDA_CHECK(hipFree(dMissingValuesCount));

}
