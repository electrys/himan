#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "vvms_cuda.h"

__global__ void himan::plugin::vvms_cuda::Calculate(const double* __restrict__ d_t,
														const double* __restrict__ d_vv,
														const double* __restrict__ d_p,
														double* __restrict__ d_vv_ms,
														options opts,
														int* d_missing)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		double P = (opts.is_constant_pressure) ? opts.p_const : d_p[idx];

		if (d_t[idx] == kFloatMissing || d_vv[idx] == kFloatMissing || P == kFloatMissing)
		{
			atomicAdd(d_missing, 1);
			d_vv_ms[idx] = kFloatMissing;
		}
		else
		{
			d_vv_ms[idx] = opts.vv_ms_scale * (287 * -d_vv[idx] * (opts.t_base + d_t[idx]) / (9.80665 * P * opts.p_scale));
		}
	}
}

void himan::plugin::vvms_cuda::Process(options& opts)
{

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* d_t = 0;
	double* d_p = 0;
	double* d_vv = 0;
	double* d_vv_ms = 0;

	int* d_missing = 0;

	CUDA_CHECK(hipMalloc((void **) &d_missing, sizeof(int)));
	CUDA_CHECK(hipMalloc((void **) &d_vv_ms, memsize));

	if (opts.t->packed_values)
	{
		d_t = opts.t->packed_values->Unpack(&stream);
		CUDA_CHECK(hipMemcpyAsync(opts.t->values, d_t, memsize, hipMemcpyDeviceToHost, stream));
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &d_t, memsize));
		CUDA_CHECK(hipMemcpyAsync(d_t, opts.t->values, memsize, hipMemcpyHostToDevice, stream));
	}

	if (opts.vv->packed_values)
	{
		d_vv = opts.vv->packed_values->Unpack(&stream);
		CUDA_CHECK(hipMemcpyAsync(opts.vv->values, d_vv, memsize, hipMemcpyDeviceToHost, stream));
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &d_vv, memsize));
		CUDA_CHECK(hipMemcpyAsync(d_vv, opts.vv->values, memsize, hipMemcpyHostToDevice, stream));
	}

	if (!opts.is_constant_pressure)
	{
		if (opts.p->packed_values)
		{
			d_p = opts.p->packed_values->Unpack(&stream);
			CUDA_CHECK(hipMemcpyAsync(opts.p->values, d_p, memsize, hipMemcpyDeviceToHost, stream));
		}
		else
		{
			CUDA_CHECK(hipMalloc((void **) &d_p, memsize));
			CUDA_CHECK(hipMemcpyAsync(d_p, opts.p->values, memsize, hipMemcpyHostToDevice, stream));
		}
	}

	int src=0;

	CUDA_CHECK(hipMemcpyAsync(d_missing, &src, sizeof(int), hipMemcpyHostToDevice, stream));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	CUDA_CHECK(hipStreamSynchronize(stream));

	Calculate <<< gridSize, blockSize, 0, stream >>> (d_t, d_vv, d_p, d_vv_ms, opts, d_missing);
	
	// block until the device has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpyAsync(&opts.missing, d_missing, sizeof(int), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK(hipMemcpyAsync(opts.vv_ms->values, d_vv_ms, memsize, hipMemcpyDeviceToHost, stream));

	CUDA_CHECK(hipStreamSynchronize(stream));
	
	CUDA_CHECK(hipFree(d_t));
	CUDA_CHECK(hipFree(d_vv));
	CUDA_CHECK(hipFree(d_vv_ms));

	if (d_p)
	{
		CUDA_CHECK(hipFree(d_p));
	}

	CUDA_CHECK(hipFree(d_missing));
	CUDA_CHECK(hipStreamDestroy(stream));
}
