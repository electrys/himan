#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "vvms_cuda.h"
#include "cuda_helper.h"

namespace himan
{

namespace plugin
{

namespace vvms_cuda
{

__global__ void UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dVVPacked,
									const unsigned char* dPPacked,
									double*  dT,
									double* dVV,
									double* dP,
									double* dVVOut,
									vvms_cuda_options opts,
									int* dMissingValuesCount);

__global__ void Calculate(const double* dT,
							const double* dVV,
							const double* dP,
							double* dVVOut,
							vvms_cuda_options opts,
							int* dMissingValuesCount);

__device__ void _Calculate(const double* __restrict__ dT,
							const double* __restrict__ dVV,
							const double* __restrict__ dP,
							double* __restrict__ dVVOut,
							vvms_cuda_options opts,
							int* dMissingValuesCount,
							int idx);

} // namespace vvms_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::vvms_cuda::UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dVVPacked,
									const unsigned char* dPPacked,
									double* dT,
									double* dVV,
									double* dP,
									double* dVVOut,
									vvms_cuda_options opts,
									int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (idx < opts.N)
	{
		if (opts.simplePackedT.HasData())
		{
			SimpleUnpack(dTPacked, dT, opts.N, opts.simplePackedT.bitsPerValue, opts.simplePackedT.binaryScaleFactor, opts.simplePackedT.decimalScaleFactor, opts.simplePackedT.referenceValue, idx);
		}

		if (opts.simplePackedVV.HasData())
		{
			SimpleUnpack(dVVPacked, dVV, opts.N, opts.simplePackedVV.bitsPerValue, opts.simplePackedVV.binaryScaleFactor, opts.simplePackedVV.decimalScaleFactor, opts.simplePackedVV.referenceValue, idx);
		}

		if (!opts.isConstantPressure && opts.simplePackedP.HasData())
		{
			SimpleUnpack(dPPacked, dP, opts.N, opts.simplePackedP.bitsPerValue, opts.simplePackedP.binaryScaleFactor, opts.simplePackedP.decimalScaleFactor, opts.simplePackedP.referenceValue, idx);
		}

		_Calculate(dT, dVV, dP, dVVOut, opts, dMissingValuesCount, idx);
	}
}

__device__ void himan::plugin::vvms_cuda::_Calculate(const double* __restrict__ dT,
														const double* __restrict__ dVV,
														const double* __restrict__ dP,
														double* __restrict__ VVOut,
														vvms_cuda_options opts, int* dMissingValuesCount, int idx)
{

	double P = (opts.isConstantPressure) ? opts.PConst : dP[idx];

	if (dT[idx] == kFloatMissing || dVV[idx] == kFloatMissing || P == kFloatMissing)
	{
		atomicAdd(dMissingValuesCount, 1);
		VVOut[idx] = kFloatMissing;
	}
	else
	{
		VVOut[idx] = 287 * -dVV[idx] * (opts.TBase + dT[idx]) / (9.81 * P);
	}
}

__global__ void himan::plugin::vvms_cuda::Calculate(const double* dT, 
														const double* dVV,
														const double* dP,
														double* dVVOut,
														vvms_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		_Calculate(dT, dVV, dP, dVVOut, opts, dMissingValuesCount, idx);
	}
	
}

void himan::plugin::vvms_cuda::DoCuda(vvms_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT = NULL;
	double* dP = NULL;
	double* dVV = NULL;
	double* dVVOut = NULL;

	unsigned char* dTPacked = NULL;
	unsigned char* dPPacked = NULL;
	unsigned char* dVVPacked = NULL;

	int *dMissingValuesCount = NULL;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));
	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	CUDA_CHECK(hipMalloc((void **) &dVV, memSize));
	CUDA_CHECK(hipMalloc((void **) &dVVOut, memSize));

	if (opts.simplePackedT.HasData())
	{
		CUDA_CHECK(hipMalloc((void **) &dTPacked, opts.simplePackedT.dataLength * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dTPacked, opts.simplePackedT.data, opts.simplePackedT.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));
	}

	if (opts.simplePackedVV.HasData())
	{
		CUDA_CHECK(hipMalloc((void **) &dVVPacked, opts.simplePackedVV.dataLength * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dVVPacked, opts.simplePackedVV.data, opts.simplePackedVV.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dVV, opts.VVIn, memSize, hipMemcpyHostToDevice));
	}

	if (!opts.isConstantPressure)
	{
		CUDA_CHECK(hipMalloc((void **) &dP, memSize));

		if (opts.simplePackedP.HasData())
		{
			CUDA_CHECK(hipMalloc((void **) &dPPacked, opts.simplePackedP.dataLength * sizeof(unsigned char)));
			CUDA_CHECK(hipMemcpy(dPPacked, opts.simplePackedP.data, opts.simplePackedP.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
		}
		else
		{
			CUDA_CHECK(hipMemcpy(dP, opts.PIn, memSize, hipMemcpyHostToDevice));
		}
	}

	int src=0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.isPackedData)
	{
		UnpackAndCalculate <<< gridDim, blockDim >>> (dTPacked, dVVPacked, dPPacked, dT, dVV, dP, dVVOut, opts, dMissingValuesCount);
	}
	else
	{
		Calculate <<< gridDim, blockDim >>> (dT, dVV, dP, dVVOut, opts, dMissingValuesCount);
	}

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.VVOut, dVVOut, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));
	CUDA_CHECK(hipFree(dVV));
	CUDA_CHECK(hipFree(dVVOut));

	CUDA_CHECK(hipFree(dMissingValuesCount));

	if (opts.simplePackedT.HasData())
	{
		CUDA_CHECK(hipFree(dTPacked));
	}

	if (opts.simplePackedVV.HasData())
	{
		CUDA_CHECK(hipFree(dVVPacked));
	}

	if (!opts.isConstantPressure)
	{
		if (opts.simplePackedP.HasData())
		{
			CUDA_CHECK(hipFree(dPPacked));
		}

		CUDA_CHECK(hipFree(dP));

	}

}
