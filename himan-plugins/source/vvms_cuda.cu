#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "vvms_cuda.h"
#include "cuda_helper.h"

namespace himan
{

namespace plugin
{

namespace vvms_cuda
{

__global__ void Calculate(const double* __restrict__ dT,
							const double* __restrict__ dVV,
							const double* __restrict__ dP,
							double* __restrict__ dVVOut,
							vvms_cuda_options opts,
							int* dMissingValuesCount);

} // namespace vvms_cuda
} // namespace plugin
} // namespace himan


__global__ void himan::plugin::vvms_cuda::Calculate(const double* __restrict__ dT,
														const double* __restrict__ dVV,
														const double* __restrict__ dP,
														double* __restrict__ VVMS,
														vvms_cuda_options opts,
														int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		double P = (opts.isConstantPressure) ? opts.PConst : dP[idx];

		if (dT[idx] == kFloatMissing || dVV[idx] == kFloatMissing || P == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			VVMS[idx] = kFloatMissing;
		}
		else
		{
			VVMS[idx] = 287 * -dVV[idx] * (opts.TBase + dT[idx]) / (9.81 * P);
		}
	}
}

void himan::plugin::vvms_cuda::DoCuda(vvms_cuda_options& opts, vvms_cuda_data& datas)
{

	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT = NULL;
	double* dP = NULL;
	double* dVV = NULL;
	double* dVVMS = NULL;

	int *dMissingValuesCount = NULL;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipHostGetDevicePointer(&dVVMS, datas.VVMS, 0));

	if (opts.pT)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dT, datas.T, 0));

	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dT, memsize));
		CUDA_CHECK(hipMemcpy(dT, datas.T, memsize, hipMemcpyHostToDevice));
	}

	if (opts.pVV)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dVV, datas.VV, 0));

	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dVV, memsize));
		CUDA_CHECK(hipMemcpy(dVV, datas.VV, memsize, hipMemcpyHostToDevice));
	}

	if (!opts.isConstantPressure)
	{
		if (opts.pP)
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dP, datas.P, 0));

		}
		else
		{
			CUDA_CHECK(hipMalloc((void **) &dP, memsize));
			CUDA_CHECK(hipMemcpy(dP, datas.P, memsize, hipMemcpyHostToDevice));
		}
	}

	int src=0;

	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	hipStream_t stream;

	CUDA_CHECK(hipStreamCreate(&stream));

	if (opts.pT)
	{
		datas.pT->Unpack(dT, &stream);
	}

	if (opts.pVV)
	{
		datas.pVV->Unpack(dVV, &stream);
	}

	if (opts.pP)
	{
		datas.pP->Unpack(dP, &stream);
	}

	Calculate <<< gridSize, blockSize, 0, stream >>> (dT, dVV, dP, dVVMS, opts, dMissingValuesCount);
	
	// block until the device has completed
	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	if (!opts.pT)
	{
		CUDA_CHECK(hipFree(dT));
	}

	if (!opts.pVV)
	{
		CUDA_CHECK(hipFree(dVV));
	}

	if (!opts.isConstantPressure && !opts.pP)
	{
		CUDA_CHECK(hipFree(dP));
	}

	CUDA_CHECK(hipFree(dMissingValuesCount));
	CUDA_CHECK(hipStreamDestroy(stream));
}
