#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

#ifdef DEBUG
#include "timer_factory.h"
#endif

namespace himan
{

namespace plugin
{

namespace vvms_cuda
{

void doCuda(const float* Tin, float TBase, const float* Pin, float PScale, const float* VVin, float* VVout, size_t N, float PConst, unsigned short deviceIndex);
void checkCUDAError(const std::string& msg);
__global__ void kernel_constant_pressure(float* Tin, float TBase, float P, float* VVin, float* VVout, size_t N);
__global__ void kernel_varying_pressure(float* Tin, float TBase, float* Pin, float PScale, float* VVin, float* VVout, size_t N);


} // namespace tpot
} // namespace plugin
} // namespace himan


const float kFloatMissing = 32700.f;

__global__ void himan::plugin::vvms_cuda::kernel_constant_pressure(float* Tin, float TBase, float P, float* VVin, float* VVout, size_t N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {

        if (Tin[idx] == kFloatMissing || P == kFloatMissing || VVin[idx] == kFloatMissing)
        {
            VVout[idx] = kFloatMissing;
        }
        else
        {
            //double VVms = 287 * -VV * (T + TBase) / (9.81 * (P * PScale));

            VVout[idx] = 287.f * -VVin[idx] * (TBase + Tin[idx]) / (9.81f * P);
        }
    }
}

__global__ void himan::plugin::vvms_cuda::kernel_varying_pressure(float* Tin, float TBase, float* Pin, float PScale, float* VVin, float* VVout, size_t N)
{

    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N)
    {

        if (Tin[idx] == kFloatMissing || Pin[idx] == kFloatMissing || VVin[idx] == kFloatMissing)
        {
            VVout[idx] = kFloatMissing;
        }
        else
        {
            //double VVms = 287 * -VV * (T + TBase) / (9.81 * (P * PScale));

            VVout[idx] = 287.f * -VVin[idx] * (TBase + Tin[idx]) / (9.81f * Pin[idx] * PScale);
        }
    }
}


void himan::plugin::vvms_cuda::doCuda(const float* Tin, float TBase, const float* Pin, float PScale, const float* VVin, float* VVout, size_t N, float PConst, unsigned short deviceIndex)
{

    //hipSetDevice(deviceIndex);
    hipSetDevice(0); // this laptop has only one GPU

    // Allocate host arrays and convert input data to float

    size_t size = N * sizeof(float);

    bool isConstantPressure = (Pin == 0 && PConst > 0);

    // Allocate device arrays

    float* dT;
    hipMalloc((void **) &dT, size);
    checkCUDAError("malloc dT");

    float* dP;

    if (!isConstantPressure)
    {
        hipMalloc((void **) &dP, size);
        checkCUDAError("malloc dP");
    }

    float *dVVin;

    hipMalloc((void **) &dVVin, size);
    checkCUDAError("malloc dVVin");

    float *dVVout;

    hipMalloc((void **) &dVVout, size);
    checkCUDAError("malloc dVVout");

    hipMemcpy(dT, Tin, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy Tin");

    if (!isConstantPressure)
    {
        hipMemcpy(dP, Pin, size, hipMemcpyHostToDevice);
        checkCUDAError("memcpy Pin");
    }

    hipMemcpy(dVVin, VVin, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy VVin");

    hipMemcpy(dVVout, VVout, size, hipMemcpyHostToDevice);
    checkCUDAError("memcpy VVout");

    // dims

    const int n_threads_per_block = 512;
    int n_blocks = N/n_threads_per_block + (N%n_threads_per_block == 0?0:1);

    dim3 dimGrid(n_blocks);
    dim3 dimBlock(n_threads_per_block);

#ifdef DEBUG
    timer* t = timer_factory::Instance()->GetTimer();
    t->Start();
#endif

    if (isConstantPressure)
    {
        kernel_constant_pressure <<< dimGrid, dimBlock >>> (dT, TBase, PConst, dVVin, dVVout, N);
    }
    else
    {
        kernel_varying_pressure <<< dimGrid, dimBlock >>> (dT, TBase, dP, PScale, dVVin, dVVout, N);
    }

    // block until the device has completed
    hipDeviceSynchronize();

    // check if kernel execution generated an error

    checkCUDAError("kernel invocation");

    // Retrieve result from device
    hipMemcpy(VVout, dVVout, size, hipMemcpyDeviceToHost);

    checkCUDAError("memcpy");

#ifdef DEBUG
    t->Stop();

    std::cout << "cudaDebug::tpot_cuda Calculation and data transfer took " << t->GetTime() << " microseconds on GPU" << std::endl;

    delete t;
#endif

    hipFree(dT);

    if (!isConstantPressure)
    {
        hipFree(dP);
    }
    hipFree(dVVin);
    hipFree(dVVout);

}

void himan::plugin::vvms_cuda::checkCUDAError(const std::string& msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        std::cout << "Cuda error (" << msg << "): " << hipGetErrorString(err) << std::endl;
        exit(1);
    }
}
