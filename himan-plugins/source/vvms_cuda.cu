#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "vvms_cuda.h"
#include "cuda_helper.h"

namespace himan
{

namespace plugin
{

namespace vvms_cuda
{

__global__ void UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dVVPacked,
									const unsigned char* dPPacked,
									double*  dT,
									double* dVV,
									double* dP,
									double* dVVOut,
									vvms_cuda_options opts,
									int* dMissingValuesCount);

__global__ void Calculate(const double* dT,
							const double* dVV,
							const double* dP,
							double* dVVOut,
							vvms_cuda_options opts,
							int* dMissingValuesCount);

__device__ void _Calculate(const double* __restrict__ dT,
							const double* __restrict__ dVV,
							const double* __restrict__ dP,
							double* __restrict__ dVVOut,
							vvms_cuda_options opts,
							int* dMissingValuesCount,
							int idx);

} // namespace vvms_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::vvms_cuda::UnpackAndCalculate(const unsigned char* dTPacked,
									const unsigned char* dVVPacked,
									const unsigned char* dPPacked,
									double* dT,
									double* dVV,
									double* dP,
									double* dVVOut,
									vvms_cuda_options opts,
									int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		SimpleUnpack(dTPacked, dT, opts.N, opts.simplePackedT.bitsPerValue, opts.simplePackedT.binaryScaleFactor, opts.simplePackedT.decimalScaleFactor, opts.simplePackedT.referenceValue, idx);
		SimpleUnpack(dVVPacked, dVV, opts.N, opts.simplePackedVV.bitsPerValue, opts.simplePackedVV.binaryScaleFactor, opts.simplePackedVV.decimalScaleFactor, opts.simplePackedVV.referenceValue, idx);
	
		if (!opts.isConstantPressure)
		{
			SimpleUnpackUnevenBytes(dPPacked, dP, opts.N, opts.simplePackedP.bitsPerValue, opts.simplePackedP.binaryScaleFactor, opts.simplePackedP.decimalScaleFactor, opts.simplePackedP.referenceValue, idx);
		}

		_Calculate(dT, dVV, dP, dVVOut, opts, dMissingValuesCount, idx);
	
	}
}

__device__ void himan::plugin::vvms_cuda::_Calculate(const double* __restrict__ dT,
														const double* __restrict__ dVV,
														const double* __restrict__ dP,
														double* __restrict__ VVOut,
														vvms_cuda_options opts, int* dMissingValuesCount, int idx)
{

	double P = (opts.isConstantPressure) ? opts.PConst : dP[idx];

	if (dT[idx] == kFloatMissing || dVV[idx] == kFloatMissing || P == kFloatMissing)
	{
		atomicAdd(dMissingValuesCount, 1);
		VVOut[idx] = kFloatMissing;
	}
	else
	{
		VVOut[idx] = 287 * -dVV[idx] * (opts.TBase + dT[idx]) / (9.81 * P);
	}
}

__global__ void himan::plugin::vvms_cuda::Calculate(const double* dT, 
														const double* dVV,
														const double* dP,
														double* dVVOut,
														vvms_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		_Calculate(dT, dVV, dP, dVVOut, opts, dMissingValuesCount, idx);
	}
	
}

void himan::plugin::vvms_cuda::DoCuda(vvms_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT;
	double* dP;
	double *dVV;
	double *dVVOut;

	unsigned char* dTPacked;
	unsigned char* dPPacked;
	unsigned char* dVVPacked;

	int *dMissingValuesCount;

	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	
	if (!opts.isConstantPressure)
	{
		CUDA_CHECK(hipMalloc((void **) &dP, memSize));
	}

	CUDA_CHECK(hipMalloc((void **) &dVV, memSize));
	CUDA_CHECK(hipMalloc((void **) &dVVOut, memSize));

	if (opts.isPackedData)
	{
		CUDA_CHECK(hipMalloc((void **) &dTPacked, opts.simplePackedT.dataLength * sizeof(unsigned char)));
		CUDA_CHECK(hipMalloc((void **) &dVVPacked, opts.simplePackedVV.dataLength * sizeof(unsigned char)));

		if (!opts.isConstantPressure)
		{
			CUDA_CHECK(hipMalloc((void **) &dPPacked, opts.simplePackedP.dataLength * sizeof(unsigned char)));
			CUDA_CHECK(hipMemcpy(dPPacked, opts.simplePackedP.data, opts.simplePackedP.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
		}

		CUDA_CHECK(hipMemcpy(dTPacked, opts.simplePackedT.data, opts.simplePackedT.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));
		CUDA_CHECK(hipMemcpy(dVVPacked, opts.simplePackedVV.data, opts.simplePackedVV.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));

	}
	else
	{

		CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));

		if (!opts.isConstantPressure)
		{
			CUDA_CHECK(hipMemcpy(dP, opts.PIn, memSize, hipMemcpyHostToDevice));
		}

		CUDA_CHECK(hipMemcpy(dVV, opts.VVIn, memSize, hipMemcpyHostToDevice));
	}

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.isPackedData)
	{
		UnpackAndCalculate <<< gridDim, blockDim >>> (dTPacked, dVVPacked, dPPacked, dT, dVV, dP, dVVOut, opts, dMissingValuesCount);
	}
	else
	{
		Calculate <<< gridDim, blockDim >>> (dT, dVV, dP, dVVOut, opts, dMissingValuesCount);
	}
	
	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.VVOut, dVVOut, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));

	if (!opts.isConstantPressure)
	{
		CUDA_CHECK(hipFree(dP));
	}

	CUDA_CHECK(hipFree(dVV));
	CUDA_CHECK(hipFree(dVVOut));
	CUDA_CHECK(hipFree(dMissingValuesCount));

	if (opts.isPackedData)
	{
		CUDA_CHECK(hipFree(dVVPacked));
		CUDA_CHECK(hipFree(dTPacked));

		if (!opts.isConstantPressure)
		{
			CUDA_CHECK(hipFree(dPPacked));
		}
	}
}
