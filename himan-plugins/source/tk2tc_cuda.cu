#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "tk2tc_cuda.h"

namespace himan
{

namespace plugin
{

namespace tk2tc_cuda
{

__global__ void UnpackAndCalculate(const unsigned char* dTPacked, double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount);
__global__ void Calculate(const double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount);

__device__ void _Calculate(const double* __restrict__ dT, double* __restrict__ dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount);

} // namespace tk2tc_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::tk2tc_cuda::UnpackAndCalculate(const unsigned char* dTPacked, double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (opts.simplePackedT.bitsPerValue%8)
		{
			SimpleUnpackUnevenBytes(dTPacked, dT, opts.N, opts.simplePackedT.bitsPerValue, opts.simplePackedT.binaryScaleFactor, opts.simplePackedT.decimalScaleFactor, opts.simplePackedT.referenceValue);
		}
		else
		{
			SimpleUnpackFullBytes(dTPacked, dT, opts.N, opts.simplePackedT.bitsPerValue, opts.simplePackedT.binaryScaleFactor, opts.simplePackedT.decimalScaleFactor, opts.simplePackedT.referenceValue);
		}
		
		_Calculate(dT, dTOut, opts, dMissingValuesCount);
	}
}

__global__ void himan::plugin::tk2tc_cuda::Calculate(const double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount)
{
	_Calculate(dT, dTOut, opts, dMissingValuesCount);
}

__device__ void himan::plugin::tk2tc_cuda::_Calculate(const double* __restrict__ dT, double* __restrict__ dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (dT[idx] == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			dTOut[idx] = kFloatMissing;
		}
		else
		{
			dTOut[idx] = dT[idx] - 273.15;
		}
	}
}

void himan::plugin::tk2tc_cuda::DoCuda(tk2tc_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT;
	unsigned char* dTPacked;
	double *dTOut;

	int *dMissingValuesCount;
	
	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	CUDA_CHECK(hipMalloc((void **) &dTOut, memSize));
	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	if (opts.isPackedData)
	{
		CUDA_CHECK(hipMalloc((void **) &dTPacked, opts.simplePackedT.dataLength * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dTPacked, opts.simplePackedT.data, opts.simplePackedT.dataLength * sizeof(unsigned char), hipMemcpyHostToDevice));

		//opts.simplePackedT.Clear();
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));
	}

	int src = 0;
	
	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.isPackedData)
	{
		UnpackAndCalculate <<< gridDim, blockDim >>> (dTPacked, dT, dTOut, opts, dMissingValuesCount);
	}
	else
	{
		Calculate <<< gridDim, blockDim >>> (dT, dTOut, opts, dMissingValuesCount);
	}

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.TOut, dTOut, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));
	CUDA_CHECK(hipFree(dTOut));
	CUDA_CHECK(hipFree(dMissingValuesCount));

	if (opts.isPackedData)
	{
		CUDA_CHECK(hipFree(dTPacked));
	}


}
