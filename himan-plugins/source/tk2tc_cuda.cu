#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "tk2tc_cuda.h"

namespace himan
{

namespace plugin
{

namespace tk2tc_cuda
{

__global__ void Calculate(const double* __restrict__ dTK, double* __restrict__ dTC, tk2tc_cuda_options opts, int* dMissingValuesCount);

} // namespace tk2tc_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::tk2tc_cuda::Calculate(const double* __restrict__ dTK,
														double* __restrict__ dTC,
														tk2tc_cuda_options opts,
														int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (dTK[idx] == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			dTC[idx] = kFloatMissing;
		}
		else
		{
			dTC[idx] = dTK[idx] - 273.15;
		}
	}
}

void himan::plugin::tk2tc_cuda::DoCuda(tk2tc_cuda_options& opts, tk2tc_cuda_data& datas)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));
	
	size_t memsize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dTK;
	unsigned char* dpTK;
	int* dbmTK;
	double *dTC;

	int *dMissingValuesCount;
	
	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	CUDA_CHECK(hipHostGetDevicePointer(&dTC, datas.TC, 0));

	if (opts.pTK)
	{
		CUDA_CHECK(hipHostGetDevicePointer(&dTK, datas.TK, 0));
		CUDA_CHECK(hipHostGetDevicePointer(&dpTK, datas.pTK.data, 0));

		if (datas.pTK.HasBitmap())
		{
			CUDA_CHECK(hipHostGetDevicePointer(&dbmTK, datas.pTK.bitmap, 0));
		}
	}
	else
	{
		CUDA_CHECK(hipMalloc((void **) &dTK, memsize));
		CUDA_CHECK(hipMemcpy(dTK, datas.TK, memsize, hipMemcpyHostToDevice));
	}

	int src = 0;
	
	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.pTK)
	{
		SimpleUnpack <<< gridDim, blockDim >>> (dpTK, dTK, dbmTK, datas.pTK.coefficients, opts.N, datas.pTK.HasBitmap());
	}

	Calculate <<< gridDim, blockDim >>> (dTK, dTC, opts, dMissingValuesCount);

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dMissingValuesCount));

	if (!opts.pTK)
	{
		CUDA_CHECK(hipFree(dTK));
	}

}
