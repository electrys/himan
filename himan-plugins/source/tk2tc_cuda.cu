#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "tk2tc_cuda.h"

namespace himan
{

namespace plugin
{

namespace tk2tc_cuda
{

__global__ void UnpackAndCalculate(const unsigned char* dTPacked, double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount);
__global__ void Calculate(const double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount);

__device__ void _Calculate(const double* __restrict__ dT, double* __restrict__ dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount);

__device__ void SimpleUnpackUnevenBytes(const unsigned char* __restrict__ d_p,
											double* __restrict__ d_u,
											size_t values_len, int bpv, double bsf, double dsf, double rv);
} // namespace tk2tc_cuda
} // namespace plugin
} // namespace himan

inline __device__ void himan::plugin::tk2tc_cuda::SimpleUnpackUnevenBytes(const unsigned char* __restrict__ d_p,
											double* __restrict__ d_u,
											size_t values_len, int bpv, double bsf, double dsf, double rv)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	int j=0;
	unsigned long lvalue;
	long bitp=bpv*idx;

	if (idx < values_len)
	{
		lvalue=0;

		for(j=0; j< bpv;j++)
		{
			lvalue <<= 1;
			int val;

			GetBitValue(d_p, bitp, &val);

			if (val) lvalue += 1;

			bitp += 1;
		}

		d_u[idx] = ((lvalue*bsf)+rv)*dsf;
	}

}

__global__ void himan::plugin::tk2tc_cuda::UnpackAndCalculate(const unsigned char* dTPacked, double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (opts.simple_packing.bitsPerValue%8)
		{
			himan::plugin::tk2tc_cuda::SimpleUnpackUnevenBytes(dTPacked, dT, opts.N, opts.simple_packing.bitsPerValue, opts.simple_packing.binaryScaleFactor, opts.simple_packing.decimalScaleFactor, opts.simple_packing.referenceValue);
		}
		else
		{
			SimpleUnpackFullBytes(dTPacked, dT, opts.N, opts.simple_packing.bitsPerValue, opts.simple_packing.binaryScaleFactor, opts.simple_packing.decimalScaleFactor, opts.simple_packing.referenceValue);
		}
		
		_Calculate(dT, dTOut, opts, dMissingValuesCount);
	}
}

__global__ void himan::plugin::tk2tc_cuda::Calculate(const double* dT, double* dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount)
{
	_Calculate(dT, dTOut, opts, dMissingValuesCount);
}

__device__ void himan::plugin::tk2tc_cuda::_Calculate(const double* __restrict__ dT, double* __restrict__ dTOut, tk2tc_cuda_options opts, int* dMissingValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.N)
	{
		if (dT[idx] == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			dTOut[idx] = kFloatMissing;
		}
		else
		{
			dTOut[idx] = dT[idx] - 273.15;
		}
	}
}

void himan::plugin::tk2tc_cuda::DoCuda(tk2tc_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	// Allocate host arrays and convert input data to double

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT;
	unsigned char* dTPacked;
	double *dTOut;

	int *dMissingValuesCount;
	
	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	CUDA_CHECK(hipMalloc((void **) &dTOut, memSize));
	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));

	if (opts.isPackedData)
	{
		CUDA_CHECK(hipMalloc((void **) &dTPacked, opts.simple_packing.N * sizeof(unsigned char)));
		CUDA_CHECK(hipMemcpy(dTPacked, opts.TInPacked, opts.simple_packing.N * sizeof(unsigned char), hipMemcpyHostToDevice));
	}
	else
	{
		CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));
	}
	
	int src = 0;
	
	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	if (opts.isPackedData)
	{
		UnpackAndCalculate <<< gridDim, blockDim >>> (dTPacked, dT, dTOut, opts, dMissingValuesCount);
	}
	else
	{
		Calculate <<< gridDim, blockDim >>> (dT, dTOut, opts, dMissingValuesCount);
	}

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.TOut, dTOut, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));
	CUDA_CHECK(hipFree(dTOut));
	CUDA_CHECK(hipFree(dMissingValuesCount));

}
