#include "hip/hip_runtime.h"
// System includes
#include <iostream>
#include <string>

// CUDA runtime
#include <hip/hip_runtime.h>

#include "cuda_helper.h"
#include "tk2tc_cuda.h"

namespace himan
{

namespace plugin
{

namespace tk2tc_cuda
{

__global__ void KernelTk2Tc(const double* __restrict__ dT, double* __restrict__ dTOut, size_t N, int* dMissingValuesCount, int* dTotalValuesCount);


} // namespace tk2tc_cuda
} // namespace plugin
} // namespace himan

__global__ void himan::plugin::tk2tc_cuda::KernelTk2Tc(const double* __restrict__ dT, double* __restrict__ dTOut, size_t N, int* dMissingValuesCount, int* dTotalValuesCount)
{

	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		atomicAdd(dTotalValuesCount, 1);
		
		if (dT[idx] == kFloatMissing)
		{
			atomicAdd(dMissingValuesCount, 1);
			dTOut[idx] = kFloatMissing;
		}
		else
		{
			dTOut[idx] = dT[idx] - 273.15;
		}
	}
}

void himan::plugin::tk2tc_cuda::DoCuda(tk2tc_cuda_options& opts)
{

	CUDA_CHECK(hipSetDevice(opts.cudaDeviceIndex));

	// Allocate host arrays and convert input data to double

	size_t memSize = opts.N * sizeof(double);

	// Allocate device arrays

	double* dT;
	double *dTOut;	
	int *dMissingValuesCount;
	int *dTotalValuesCount;
	
	CUDA_CHECK(hipMalloc((void **) &dT, memSize));
	CUDA_CHECK(hipMalloc((void **) &dTOut, memSize));
	CUDA_CHECK(hipMalloc((void **) &dMissingValuesCount, sizeof(int)));
	CUDA_CHECK(hipMalloc((void **) &dTotalValuesCount, sizeof(int)));

	int src = 0;
	
	CUDA_CHECK(hipMemcpy(dT, opts.TIn, memSize, hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dMissingValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));
	CUDA_CHECK(hipMemcpy(dTotalValuesCount, &src, sizeof(int), hipMemcpyHostToDevice));

	// dims

	const int blockSize = 512;
	const int gridSize = opts.N/blockSize + (opts.N%blockSize == 0?0:1);

	dim3 gridDim(gridSize);
	dim3 blockDim(blockSize);

	KernelTk2Tc <<< gridDim, blockDim >>> (dT, dTOut, opts.N, dMissingValuesCount, dTotalValuesCount);

	// block until the device has completed
	CUDA_CHECK(hipDeviceSynchronize());

	// check if kernel execution generated an error

	CUDA_CHECK_ERROR_MSG("Kernel invocation");

	// Retrieve result from device
	CUDA_CHECK(hipMemcpy(opts.TOut, dTOut, memSize, hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.missingValuesCount, dMissingValuesCount, sizeof(int), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(&opts.totalValuesCount, dTotalValuesCount, sizeof(int), hipMemcpyDeviceToHost));

	CUDA_CHECK(hipFree(dT));
	CUDA_CHECK(hipFree(dTOut));
	CUDA_CHECK(hipFree(dMissingValuesCount));
	CUDA_CHECK(hipFree(dTotalValuesCount));

}
