#include "hip/hip_runtime.h"
/**
 * @file simple_packed.cu
 *
 * @date Aug 27, 2013
 * @author partio
 */

#include "simple_packed.h"

#include "cuda_helper.h"

using namespace himan;

__host__
double* simple_packed::Unpack(hipStream_t* stream)
{
	if (!packedLength)
	{
		return 0;
	}

	// We need to create a stream if no stream is specified since dereferencing
	// a null pointer is, well, not a good thing.

	bool destroyStream = false;
	
	if (!stream)
	{
		stream = new hipStream_t;
		CUDA_CHECK(hipStreamCreate(stream));
		destroyStream = true;
	}

	int blockSize = 512;
	int gridSize = unpackedLength / blockSize + (unpackedLength % blockSize == 0 ? 0 : 1);

	double*			d_u = 0; // device-unpacked data
	unsigned char*	d_p = 0; // device-packed data
	int*			d_b = 0; // device-bitmap

	CUDA_CHECK(hipMalloc((void**) (&d_u), unpackedLength * sizeof(double)));

	CUDA_CHECK(hipMalloc((void**) (&d_p), packedLength * sizeof(unsigned char)));
	CUDA_CHECK(hipMemcpyAsync(d_p, data, packedLength * sizeof(unsigned char), hipMemcpyHostToDevice, *stream));

	if (HasBitmap())
	{
		CUDA_CHECK(hipMalloc((void**) (&d_b), bitmapLength * sizeof(int)));
		CUDA_CHECK(hipMemcpyAsync(d_b, bitmap, bitmapLength * sizeof(int), hipMemcpyHostToDevice, *stream));
		CUDA_CHECK(hipStreamSynchronize(*stream));
	}

	simple_packed_util::Unpack <<< gridSize, blockSize, 0, *stream >>> (d_p, d_u, d_b, coefficients, HasBitmap(), unpackedLength);

	CUDA_CHECK(hipFree(d_p));

	if (HasBitmap())
	{
		CUDA_CHECK(hipFree(d_b));
	}

	if (destroyStream)
	{
		CUDA_CHECK(hipStreamDestroy(*stream));
		delete stream;
	}

	return d_u;

}

__global__
void simple_packed_util::Unpack(unsigned char* d_p, double* d_u, int* d_b, simple_packed_coefficients coeff, bool hasBitmap, size_t N)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < N)
	{
		if (coeff.bitsPerValue % 8) // modulo is expensive but "Compiler will convert literal power-of-2 divides to bitwise shifts"
		{
			UnpackUnevenBytes(d_p, d_u, d_b, coeff, hasBitmap, idx);
		}
		else
		{
			UnpackFullBytes(d_p, d_u, d_b, coeff, hasBitmap, idx);
		}
	}
}

__device__
void simple_packed_util::GetBitValue(unsigned char* p, long bitp, int *val)
{
	p += (bitp >> 3);
	*val = (*p&(1<<(7-(bitp%8))));
}

__device__
void simple_packed_util::UnpackFullBytes(unsigned char* __restrict__ d_p, double* __restrict__ d_u, int* __restrict__ d_b, simple_packed_coefficients coeff, bool hasBitmap, int idx)
{
	int bc;
	unsigned long lvalue;

	int l = coeff.bitsPerValue / 8;

	int bm = idx;
	int value_found = 1;

	if (hasBitmap)
	{
		bm = d_b[idx];

		if (bm == 0)
		{
			d_u[idx] = kFloatMissing;
			value_found = 0;
		}
		else
		{
			bm--;
		}
	}

	if (value_found)
	{
		size_t o = bm*l;

		lvalue	= 0;
		lvalue	<<= 8;
		lvalue |= d_p[o++] ;

		for ( bc=1; bc<l; bc++ )
		{
			lvalue <<= 8;
			lvalue |= d_p[o++] ;
		}

		d_u[idx] = ((lvalue * coeff.binaryScaleFactor) + coeff.referenceValue) * coeff.decimalScaleFactor;
	}
}

__device__
void simple_packed_util::UnpackUnevenBytes(unsigned char* __restrict__ d_p, double* __restrict__ d_u, int* __restrict__ d_b, simple_packed_coefficients coeff, bool hasBitmap, int idx)
{
	int j=0;
	unsigned long lvalue;

	int bm = idx;
	int value_found = 1;

	/*
	 * Check if bitmap is set.
	 * If bitmap is set and indicates that value for this element is missing, do
	 * not proceed to calculating phase.
	 *
	 * If bitmap is set and indicates that value exists for this element, the index
	 * for the actual data is the one indicated by the bitmap array. From this index
	 * we reduce one (1) because that one is added to the value in unpack_bitmap.
	 */

	if (hasBitmap)
	{
		bm = d_b[idx];

		if (bm == 0)
		{
			d_u[idx] = kFloatMissing;
			value_found = 0;
		}
		else
		{
			bm--;
		}
	}

	if (value_found)
	{
		long bitp=coeff.bitsPerValue*bm;

		lvalue=0;

		for(j=0; j< coeff.bitsPerValue; j++)
		{
			lvalue <<= 1;
			int val;

			GetBitValue(d_p, bitp, &val);

			if (val) lvalue += 1;

			bitp += 1;
		}

		d_u[idx] = ((lvalue * coeff.binaryScaleFactor) + coeff.referenceValue) * coeff.decimalScaleFactor;
	}

}