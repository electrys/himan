#include "hip/hip_runtime.h"
#include "cuda_plugin_helper.h"
#include "info_simple.h"
#include "interpolate.h"
#include "numerical_functions.h"
#include <thrust/sort.h>

#include "stereographic_grid.h"

// these functions are defined in lambert_conformal_grid.cpp
extern double GetStandardParallel(himan::grid* g, int parallelno);
extern double GetOrientation(himan::grid* g);

const double kEpsilon = 1e-6;
using himan::IsMissingDouble;

struct point
{
	double x;
	double y;

	__host__ __device__ point() : x(himan::MissingDouble()), y(himan::MissingDouble()) {}
	__host__ __device__ point(double _x, double _y) : x(_x), y(_y) {}
};

__host__ __device__ unsigned int Index(unsigned int x, unsigned int y, unsigned int sx) { return y * sx + x; }
__host__ __device__ unsigned int Index(point p, unsigned int sx)
{
	return Index(static_cast<unsigned int>(p.x), static_cast<unsigned int>(p.y), sx);
}

__global__ void Swap(double* __restrict__ arr, size_t ni, size_t nj)
{
	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	// Flip with regards to x axis

	if (idx < nj * ni * 0.5)
	{
		const int i = fmod(static_cast<double>(idx), static_cast<double>(ni));
		const int j = floor(static_cast<double>(idx / ni));

		double upper = arr[idx];
		double lower = arr[Index(i, nj - 1 - j, ni)];

		arr[idx] = lower;
		arr[Index(i, nj - 1 - j, ni)] = upper;
	}
}

void CreateGrid(himan::info& sourceInfo, himan::info& targetInfo, ::point* grid)
{
	targetInfo.ResetLocation();

	int i = 0;

	while (targetInfo.NextLocation())
	{
		himan::point gp = sourceInfo.Grid()->XY(targetInfo.LatLon());

		grid[i].x = gp.X();
		grid[i].y = gp.Y();
		i++;
	}
}

__device__ double Mode(double* arr)
{
	thrust::sort(thrust::seq, arr, arr + 4);

	double num = arr[0];
	double mode = himan::MissingDouble();

	int count = 1;
	int modeCount = 0;

	bool multiModal = false;

	for (int i = 1; i < 4; i++)
	{
		double val = arr[i];

		if (fabs(val - num) < kEpsilon)
		{
			// increase occurrences for this number
			count++;

			if (count == modeCount)
			{
				multiModal = true;
			}
			else if (count > modeCount)
			{
				modeCount = count;
				mode = num;
				multiModal = false;
			}
		}
		else
		{
			// value changed
			count = 1;
			num = val;
		}
	}

	double ret = himan::MissingDouble();

	if (!multiModal)
	{
		ret = mode;
	}

	return ret;
}

__device__ bool IsInsideGrid(point& gp, size_t size_x, size_t size_y)
{
	// if interpolated grid points are negative, it means that we are outside the grid

	// sometime first grid point is -0, so we subtract a small value from first
	// grid point accept that value as well

	if (gp.x >= (0 - kEpsilon) && gp.y >= (0 - kEpsilon) &&

	    // if interpolated grid points are larger than source grid in x or y
	    // direction, it means again that we are outside of the area

	    ((fabs(gp.x - (size_x - 1)) < kEpsilon || __double2uint_ru(gp.x) < size_x) &&
	     (fabs(gp.y - (size_y - 1)) < kEpsilon || __double2uint_ru(gp.y) < size_y)))
	{
		return true;
	}

#ifdef EXTRADEBUG
	bool lc = gp.x >= (0 - kEpsilon) && gp.y >= (0 - kEpsilon);
	bool uc = (fabs(gp.x - (size_x - 1)) < kEpsilon || __double2uint_ru(gp.x) < size_x) &&
	          (fabs(gp.y - (size_y - 1)) < kEpsilon || __double2uint_ru(gp.y) < size_y);

	printf("gp x:%f y:%f discarded [%ld,%ld]: lower cond --> x:%d y:%d upper cond x:%d y:%d\n", gp.x, gp.y, size_x,
	       size_y, gp.x >= (0 - kEpsilon), gp.y >= (0 - kEpsilon), lc, uc);

#endif

	return false;
}

__device__ double NearestPointInterpolation(const double* __restrict__ d_source, himan::info_simple& sourceInfo,
                                            const point& gp)
{
	int rx = rint(gp.x);
	int ry = rint(gp.y);

	assert(rx >= 0 && rx <= sourceInfo.size_x);
	assert(ry >= 0 && ry <= sourceInfo.size_y);

	double npValue = d_source[Index(rx, ry, sourceInfo.size_x)];

	// Sometimes nearest point value is missing, but there is another point almost as close that
	// is not missing. Should we try to use that instead? This would mean that the interpolation
	// result would in some cases contain less missing values, but the cost is an extra branch
	// for *every* nearest point interpolation ever done!

	return npValue;
}

__device__ double BiLinearInterpolation(const double* __restrict__ d_source, himan::info_simple& sourceInfo,
                                        const point& gp)
{
	double ret = himan::MissingDouble();

	// Find all four neighboring points

	point a(floor(gp.x), ceil(gp.y));
	point b(ceil(gp.x), ceil(gp.y));
	point c(floor(gp.x), floor(gp.y));
	point d(ceil(gp.x), floor(gp.y));

	// Assure neighboring points are inside grid and get values

	size_t size_x = sourceInfo.size_x;
	size_t size_y = sourceInfo.size_y;

	double av = himan::MissingDouble(), bv = himan::MissingDouble(), cv = himan::MissingDouble(), dv = himan::MissingDouble();

	if (IsInsideGrid(a, size_x, size_y))
	{
		av = d_source[Index(a, size_x)];
	}
	if (IsInsideGrid(b, size_x, size_y))
	{
		bv = d_source[Index(b, size_x)];
	}
	if (IsInsideGrid(c, size_x, size_y))
	{
		cv = d_source[Index(c, size_x)];
	}
	if (IsInsideGrid(d, size_x, size_y))
	{
		dv = d_source[Index(d, size_x)];
	}

	// Distance of interpolated point to neighboring points

	point dist(gp.x - c.x, gp.y - c.y);

	assert(dist.x >= 0 && dist.x <= 1);
	assert(dist.y >= 0 && dist.y <= 1);

	// If interpolated point is very close to source grid point, pick
	// the point value directly

	// This is preferred since nearest point is faster than bilinear, and
	// if wanted grid point =~ source grid point, the bilinear interpolation
	// value will be very close to nearest point value

	using namespace himan::numerical_functions::interpolation;

	if ((dist.x < kEpsilon || fabs(dist.x - 1) < kEpsilon) && (dist.y < kEpsilon || fabs(dist.y - 1) < kEpsilon))
	{
		ret = NearestPointInterpolation(d_source, sourceInfo, gp);
	}

	// All values present, regular bilinear interpolation

	else if (!IsMissingDouble(av) && !IsMissingDouble(bv) && !IsMissingDouble(cv) && !IsMissingDouble(dv))
	{
		ret = BiLinear(dist.x, dist.y, av, bv, cv, dv);
	}

	// x or y is at grid edge

	else if (fabs(dist.y) < kEpsilon && !IsMissingDouble(cv) && !IsMissingDouble(dv))
	{
		ret = Linear(dist.x, cv, dv);
	}

	else if (fabs(dist.y - 1) < kEpsilon && !IsMissingDouble(av) && !IsMissingDouble(bv))
	{
		ret = Linear(dist.x, av, bv);
	}

	else if (fabs(dist.x) < kEpsilon && !IsMissingDouble(cv) && !IsMissingDouble(av))
	{
		ret = Linear(dist.y, cv, av);
	}

	else if (fabs(dist.x - 1) < kEpsilon && !IsMissingDouble(av) && !IsMissingDouble(bv))
	{
		ret = Linear(dist.y, dv, bv);
	}

	// One point missing; these "triangulation" methods have been copied from NFmiInterpolation.cpp

	else if (IsMissingDouble(av) && !IsMissingDouble(bv) && !IsMissingDouble(cv) && !IsMissingDouble(dv))
	{
		double wsum = (dist.x * dist.y + (1 - dist.x) * (1 - dist.y) + dist.x * (1 - dist.y));

		ret = ((1 - dist.x) * (1 - dist.y) * cv + dist.x * (1 - dist.y) * dv + dist.x * dist.y * bv) / wsum;
	}
	else if (!IsMissingDouble(av) && IsMissingDouble(bv) && !IsMissingDouble(cv) && !IsMissingDouble(dv))
	{
		double wsum = ((1 - dist.x) * dist.y + (1 - dist.x) * (1 - dist.y) + dist.x * (1 - dist.y));

		ret = ((1 - dist.x) * (1 - dist.y) * cv + dist.x * (1 - dist.y) * dv + (1 - dist.x) * dist.y * av) / wsum;
	}
	else if (!IsMissingDouble(av) && !IsMissingDouble(bv) && IsMissingDouble(cv) && !IsMissingDouble(dv))
	{
		double wsum = ((1 - dist.x) * dist.y + dist.x * dist.y + dist.x * (1 - dist.y));

		ret = (dist.x * (1 - dist.y) * dv + (1 - dist.x) * dist.y * av + dist.x * dist.y * bv) / wsum;
	}
	else if (!IsMissingDouble(av) && !IsMissingDouble(bv) && !IsMissingDouble(cv) && IsMissingDouble(dv))
	{
		double wsum = ((1 - dist.x) * (1 - dist.y) + (1 - dist.x) * dist.y + dist.x * dist.y);

		ret = ((1 - dist.x) * (1 - dist.y) * cv + (1 - dist.x) * dist.y * av + dist.x * dist.y * bv) / wsum;
	}

#ifdef EXTRADEBUG
	else
	{
		printf("More than one point missing for gp x: %f y:%f --> a:%f b:%f c:%f d:%f | dx:%f dy:%f\n", gp.x, gp.y, av,
		       bv, cv, dv, dist.x, dist.y);
	}

	if ((ret != ret))
	{
		printf("gpx:%f gpy:%f [%ld %ld] |  dist x:%f y:%f\n", gp.x, gp.y, size_x, size_y, dist.x, dist.y);
		printf("av:%f bv:%f cv:%f dv:%f | interp:%f\n", av, bv, cv, dv, ret);
		printf("ax:%f ay:%f bx:%f by:%f cx:%f cy:%f dx:%f dy:%f\n", a.x, a.y, b.x, b.y, c.x, c.y, d.x, d.y);
		printf("is inside grid: a:%d b:%d c:%d d:%d\n", IsInsideGrid(a, size_x, size_y),
		       IsInsideGrid(b, size_x, size_y), IsInsideGrid(c, size_x, size_y), IsInsideGrid(d, size_x, size_y));
	}

#endif

	return ret;
}

__device__ double NearestPointValueInterpolation(const double* __restrict__ d_source, himan::info_simple& sourceInfo,
                                                 const point& gp)
{
	double ret = himan::MissingDouble();

	// Find all four neighboring points

	point a(floor(gp.x), ceil(gp.y));
	point b(ceil(gp.x), ceil(gp.y));
	point c(floor(gp.x), floor(gp.y));
	point d(ceil(gp.x), floor(gp.y));

	// Assure neighboring points are inside grid

	size_t size_x = sourceInfo.size_x;
	size_t size_y = sourceInfo.size_y;

	if (!IsInsideGrid(a, size_x, size_y) || !IsInsideGrid(b, size_x, size_y) || !IsInsideGrid(c, size_x, size_y) ||
	    !IsInsideGrid(d, size_x, size_y))
	{
		return ret;
	}

	// Neighbor values

	double av = d_source[Index(a, size_x)];
	double bv = d_source[Index(b, size_x)];
	double cv = d_source[Index(c, size_x)];
	double dv = d_source[Index(d, size_x)];

	// Find mode of neighboring points

	double arr[4] = {av, bv, cv, dv};
	double mode = Mode(arr);

	if (!IsMissingDouble(mode))
	{
		return mode;
	}

	double bilin = BiLinearInterpolation(d_source, sourceInfo, gp);

	arr[0] = fabs(av - bilin);
	arr[1] = fabs(bv - bilin);
	arr[2] = fabs(cv - bilin);
	arr[3] = fabs(dv - bilin);

	mode = Mode(arr);

	if (!IsMissingDouble(mode))
	{
		double min = fmin(arr[0], fmin(arr[1], fmin(arr[2], arr[3])));

		if (fabs(mode - min) < kEpsilon)
		{
			ret = bilin - mode;
		}
		else
		{
			ret = bilin - min;
		}
	}
	else
	{
		// no mode
		double min = fmin(arr[0], fmin(arr[1], fmin(arr[2], arr[3])));
		ret = bilin - min;
	}

	return ret;
}

__global__ void InterpolateCudaKernel(const double* __restrict__ d_source, double* __restrict__ d_target,
                                      const point* __restrict__ d_grid, himan::info_simple sourceInfo,
                                      himan::info_simple targetInfo)
{
	// idx is our pointer to the TARGET data in linear format

	const int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < targetInfo.size_x * targetInfo.size_y)
	{
		// next we need to get x and y of the 'idx' in the source grid coordinates
		// to do that we first determine the i and j of the target grid coordinates

		const int i = fmod(static_cast<double>(idx), static_cast<double>(targetInfo.size_x));
		const int j = floor(static_cast<double>(idx / targetInfo.size_x));

		// with i and j we can get the grid point coordinates in the source grid

		point gp = d_grid[Index(i, j, targetInfo.size_x)];

		double interp = himan::MissingDouble();

		if (IsInsideGrid(gp, sourceInfo.size_x, sourceInfo.size_y))
		{
			// targetInfo.interpolation = himan::kNearestPointValue;

			switch (targetInfo.interpolation)
			{
				case himan::kBiLinear:
					interp = BiLinearInterpolation(d_source, sourceInfo, gp);
					break;

				case himan::kNearestPoint:
					interp = NearestPointInterpolation(d_source, sourceInfo, gp);
					break;

				case himan::kNearestPointValue:
					interp = NearestPointValueInterpolation(d_source, sourceInfo, gp);
					break;
			}
		}
#ifdef EXTRADEBUG
		else
		{
			printf("grid point x:%f y:%f discarded [%ld,%ld]\n", gp.x, gp.y, sourceInfo.size_x, sourceInfo.size_y);
		}
#endif
		d_target[idx] = interp;

		assert(interp == interp || IsMissingDouble(interp));  // no NaN
		assert(interp < 1e30 || IsMissingDouble(interp));     // No crazy values
	}
}

bool InterpolateAreaGPU(himan::info& base, himan::info& source, himan::matrix<double>& targetData)
{
	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	if (base.Param().InterpolationMethod() == himan::kUnknownInterpolationMethod)
	{
		base.Param().InterpolationMethod(himan::kBiLinear);
	}
	else
	{
		auto newMethod =
		    himan::interpolate::InterpolationMethod(source.Param().Name(), base.Param().InterpolationMethod());
		auto newParam = base.Param();
		newParam.InterpolationMethod(newMethod);

		base.SetParam(newParam);
	}

	// Determine all grid point coordinates that need to be interpolated.
	const size_t N = base.SizeLocations();

	point* grid_ = 0;
	point* d_grid = 0;

	CUDA_CHECK(hipHostMalloc((void**)&grid_, N * sizeof(::point)));

	CreateGrid(source, base, grid_);

	CUDA_CHECK(hipMalloc((void**)&d_grid, sizeof(::point) * N));
	CUDA_CHECK(hipMemcpyAsync(d_grid, grid_, sizeof(::point) * N, hipMemcpyHostToDevice, stream));

	double* d_source = 0;
	double* d_target = 0;

	CUDA_CHECK(hipMalloc((void**)&d_source, source.SizeLocations() * sizeof(double)));
	CUDA_CHECK(hipMalloc((void**)&d_target, N * sizeof(double)));

	auto sourceInfo = source.ToSimple();
	auto targetInfo = base.ToSimple();
	targetInfo->values = targetData.ValuesAsPOD();

	assert(targetInfo->values);

	PrepareInfo(sourceInfo, d_source, stream);
	PrepareInfo(targetInfo);

	const int bs = 256;
	const int gs = N / bs + (N % bs == 0 ? 0 : 1);

	InterpolateCudaKernel<<<gs, bs, 0, stream>>>(d_source, d_target, d_grid, *sourceInfo, *targetInfo);

	CUDA_CHECK(hipStreamSynchronize(stream));

	CUDA_CHECK(hipHostFree(grid_));
	himan::ReleaseInfo(sourceInfo);

	himan::ReleaseInfo(targetInfo, d_target, stream);

	CUDA_CHECK(hipFree(d_source));
	CUDA_CHECK(hipFree(d_target));
	CUDA_CHECK(hipFree(d_grid));

	CUDA_CHECK(hipStreamDestroy(stream));

	return true;
}

__global__ void RotateLambert(double* __restrict__ d_u, double* __restrict__ d_v, const double* __restrict__ d_lon,
                              double cone, double orientation, himan::info_simple opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.size_x * opts.size_y)
	{
		double U = d_u[idx];
		double V = d_v[idx];

		int i = fmod(static_cast<double>(idx), static_cast<double>(opts.size_x));
		int j = floor(static_cast<double>(idx / opts.size_x));

		double londiff = d_lon[idx] - orientation;
		const double angle = cone * londiff * himan::constants::kDeg;
		double sinx, cosx;
		sincos(angle, &sinx, &cosx);
		d_u[idx] = cosx * U + sinx * V;
		d_v[idx] = -1 * sinx * U + cosx * V;
	}
}

__global__ void RotateRotatedLatitudeLongitude(double* __restrict__ d_u, double* __restrict__ d_v,
                                               himan::info_simple opts)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < opts.size_x * opts.size_y)
	{
		double U = d_u[idx];
		double V = d_v[idx];

		// Rotated to regular coordinates

		int i = fmod(static_cast<double>(idx), static_cast<double>(opts.size_x));  // idx - j * opts.size_x;
		int j = floor(static_cast<double>(idx / opts.size_x));

		double lon = opts.first_lon + i * opts.di;

		double lat = himan::MissingDouble();

		if (opts.j_scans_positive)
		{
			lat = opts.first_lat + j * opts.dj;
		}
		else
		{
			lat = opts.first_lat - j * opts.dj;
		}

		double SinYPole = sin((opts.south_pole_lat + 90.) * himan::constants::kDeg);
		double CosYPole = cos((opts.south_pole_lat + 90.) * himan::constants::kDeg);

		double SinXRot, CosXRot, SinYRot, CosYRot;

		sincos(lon * himan::constants::kDeg, &SinXRot, &CosXRot);
		sincos(lat * himan::constants::kDeg, &SinYRot, &CosYRot);

		double SinYReg = CosYPole * SinYRot + SinYPole * CosYRot * CosXRot;

		SinYReg = fmin(fmax(SinYReg, -1.), 1.);

		double YReg = asin(SinYReg) * himan::constants::kRad;

		double CosYReg = cos(YReg * himan::constants::kDeg);

		double CosXReg = (CosYPole * CosYRot * CosXRot - SinYPole * SinYRot) / CosYReg;

		CosXReg = fmin(fmax(CosXReg, -1.), 1.);
		double SinXReg = CosYRot * SinXRot / CosYReg;

		double XReg = acos(CosXReg) * himan::constants::kRad;

		if (SinXReg < 0.)
		{
			XReg = -XReg;
		}
		XReg += opts.south_pole_lon;

		// UV to earth relative

		double zxmxc = himan::constants::kDeg * (XReg - opts.south_pole_lon);

		double sinxmxc, cosxmxc;

		sincos(zxmxc, &sinxmxc, &cosxmxc);

		double PA = cosxmxc * CosXRot + CosYPole * sinxmxc * SinXRot;
		double PB =
		    CosYPole * sinxmxc * CosXRot * SinYRot + SinYPole * sinxmxc * CosYRot - cosxmxc * SinXRot * SinYRot;
		double PC = (-SinYPole) * SinXRot / CosYReg;
		double PD = (CosYPole * CosYRot - SinYPole * CosXRot * SinYRot) / CosYReg;

		double newU = PA * U + PB * V;
		double newV = PC * U + PD * V;

		d_u[idx] = newU;
		d_v[idx] = newV;
	}
}

void RotateVectorComponentsGPU(himan::info& UInfo, himan::info& VInfo)
{
	const size_t N = UInfo.SizeLocations();
	const int bs = 256;
	const int gs = N / bs + (N % bs == 0 ? 0 : 1);

	hipStream_t stream;
	CUDA_CHECK(hipStreamCreate(&stream));

	double* d_u = 0;
	double* d_v = 0;
	double* d_lon = 0;

	CUDA_CHECK(hipMalloc((void**)&d_u, N * sizeof(double)));
	CUDA_CHECK(hipMalloc((void**)&d_v, N * sizeof(double)));

	auto USimple = UInfo.ToSimple();
	auto VSimple = VInfo.ToSimple();

	PrepareInfo(USimple, d_u, stream);
	PrepareInfo(VSimple, d_v, stream);

	switch (UInfo.Grid()->Type())
	{
		case himan::kRotatedLatitudeLongitude:
			RotateRotatedLatitudeLongitude<<<gs, bs, 0, stream>>>(d_u, d_v, *USimple);
			break;

		case himan::kLambertConformalConic:
		{
			CUDA_CHECK(hipMalloc((void**)&d_lon, N * sizeof(double)));

			double* lon = 0;

			CUDA_CHECK(hipHostMalloc((void**)&lon, N * sizeof(double)));

			for (UInfo.ResetLocation(); UInfo.NextLocation();)
			{
				lon[UInfo.LocationIndex()] = UInfo.LatLon().X();
			}

			CUDA_CHECK(hipMemcpyAsync(d_lon, lon, N * sizeof(double), hipMemcpyHostToDevice));

			const double latin1 = GetStandardParallel(UInfo.Grid(), 1);
			const double latin2 = GetStandardParallel(UInfo.Grid(), 2);
			const double orientation = GetOrientation(UInfo.Grid());

			assert(!himan::IsKHPMissingValue(latin1) && !himan::IsKHPMissingValue(orientation));
			double cone;

			using himan::constants::kDeg;

			if (latin1 == latin2)
			{
				cone = sin(fabs(latin1) * kDeg);
			}
			else
			{
				cone = (log(cos(latin1 * kDeg)) - log(cos(latin2 * kDeg))) /
				       (log(tan((90 - fabs(latin1)) * kDeg * 0.5)) - log(tan(90 - fabs(latin2)) * kDeg * 0.5));
			}

			RotateLambert<<<gs, bs, 0, stream>>>(d_u, d_v, d_lon, cone, orientation, *USimple);

			CUDA_CHECK(hipStreamSynchronize(stream));
			CUDA_CHECK(hipHostFree(lon));
		}
		break;

		case himan::kStereographic:
		{
			const double orientation = dynamic_cast<himan::stereographic_grid*>(UInfo.Grid())->Orientation();
			CUDA_CHECK(hipMalloc((void**)&d_lon, N * sizeof(double)));

			double* lon = 0;

			CUDA_CHECK(hipHostMalloc((void**)&lon, N * sizeof(double)));

			for (UInfo.ResetLocation(); UInfo.NextLocation();)
			{
				lon[UInfo.LocationIndex()] = UInfo.LatLon().X();
			}

			CUDA_CHECK(hipMemcpyAsync(d_lon, lon, N * sizeof(double), hipMemcpyHostToDevice));

			RotateLambert<<<gs, bs, 0, stream>>>(d_u, d_v, d_lon, 1, orientation, *USimple);

			CUDA_CHECK(hipStreamSynchronize(stream));
			CUDA_CHECK(hipHostFree(lon));
		}
		break;

		default:
			break;
	}
	CUDA_CHECK(hipStreamSynchronize(stream));

	himan::ReleaseInfo(USimple, d_u, stream);
	himan::ReleaseInfo(VSimple, d_v, stream);

	CUDA_CHECK(hipFree(d_u));
	CUDA_CHECK(hipFree(d_v));

	if (d_lon)
	{
		CUDA_CHECK(hipFree(d_lon));
	}

	CUDA_CHECK(hipStreamDestroy(stream));

	if (UInfo.Grid()->IsPackedData())
	{
		UInfo.Grid()->PackedData().Clear();
	}
	if (VInfo.Grid()->IsPackedData())
	{
		VInfo.Grid()->PackedData().Clear();
	}
}
